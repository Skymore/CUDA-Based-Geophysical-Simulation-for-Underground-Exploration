#include "hip/hip_runtime.h"
/************************************************************************************
* Author: Tao Rui
* 版本: V1.0 单卡，Linux版
* 说明:
*		计算第二部分的并行。
************************************************************************************/
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <string.h>
#include "unistd.h"
#include "global_variables.cpp"
#include <unistd.h>  //linux
//#include <direct.h>  //windows


__global__ void print_dev_matrix(float *A, int i, int j, int k, int xdim, int ydim, int zdim)
{
	int	idx = i * ydim*zdim + j * zdim + k;
	printf("dev_Matrix[%d][%d][%d] = %8f\n", i, j, k, A[idx]);
}
/************************************************************************************
* GPU计算单个矩阵的函数
************************************************************************************/
dim3 gridUHyz(npml, nx - 1); 
dim3 blockUHyz(nz);
__global__ void gpu_UHyz(float *UHyz, float *RBHyz, float *RAHyz, float *Ez)
{
	/*
	in0 UHyz  nx+1 ny     nz
	in1 RBHyz nx-1 2*npml nz
	in2 RAHyz nx-1 2*npml nz
	in3 Ez    nx+1  ny+1  nz
	UHyz = UHyz * RBHyz + RAHyz * (Ez - Ez) / dy
	运算块大小 nx-1 * npml * nz
	UHyz由5个矩阵相乘或相加得来。
	y维分为了两块

	UHyz(2:nx, [1:npml ny-npml+1:ny], :)=RBHyz .* UHyz(2:nx, [1:npml ny-npml+1:ny], :)...
	+RAHyz ./ dy .* (Ez(2:nx, [2:npml+1 ny-npml+2:ny+1], :) - Ez(2:nx, [1:npml ny-npml+1:ny], :));
	*/

	int ix = blockIdx.y;   // ix in [0, nx - 1)
	int iy = blockIdx.x;   // iy in [0, npml)
	int iz = threadIdx.x;  // iz in [0, nz)

	int lid0 = (ix + 1)*ny*nz + iy * nz + iz; // checked!
	int rid0 = (ix + 1)*ny*nz + (iy + ny - npml) * nz + iz;  //checked!

	int lid1 = ix * (2 * npml)*nz + iy * nz + iz; // checked!
	int rid1 = ix * (2 * npml)*nz + (iy + npml) * nz + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid3 = (ix + 1)*(ny + 1)*nz + (iy + 1)*nz + iz; // checked!
	int rid3 = (ix + 1)*(ny + 1)*nz + (iy + ny - npml + 1)*nz + iz; // checked!

	int lid4 = (ix + 1)*(ny + 1)*nz + iy * nz + iz; // checked!
	int rid4 = (ix + 1)*(ny + 1)*nz + (iy + ny - npml)*nz + iz; // checked!

	UHyz[lid0] = UHyz[lid0] * RBHyz[lid1] + RAHyz[lid2] * (Ez[lid3] - Ez[lid4]) / dy;
	UHyz[rid0] = UHyz[rid0] * RBHyz[rid1] + RAHyz[rid2] * (Ez[rid3] - Ez[rid4]) / dy;
}
dim3 gridUHzy(nx - 1, ny);
dim3 blockUHzy(npml);
__global__ void gpu_UHzy(float *UHzy, float *RBHzy, float *RAHzy, float *Ey)
{
	/*
	in0 UHzy  --size--  nx+1  ny  nz
	in1 RBHzy --size--  nx-1  ny  2*npml
	in2 RAHzy --size--  nx-1  ny  2*npml
	in3 Ey    --size--  nx+1  ny  nz+1
	UHyz = UHyz * RBHyz + RAHyz * (Ez - Ez) / dy
	运算块大小 nx-1 * ny * (5 *npml)
	UHyz由5个矩阵相乘或相加得来。
	z维分为了两块
	UHzy(2:nx, :, [1:npml nz-npml+1:nz])=RBHzy.*UHzy(2:nx, :, [1:npml nz-npml+1:nz])
	+RAHzy./dz.*(Ey(2:nx, :, [2:npml+1 nz-npml+2:nz+1])-Ey(2:nx, :, [1:npml nz-npml+1:nz]));
	*/

	int ix = blockIdx.x;  // ix in [0, nx - 1)
	int iy = blockIdx.y;  // iy in [0, ny)
	int iz = threadIdx.x; // ix in [0, npml)

	int lid0 = (ix + 1) * ny * nz + iy * nz + iz; //checked!
	int rid0 = (ix + 1) * ny * nz + iy * nz + iz + nz - npml; //checked!

	int lid1 = ix * ny * (2 * npml) + iy * (2 * npml) + iz; //checked!
	int rid1 = ix * ny * (2 * npml) + iy * (2 * npml) + iz + npml; //checked!

	int lid2 = lid1;
	int rid2 = rid1;

	int lid4 = (ix + 1) * ny * (nz + 1) + iy * (nz + 1) + iz; //checked!
	int rid4 = (ix + 1) * ny * (nz + 1) + iy * (nz + 1) + iz + nz - npml; //checked!

	int lid3 = lid4 + 1;
	int rid3 = rid4 + 1;

	UHzy[lid0] = UHzy[lid0] * RBHzy[lid1] + RAHzy[lid2] * (Ey[lid3] - Ey[lid4]) / dz;
	UHzy[rid0] = UHzy[rid0] * RBHzy[rid1] + RAHzy[rid2] * (Ey[rid3] - Ey[rid4]) / dz;
}
dim3 gridUHzx(nx, ny - 1);
dim3 blockUHzx(npml);
__global__ void gpu_UHzx(float *UHzx, float *RBHzx, float *RAHzx, float *Ex)
{
	/*
	in0 UHzx  --size--  nx   ny + 1  nz
	in1 RBHzx --size--  nx   ny - 1  2 * npml
	in2 RAHzx --size--  nx   ny - 1  2 * npml
	in3 Ex    --size--  nx   ny + 1  nz + 1
	UHzx = UHzx * RBHzx + RAHzx * (Ez - Ez) / dy
	运算块大小 nx * ny - 1 * npml
	UHzx由5个矩阵相乘或相加得来。
	z维分为了两块  1:npml    -npml:0
	UHzx(:, 2:ny, [1:npml nz - npml + 1:nz])=RBHzx. * UHzx(:, 2:ny, [1:npml nz - npml + 1:nz])
	+RAHzx./dz.*(Ex(:, 2:ny, [2:npml + 1 nz - npml + 2:nz + 1]) - Ex(:, 2:ny, [1:npml nz - npml + 1:nz]));
	*/

	int ix = blockIdx.x;  // ix in [0, nx)
	int iy = blockIdx.y;  // iy in [0, ny - 1)
	int iz = threadIdx.x; // iz in [0, npml)

	int lid0 = ix * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid0 = ix * (ny + 1) * nz + (iy + 1) * nz + iz + nz - npml; // checked!

	int lid1 = ix * (ny - 1) * (2 * npml) + iy * (2 * npml) + iz; // checked!
	int rid1 = ix * (ny - 1) * (2 * npml) + iy * (2 * npml) + iz + npml; // checked!

	int lid2 = lid1;
	int rid2 = rid1;

	int lid4 = ix * (ny + 1) * (nz + 1) + (iy + 1) * (nz + 1) + iz; // checked!
	int rid4 = ix * (ny + 1) * (nz + 1) + (iy + 1) * (nz + 1) + iz + nz - npml; // checked!

	int lid3 = lid4 + 1;
	int rid3 = rid4 + 1;

	UHzx[lid0] = UHzx[lid0] * RBHzx[lid1] + RAHzx[lid2] * (Ex[lid3] - Ex[lid4]) / dz;
	UHzx[rid0] = UHzx[rid0] * RBHzx[rid1] + RAHzx[rid2] * (Ex[rid3] - Ex[rid4]) / dz;
}
dim3 gridUHxz(npml, ny - 1);
dim3 blockUHxz(nz);
__global__ void gpu_UHxz(float *UHxz, float *RBHxz, float *RAHxz, float *Ez)
{
	/*
	in0 UHxz  --size--  nx       ny + 1  nz
	in1 RBHxz --size--  2*npml   ny - 1  nz
	in2 RAHxz --size--  2*npml   ny - 1  nz
	in3 Ez    --size--  nx + 1   ny + 1  nz
	UHxz = UHxz * RBHxz + RAHxz * (Ez - Ez) / dx
	运算块大小 npml * ny - 1 * nz
	UHxz由5个矩阵相乘或相加得来。
	x维分为了两块  1:npml    -npml:0
	UHxz([1:npml nx-npml+1:nx], 2:ny, :)=RBHxz.*UHxz([1:npml nx-npml+1:nx], 2:ny, :)...
	+RAHxz./dx.*(Ez([2:npml+1 nx-npml+2:nx+1], 2:ny, :)-Ez([1:npml nx-npml+1:nx], 2:ny, :));
	*/
	int ix = blockIdx.x;  // ix in [0, npml)
	int iy = blockIdx.y;  // iy in [0, ny - 1)
	int iz = threadIdx.x; // iz in [0, nz)

	int lid0 = ix * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid0 = (ix + nx - npml) * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!

	int lid1 = ix * (ny - 1) * nz + iy * nz + iz; // checked!
	int rid1 = (ix + npml) * (ny - 1) * nz + iy * nz + iz; // checked!

	int lid2 = lid1;
	int rid2 = rid1;

	int lid4 = ix * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid4 = (ix + nx - npml) * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!

	int lid3 = lid4 + (ny + 1) * nz;
	int rid3 = rid4 + (ny + 1) * nz;

	UHxz[lid0] = UHxz[lid0] * RBHxz[lid1] + RAHxz[lid2] * (Ez[lid3] - Ez[lid4]) / dx;
	UHxz[rid0] = UHxz[rid0] * RBHxz[rid1] + RAHxz[rid2] * (Ez[rid3] - Ez[rid4]) / dx;
}
dim3 gridUHxy(npml, ny);
dim3 blockUHxy(nz - 1);
__global__ void gpu_UHxy(float *UHxy, float *RBHxy, float *RAHxy, float *Ey)
{
	/*
	in0 UHxy  --size--  nx       ny      nz + 1
	in1 RBHxy --size--  2*npml   ny      nz - 1
	in2 RAHxy --size--  2*npml   ny      nz - 1
	in3 EY    --size--  nx + 1   ny      nz + 1
	UHxy = UHxy * RBHxy + RAHxy * (Ez - Ez) / dx
	运算块大小 npml * ny * nz - 1
	UHxy由5个矩阵相乘或相加得来。
	x维分为了两块  1:npml    -npml:0
	UHxy([1:npml nx-npml+1:nx], :, 2:nz)=RBHxy.*UHxy([1:npml nx-npml+1:nx], :, 2:nz)...
	+RAHxy./dx.*(Ey([2:npml+1 nx-npml+2:nx+1], :, 2:nz)-Ey([1:npml nx-npml+1:nx], :, 2:nz));
	*/
	int ix = blockIdx.x;  // ix in [0, npml)
	int iy = blockIdx.y;  // iy in [0, ny)
	int iz = threadIdx.x; // iz in [0, nz - 1)

	int lid0 = ix * ny * (nz + 1) + iy * (nz + 1) + iz + 1; // checked!
	int rid0 = (ix + nx - npml) * ny * (nz + 1) + iy * (nz + 1) + iz + 1; //checked

	int lid1 = ix * ny * (nz - 1) + iy * (nz - 1) + iz; // checked!
	int rid1 = (ix + npml) * ny * (nz - 1) + iy * (nz - 1) + iz; // checked!

	int lid2 = lid1;
	int rid2 = rid1;

	int lid4 = ix * ny * (nz + 1) + iy * (nz + 1) + iz + 1; // checked!
	int rid4 = (ix + nx - npml) * ny * (nz + 1) + iy * (nz + 1) + iz + 1; // checked!

	int lid3 = lid4 + ny * (nz + 1);
	int rid3 = rid4 + ny * (nz + 1);

	UHxy[lid0] = UHxy[lid0] * RBHxy[lid1] + RAHxy[lid2] * (Ey[lid3] - Ey[lid4]) / dx;
	UHxy[rid0] = UHxy[rid0] * RBHxy[rid1] + RAHxy[rid2] * (Ey[rid3] - Ey[rid4]) / dx;
}
dim3 gridUHyx(npml, nx);
dim3 blockUHyx(nz - 1);
__global__ void gpu_UHyx(float *UHyx, float *RBHyx, float *RAHyx, float *Ex)
{
	/*
	in0 UHyx  nx   ny     nz + 1
	in1 RBHyx nx   2*npml nz - 1
	in2 RAHyx nx   2*npml nz - 1
	in3 Ex    nx   ny + 1 nz + 1
	UHyx = UHyx * RBHyx + RAHyx * (Ex - Ex) / dy
	运算块大小 nx * npml * nz - 1
	UHyx由5个矩阵相乘或相加得来。
	y维分为了两块

	UHyx(:, [1:npml ny-npml+1:ny], 2:nz)=RBHyx.*UHyx(:, [1:npml ny-npml+1:ny], 2:nz)...
	+RAHyx./dy.*(Ex(:, [2:npml+1 ny-npml+2:ny+1], 2:nz)-Ex(:, [1:npml ny-npml+1:ny], 2:nz));
	*/
	int ix = blockIdx.y;   // ix in [0, nx)
	int iy = blockIdx.x;   // iy in [0, npml)
	int iz = threadIdx.x;  // iz in [0, nz - 1)

	int lid0 = ix * ny * (nz + 1) + iy * (nz + 1) + iz + 1; // checked!
	int rid0 = ix * ny * (nz + 1) + (iy + ny - npml) * (nz + 1) + iz + 1;  //checked!

	int lid1 = ix * (2 * npml) * (nz - 1) + iy * (nz - 1) + iz; // checked!
	int rid1 = ix * (2 * npml) * (nz - 1) + (iy + npml) * (nz - 1) + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!


	int lid4 = ix * (ny + 1) * (nz + 1) + iy * (nz + 1) + iz + 1; // checked!
	int rid4 = ix * (ny + 1) * (nz + 1) + (iy + ny - npml) * (nz + 1) + iz + 1; // checked!


	int lid3 = lid4 + (nz + 1); // checked!
	int rid3 = rid4 + (nz + 1); // checked!

	UHyx[lid0] = UHyx[lid0] * RBHyx[lid1] + RAHyx[lid2] * (Ex[lid3] - Ex[lid4]) / dy;
	UHyx[rid0] = UHyx[rid0] * RBHyx[rid1] + RAHyx[rid2] * (Ex[rid3] - Ex[rid4]) / dy;
}
dim3 gridHx(nx - 1, ny);
dim3 blockHx(nz);
__global__ void gpu_Hx(float *Hx, float *CPHx, float *CQHx, float *ky_Hx, float *kz_Hx, float *Ez, float *Ey, float *UHyz, float *UHzy)
{
	//
	// * 运算块大小 nx - 1 * ny * nz
	// * Hx(2:nx,:,:)
	//
	int ix = blockIdx.x + 1;
	int iy = blockIdx.y;
	int iz = threadIdx.x;

	int idx = ix * ny * nz + iy * nz + iz;
	int idxEz = ix * (ny + 1)*nz + iy * nz + iz;
	int idxEy = ix * ny * (nz + 1) + iy * (nz + 1) + iz;

	int deltaEz = nz;
	int deltaEy = 1;
	float CQH = CQHx[idx];

	Hx[idx] = Hx[idx] * CPHx[idx]
		- CQH / ky_Hx[idx] * (Ez[idxEz + deltaEz] - Ez[idxEz]) / dy
		+ CQH / kz_Hx[idx] * (Ey[idxEy + deltaEy] - Ey[idxEy]) / dz
		- CQH * UHyz[idx]
		+ CQH * UHzy[idx];
}
dim3 gridHy(nx, ny - 1);
dim3 blockHy(nz);
__global__ void gpu_Hy(float *Hy, float *CPHy, float *CQHy, float *kz_Hy, float *kx_Hy, float *Ex, float *Ez, float *UHzx, float *UHxz)
{
	//
	// * 运算块大小 nx * ny -1 * nz
	// * Hy(:,2:ny,:)
	//
	int ix = blockIdx.x;
	int iy = blockIdx.y + 1;
	int iz = threadIdx.x;

	int idx = ix * (ny + 1)*nz + iy * nz + iz;
	int idxEx = ix * (ny + 1)*(nz + 1) + iy * (nz + 1) + iz;
	int idxEz = ix * (ny + 1)*nz + iy * nz + iz;

	int deltaEx = 1;
	int deltaEz = (ny + 1)*nz;
	float CQH = CQHy[idx];

	Hy[idx] = Hy[idx] * CPHy[idx]
		- CQH / kz_Hy[idx] * (Ex[idxEx + deltaEx] - Ex[idxEx]) / dz
		+ CQH / kx_Hy[idx] * (Ez[idxEz + deltaEz] - Ez[idxEz]) / dx
		- CQH * UHzx[idx]
		+ CQH * UHxz[idx];
}
dim3 gridHz(nx, ny);
dim3 blockHz(nz - 1);
__global__ void gpu_Hz(float *Hz, float *CPHz, float *CQHz, float *kx_Hz, float *ky_Hz, float *Ey, float *Ex, float *UHxy, float *UHyx)
{
	//
	// * 运算块大小 nx * ny * nz -1
	// * Hz(:,;,2:nz)
	// * Hz大小为nx ny nz+1
	//
	int ix = blockIdx.x;
	int iy = blockIdx.y;
	int iz = threadIdx.x + 1;

	int idx = ix * ny * (nz + 1) + iy * (nz + 1) + iz;
	int idxEy = ix * ny * (nz + 1) + iy * (nz + 1) + iz;
	int idxEx = ix * (ny + 1) * (nz + 1) + iy * (nz + 1) + iz;
	int deltaEy = ny * (nz + 1);
	int deltaEx = nz + 1;
	float CQH = CQHz[idx];

	Hz[idx] = Hz[idx] * CPHz[idx]
		- CQH / kx_Hz[idx] * (Ey[idxEy + deltaEy] - Ey[idxEy]) / dx
		+ CQH / ky_Hz[idx] * (Ex[idxEx + deltaEx] - Ex[idxEx]) / dy
		- CQH * UHxy[idx]
		+ CQH * UHyx[idx];
}
dim3 gridUEyz(npml - 1, nx);
dim3 blockUEyz(nz - 1);
__global__ void gpu_UEyz(float *UEyz, float *RBEyz, float *RAEyz, float *Hz)
{
	/*
	dim3 blockUEyz(nz - 1);
	dim3 gridUEyz(npml - 1, nx);

	in0 UEyz  nx   ny + 1     nz + 1
	in1 RBEyz nx   2*(npml-1) nz - 1
	in2 RAEyz nx   2*(npml-1) nz - 1
	in3 Hz    nx   ny         nz + 1

	运算块大小 nx * npml - 1 * nz - 1

	UEyz(:, [2:npml ny-npml+2:ny], 2:nz)=RBEyz .* UEyz(:, [2:npml ny-npml+2:ny], 2:nz)...
	+RAEyz ./ dy .* (Hz(:, [2:npml ny-npml+2:ny], 2:nz) - Hz(:, [1:npml-1 ny-npml+1:ny-1], 2:nz));
	*/
	int ix = blockIdx.y;   // ix in [0, nx)
	int iy = blockIdx.x;   // iy in [0, npml - 1)
	int iz = threadIdx.x;  // iz in [0, nz - 1)

	int lid0 = ix * (ny + 1) * (nz + 1) + (iy + 1) * (nz + 1) + (iz + 1); // checked!
	int rid0 = ix * (ny + 1) * (nz + 1) + (iy + 1 + ny - npml) * (nz + 1) + (iz + 1);  //checked!

	int lid1 = ix * (2 * (npml - 1)) * (nz - 1) + iy * (nz - 1) + iz; // checked!
	int rid1 = ix * (2 * (npml - 1)) * (nz - 1) + (iy + npml - 1) * (nz - 1) + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = ix * ny * (nz + 1) + iy * (nz + 1) + (iz + 1); // checked!
	int rid4 = ix * ny * (nz + 1) + (iy + ny - npml) * (nz + 1) + (iz + 1); // checked!

	int lid3 = lid4 + (nz + 1); // checked!
	int rid3 = rid4 + (nz + 1); // checked!

	UEyz[lid0] = UEyz[lid0] * RBEyz[lid1] + RAEyz[lid2] * (Hz[lid3] - Hz[lid4]) / dy;
	UEyz[rid0] = UEyz[rid0] * RBEyz[rid1] + RAEyz[rid2] * (Hz[rid3] - Hz[rid4]) / dy;
}
dim3 gridUEyx(npml - 1, nx);
dim3 blockUEyx(nz - 1);
__global__ void gpu_UEyx(float *UEyx, float *RBEyx, float *RAEyx, float *Hx)
{
	/*
	dim3 blockUEyx(nz - 1);
	dim3 gridUEyx(npml - 1, nx);

	in0 UEyx  nx + 1 ny + 1     nz
	in1 RBEyx nx - 1 2*(npml-1) nz
	in2 RAEyx nx - 1 2*(npml-1) nz
	in3 Hx    nx + 1 ny         nz

	运算块大小 nx * npml-1 * nz-1

	UEyx(2:nx, [2:npml ny-npml+2:ny], :)=RBEyx .* UEyx(2:nx, [2:npml ny-npml+2:ny], :)...
	+RAEyx ./ dy .* (Hx(2:nx, [2:npml ny-npml+2:ny], :) - Hx(2:nx, [1:npml-1 ny-npml+1:ny-1], :));
	*/
	int ix = blockIdx.y;   // ix in [0, nx)
	int iy = blockIdx.x;   // iy in [0, npml - 1)
	int iz = threadIdx.x;  // iz in [0, nz - 1)

	int lid0 = (ix + 1) * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid0 = (ix + 1) * (ny + 1) * nz + (iy + 1 + ny - npml) * nz + iz;  //checked!

	int lid1 = ix * (2 * (npml - 1)) * nz + iy * nz + iz; // checked!
	int rid1 = ix * (2 * (npml - 1)) * nz + (iy + npml - 1) * nz + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = (ix + 1) * ny * nz + iy * nz + iz; // checked!
	int rid4 = (ix + 1) * ny * nz + (iy + ny - npml) * nz + iz; // checked!

	int lid3 = lid4 + nz; // checked!
	int rid3 = rid4 + nz; // checked!

	UEyx[lid0] = UEyx[lid0] * RBEyx[lid1] + RAEyx[lid2] * (Hx[lid3] - Hx[lid4]) / dy;
	UEyx[rid0] = UEyx[rid0] * RBEyx[rid1] + RAEyx[rid2] * (Hx[rid3] - Hx[rid4]) / dy;
}
dim3 gridUExy(npml - 1, ny - 1);
dim3 blockUExy(nz);
__global__ void gpu_UExy(float *UExy, float *RBExy, float *RAExy, float *Hy)
{
	/*
	dim3 blockUExy(nz);
	dim3 gridUExy(npml - 1, ny - 1);

	in0 UExy  nx + 1     ny + 1 nz
	in1 RBExy 2*(npml-1) ny - 1 nz
	in2 RAExy 2*(npml-1) ny - 1 nz
	in3 Hy    nx         ny + 1 nz

	运算块大小 npml-1 * ny-1 * nz

	UExy([2:npml nx-npml+2:nx], 2:ny, :)=RBExy .* UExy([2:npml nx-npml+2:nx], 2:ny, :)...
	+RAExy ./ dx .* (Hy([2:npml nx-npml+2:nx], 2:ny, :) - Hy([1:npml-1 nx-npml+1:nx-1], 2:ny, :));
	*/
	int ix = blockIdx.x;   // ix in [0, npml - 1)
	int iy = blockIdx.y;   // iy in [0, ny - 1)
	int iz = threadIdx.x;  // iz in [0, nz)

	int lid0 = (ix + 1) * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid0 = (ix + 1 + nx - npml) * (ny + 1) * nz + (iy + 1) * nz + iz;  //checked!

	int lid1 = ix * (ny - 1) * nz + iy * nz + iz; // checked!
	int rid1 = (ix + npml - 1) * (ny - 1) * nz + iy * nz + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = ix * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid4 = (ix + nx - npml) * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!

	int lid3 = lid4 + (ny + 1) * nz; // checked!
	int rid3 = rid4 + (ny + 1) * nz; // checked!

	UExy[lid0] = UExy[lid0] * RBExy[lid1] + RAExy[lid2] * (Hy[lid3] - Hy[lid4]) / dx;
	UExy[rid0] = UExy[rid0] * RBExy[rid1] + RAExy[rid2] * (Hy[rid3] - Hy[rid4]) / dx;
}
dim3 gridUExz(npml - 1, ny);
dim3 blockUExz(nz - 1);
__global__ void gpu_UExz(float *UExz, float *RBExz, float *RAExz, float *Hz)
{
	/*
	dim3 blockUExz(nz - 1);
	dim3 gridUExz(npml - 1, ny);

	in0 UExz  nx + 1     ny     nz + 1
	in1 RBExz 2*(npml-1) ny     nz - 1
	in2 RAExz 2*(npml-1) ny     nz - 1
	in3 Hz    nx         ny     nz + 1
	运算块大小 npml-1 * ny * nz-1

	UExz([2:npml nx-npml+2:nx], :, 2:nz)=RBExz .* UExz([2:npml nx-npml+2:nx], :, 2:nz)...
	+RAExz ./ dx .* (Hz([2:npml nx-npml+2:nx], :, 2:nz) - Hz([1:npml-1 nx-npml+1:nx-1], :, 2:nz));
	*/
	int ix = blockIdx.x;   // ix in [0, npml - 1)
	int iy = blockIdx.y;   // iy in [0, ny)
	int iz = threadIdx.x;  // iz in [0, nz - 1)

	int lid0 = (ix + 1) * ny * (nz + 1) + iy * (nz + 1) + (iz + 1); // checked!
	int rid0 = (ix + 1 + nx - npml) * ny * (nz + 1) + iy * (nz + 1) + (iz + 1);  //checked!

	int lid1 = ix * ny * (nz - 1) + iy * (nz - 1) + iz; // checked!
	int rid1 = (ix + npml - 1) * ny * (nz - 1) + iy * (nz - 1) + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = ix * ny * (nz + 1) + iy * (nz + 1) + (iz + 1); // checked!
	int rid4 = (ix + nx - npml) * ny * (nz + 1) + iy * (nz + 1) + (iz + 1); // checked!

	int lid3 = lid4 + ny * (nz + 1); // checked!
	int rid3 = rid4 + ny * (nz + 1); // checked!

	UExz[lid0] = UExz[lid0] * RBExz[lid1] + RAExz[lid2] * (Hz[lid3] - Hz[lid4]) / dx;
	UExz[rid0] = UExz[rid0] * RBExz[rid1] + RAExz[rid2] * (Hz[rid3] - Hz[rid4]) / dx;
}
dim3 gridUEzx(nx - 1, ny);
dim3 blockUEzx(npml - 1);
__global__ void gpu_UEzx(float *UEzx, float *RBEzx, float *RAEzx, float *Hx)
{
	/*
	dim3 blockUEzx(npml - 1);
	dim3 gridUEzx(nx - 1, ny);

	in0 UEzx  nx + 1     ny     nz + 1
	in1 RBEzx nx - 1     ny     2*(npml-1)
	in2 RAEzx nx - 1     ny     2*(npml-1)
	in3 Hx    nx + 1     ny     nz

	运算块大小 nx-1 * ny * npml-1

	UEzx(2:nx, :, [2:npml nz-npml+2:nz])=RBEzx .* UEzx(2:nx, :, [2:npml nz-npml+2:nz])...
	+RAEzx ./ dz .* (Hx(2:nx, :, [2:npml nz-npml+2:nz]) - Hx(2:nx, :, [1:npml-1 nz-npml+1:nz-1]));
	*/
	int ix = blockIdx.x;   // ix in [0, nx)
	int iy = blockIdx.y;   // iy in [0, npml - 1)
	int iz = threadIdx.x;  // iz in [0, nz)

	int lid0 = (ix + 1) * ny * (nz + 1) + iy * (nz + 1) + (iz + 1); // checked!
	int rid0 = (ix + 1) * ny * (nz + 1) + iy * (nz + 1) + (iz + 1 + nz - npml);  //checked!

	int lid1 = ix * ny * (2 * (npml - 1)) + iy * (2 * (npml - 1)) + iz; // checked!
	int rid1 = ix * ny * (2 * (npml - 1)) + iy * (2 * (npml - 1)) + (iz + npml - 1); // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = (ix + 1) * ny * nz + iy * nz + iz; // checked!
	int rid4 = (ix + 1) * ny * nz + iy * nz + (iz + nz - npml); // checked!

	int lid3 = lid4 + 1; // checked!
	int rid3 = rid4 + 1; // checked!

	UEzx[lid0] = UEzx[lid0] * RBEzx[lid1] + RAEzx[lid2] * (Hx[lid3] - Hx[lid4]) / dz;
	UEzx[rid0] = UEzx[rid0] * RBEzx[rid1] + RAEzx[rid2] * (Hx[rid3] - Hx[rid4]) / dz;
}
dim3 gridUEzy(nx, ny - 1);
dim3 blockUEzy(npml - 1);
__global__ void gpu_UEzy(float *UEzy, float *RBEzy, float *RAEzy, float *Hy)
{
	/*
	dim3 blockUEzy(npml - 1);
	dim3 gridUEzy(nx, ny - 1);

	in0 UEzy  nx      ny + 1    nz + 1
	in1 RBEzy nx      ny - 1    2*(npml-1)
	in2 RAEzy nx      ny - 1    2*(npml-1)
	in3 Hy    nx      ny + 1    nz

	运算块大小 nx * ny - 1 * npml-1

	UEzy(:, 2:ny, [2:npml nz-npml+2:nz])=RBEzy.*UEzy(:, 2:ny, [2:npml nz-npml+2:nz])...
	+RAEzy./dz.*(Hy(:, 2:ny, [2:npml nz-npml+2:nz])-Hy(:, 2:ny, [1:npml-1 nz-npml+1:nz-1]));
	*/
	int ix = blockIdx.x;   // ix in [0, nx)
	int iy = blockIdx.y;   // iy in [0, npml - 1)
	int iz = threadIdx.x;  // iz in [0, nz)

	int lid0 = ix * (ny + 1) * (nz + 1) + (iy + 1) * (nz + 1) + (iz + 1); // checked!
	int rid0 = ix * (ny + 1) * (nz + 1) + (iy + 1) * (nz + 1) + (iz + 1 + nz - npml);  //checked!

	int lid1 = ix * (ny - 1) * (2 * (npml - 1)) + iy * (2 * (npml - 1)) + iz; // checked!
	int rid1 = ix * (ny - 1) * (2 * (npml - 1)) + iy * (2 * (npml - 1)) + (iz + npml - 1); // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = ix * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid4 = ix * (ny + 1) * nz + (iy + 1) * nz + (iz + nz - npml); // checked!

	int lid3 = lid4 + 1; // checked!
	int rid3 = rid4 + 1; // checked!

	UEzy[lid0] = UEzy[lid0] * RBEzy[lid1] + RAEzy[lid2] * (Hy[lid3] - Hy[lid4]) / dz;
	UEzy[rid0] = UEzy[rid0] * RBEzy[rid1] + RAEzy[rid2] * (Hy[rid3] - Hy[rid4]) / dz;
}
dim3 gridEx(nx, ny - 1);
dim3 blockEx(nz - 1);
__global__ void gpu_Ex(float *Ex, float *CAEx, float *CBEx, float *ky_Ex, float *kz_Ex, float *Hz, float *Hy, float *UEyz, float *UEzy)
{
	//
	// * dim3 blockEx(nz-1);
	// * dim3 gridEx(nx, ny-1);
	// * 运算块大小 nx * ny-1 * nz-1
	// * Ex(:, 2:ny, 2:nz)
	//
	int ix = blockIdx.x;      // ix in [0, nx)
	int iy = blockIdx.y + 1;  // iy in [1, ny)
	int iz = threadIdx.x + 1; // iz in [1, nz)

	int idx = ix * (ny + 1) * (nz + 1) + iy * (nz + 1) + iz;
	int idxHz = ix * ny * (nz + 1) + iy * (nz + 1) + iz;
	int idxHy = ix * (ny + 1)*nz + iy * nz + iz;
	int deltaHz = nz + 1;
	int deltaHy = 1;
	float CBE = CBEx[idx];

	Ex[idx] = Ex[idx] * CAEx[idx]
		+ CBE / ky_Ex[idx] * (Hz[idxHz] - Hz[idxHz - deltaHz]) / dy
		- CBE / kz_Ex[idx] * (Hy[idxHy] - Hy[idxHy - deltaHy]) / dz
		+ CBE * UEyz[idx]
		- CBE * UEzy[idx];
}
dim3 gridEy(nx - 1, ny);
dim3 blockEy(nz - 1);
__global__ void gpu_Ey(float *Ey, float *CAEy, float *CBEy, float *kz_Ey, float *kx_Ey, float *Hx, float *Hz, float *UEzx, float *UExz)
{
	//
	// * dim3 blockEy(nz-1);
	// * dim3 gridEy(nx-1, ny);
	// * 运算块大小 nx-1 * ny * nz-1
	// * Ey(2:nx, :, 2:nz)
	//
	int ix = blockIdx.x + 1;  // ix in [1, nx)
	int iy = blockIdx.y;      // iy in [0, ny)
	int iz = threadIdx.x + 1; // iz in [1, nz)

	int idx = ix * ny * (nz + 1) + iy * (nz + 1) + iz;
	int idxHx = ix * ny * nz + iy * nz + iz;
	int idxHz = ix * ny * (nz + 1) + iy * (nz + 1) + iz;
	int deltaHx = 1;
	int deltaHz = ny * (nz + 1);
	float CBE = CBEy[idx];

	Ey[idx] = Ey[idx] * CAEy[idx]
		+ CBE / kz_Ey[idx] * (Hx[idxHx] - Hx[idxHx - deltaHx]) / dz
		- CBE / kx_Ey[idx] * (Hz[idxHz] - Hz[idxHz - deltaHz]) / dx
		+ CBE * UEzx[idx]
		- CBE * UExz[idx];
}
dim3 gridEz(nx - 1, ny - 1);
dim3 blockEz(nz);
__global__ void gpu_Ez(float *Ez, float *CAEz, float *CBEz, float *kx_Ez, float *ky_Ez, float *Hy, float *Hx, float *UExy, float *UEyx)
{
	//
	// * dim3 blockEz(nz);
	// * dim3 gridEz(nx-1, ny-1);
	// * 运算块大小 nx-1 * ny-1 * nz
	// * Ez(2:nx, 2:ny, :)
	// * Ez大小为nx ny nz+1
	//
	int ix = blockIdx.x + 1; // ix in [1, nx)
	int iy = blockIdx.y + 1; // iy in [1, ny)
	int iz = threadIdx.x;    // iz in [0, nz)

	int idx = ix * (ny + 1) * nz + iy * nz + iz;
	int idxHy = ix * (ny + 1) * nz + iy * nz + iz;
	int idxHx = ix * ny * nz + iy * nz + iz;
	int deltaHy = (ny + 1) * nz;
	int deltaHx = nz;
	float CBE = CBEz[idx];

	Ez[idx] = Ez[idx] * CAEz[idx]
		+ CBE / kx_Ez[idx] * (Hy[idxHy] - Hy[idxHy - deltaHy]) / dx
		- CBE / ky_Ez[idx] * (Hx[idxHx] - Hx[idxHx - deltaHx]) / dy
		+ CBE * UExy[idx]
		- CBE * UEyx[idx];
}

dim3 grid_zheng_1(npmlc, ny - 2 * npml);
dim3 grid_zheng_2(nx - 2 * npml, npmlc);
dim3 grid_zheng_3(nx - 2 * npml, ny - 2 * npml);
dim3 grid_zheng_last(nx - 2 * npml, ny - 2 * npml);
dim3 block_zheng_1(nz - 2 * npml);
dim3 block_zheng_2(nz - 2 * npml);
dim3 block_zheng_3(npmlc);
dim3 block_zheng_last(nz - 2 * npml);
__global__ void gpu_zheng_1(
	float *dev_Ex_zheng, float *dev_Ey_zheng, float *dev_Ez_zheng,
	float *dev_Hx_zheng, float *dev_Hy_zheng, float *dev_Hz_zheng,
	float *dev_Ex, float *dev_Ey, float *dev_Ez,
	float *dev_Hx, float *dev_Hy, float *dev_Hz,
	int j)
{
	int ix = blockIdx.x;
	int iy = blockIdx.y;
	int iz = threadIdx.x;

	int lidzheng; //**_zheng_* 前半部分的位置
	int ridzheng; //**_zheng_* 后半部分的位置
	int lidEx, lidEy, lidEz, lidHx, lidHy, lidHz;
	int ridEx, ridEy, ridEz, ridHx, ridHy, ridHz;

	lidzheng =
		j * (2 * npmlc) * (ny - 2 * npml) * (nz - 2 * npml) +
		ix * (ny - 2 * npml) * (nz - 2 * npml) +
		iy * (nz - 2 * npml) +
		iz;
	lidEx =
		(ix + npml) * (ny + 1) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);
	lidEy =
		(ix + npml) * (ny + 0) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);
	lidEz =
		(ix + npml) * (ny + 1) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHx =
		(ix + npml) * (ny + 0) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHy =
		(ix + npml) * (ny + 1) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHz =
		(ix + npml) * (ny + 0) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);

	ridzheng = lidzheng + (ny - 2 * npml) * (nz - 2 * npml) * (npmlc);
	ridEx = lidEx + (ny + 1) * (nz + 1) * (nx - 2 * npml - npmlc);
	ridEy = lidEy + (ny + 0) * (nz + 1) * (nx - 2 * npml - npmlc);
	ridEz = lidEz + (ny + 1) * (nz + 0) * (nx - 2 * npml - npmlc);
	ridHx = lidHx + (ny + 0) * (nz + 0) * (nx - 2 * npml - npmlc);
	ridHy = lidHy + (ny + 1) * (nz + 0) * (nx - 2 * npml - npmlc);
	ridHz = lidHz + (ny + 0) * (nz + 1) * (nx - 2 * npml - npmlc);

	dev_Ex_zheng[lidzheng] = dev_Ex[lidEx];
	dev_Ey_zheng[lidzheng] = dev_Ey[lidEy];
	dev_Ez_zheng[lidzheng] = dev_Ez[lidEz];
	dev_Hx_zheng[lidzheng] = dev_Hx[lidHx];
	dev_Hy_zheng[lidzheng] = dev_Hy[lidHy];
	dev_Hz_zheng[lidzheng] = dev_Hz[lidHz];
	dev_Ex_zheng[ridzheng] = dev_Ex[ridEx];
	dev_Ey_zheng[ridzheng] = dev_Ey[ridEy];
	dev_Ez_zheng[ridzheng] = dev_Ez[ridEz];
	dev_Hx_zheng[ridzheng] = dev_Hx[ridHx];
	dev_Hy_zheng[ridzheng] = dev_Hy[ridHy];
	dev_Hz_zheng[ridzheng] = dev_Hz[ridHz];
}

__global__ void gpu_zheng_2(
	float *dev_Ex_zheng, float *dev_Ey_zheng, float *dev_Ez_zheng,
	float *dev_Hx_zheng, float *dev_Hy_zheng, float *dev_Hz_zheng,
	float *dev_Ex, float *dev_Ey, float *dev_Ez,
	float *dev_Hx, float *dev_Hy, float *dev_Hz,
	int j)
{
	int ix = blockIdx.x;
	int iy = blockIdx.y;
	int iz = threadIdx.x;

	int lidzheng; //**_zheng_* 前半部分的位置
	int ridzheng; //**_zheng_* 后半部分的位置
	int lidEx, lidEy, lidEz, lidHx, lidHy, lidHz;
	int ridEx, ridEy, ridEz, ridHx, ridHy, ridHz;

	lidzheng =
		j * (nx - 2 * npml) * (2 * npmlc) * (nz - 2 * npml) +
		ix * (2 * npmlc) * (nz - 2 * npml) +
		iy * (nz - 2 * npml) +
		iz;
	lidEx =
		(ix + npml) * (ny + 1) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);
	lidEy =
		(ix + npml) * (ny + 0) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);
	lidEz =
		(ix + npml) * (ny + 1) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHx =
		(ix + npml) * (ny + 0) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHy =
		(ix + npml) * (ny + 1) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHz =
		(ix + npml) * (ny + 0) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);


	ridzheng = lidzheng + (nz - 2 * npml) * (npmlc);
	ridEx = lidEx + (nz + 1) * (ny - 2 * npml - npmlc);
	ridEy = lidEy + (nz + 1) * (ny - 2 * npml - npmlc);
	ridEz = lidEz + (nz + 0) * (ny - 2 * npml - npmlc);
	ridHx = lidHx + (nz + 0) * (ny - 2 * npml - npmlc);
	ridHy = lidHy + (nz + 0) * (ny - 2 * npml - npmlc);
	ridHz = lidHz + (nz + 1) * (ny - 2 * npml - npmlc);

	dev_Ex_zheng[lidzheng] = dev_Ex[lidEx];
	dev_Ey_zheng[lidzheng] = dev_Ey[lidEy];
	dev_Ez_zheng[lidzheng] = dev_Ez[lidEz];
	dev_Hx_zheng[lidzheng] = dev_Hx[lidHx];
	dev_Hy_zheng[lidzheng] = dev_Hy[lidHy];
	dev_Hz_zheng[lidzheng] = dev_Hz[lidHz];
	dev_Ex_zheng[ridzheng] = dev_Ex[ridEx];
	dev_Ey_zheng[ridzheng] = dev_Ey[ridEy];
	dev_Ez_zheng[ridzheng] = dev_Ez[ridEz];
	dev_Hx_zheng[ridzheng] = dev_Hx[ridHx];
	dev_Hy_zheng[ridzheng] = dev_Hy[ridHy];
	dev_Hz_zheng[ridzheng] = dev_Hz[ridHz];
}

__global__ void gpu_zheng_3(
	float *dev_Ex_zheng, float *dev_Ey_zheng, float *dev_Ez_zheng,
	float *dev_Hx_zheng, float *dev_Hy_zheng, float *dev_Hz_zheng,
	float *dev_Ex, float *dev_Ey, float *dev_Ez,
	float *dev_Hx, float *dev_Hy, float *dev_Hz,
	int j)
{
	int ix = blockIdx.x;
	int iy = blockIdx.y;
	int iz = threadIdx.x;

	int lidzheng; //**_zheng_* 前半部分的位置
	int ridzheng; //**_zheng_* 后半部分的位置
	int lidEx, lidEy, lidEz, lidHx, lidHy, lidHz;
	int ridEx, ridEy, ridEz, ridHx, ridHy, ridHz;

	lidzheng =
		j * (nx - 2 * npml) * (ny - 2 * npml) * (2 * npmlc) +
		ix * (ny - 2 * npml) * (2 * npmlc) +
		iy * (2 * npmlc) +
		iz;
	lidEx =
		(ix + npml) * (ny + 1) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);
	lidEy =
		(ix + npml) * (ny + 0) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);
	lidEz =
		(ix + npml) * (ny + 1) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHx =
		(ix + npml) * (ny + 0) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHy =
		(ix + npml) * (ny + 1) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHz =
		(ix + npml) * (ny + 0) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);


	ridzheng = lidzheng + (npmlc);
	ridEx = lidEx + (nz - 2 * npml - npmlc);
	ridEy = lidEy + (nz - 2 * npml - npmlc);
	ridEz = lidEz + (nz - 2 * npml - npmlc);
	ridHx = lidHx + (nz - 2 * npml - npmlc);
	ridHy = lidHy + (nz - 2 * npml - npmlc);
	ridHz = lidHz + (nz - 2 * npml - npmlc);

	dev_Ex_zheng[lidzheng] = dev_Ex[lidEx];
	dev_Ey_zheng[lidzheng] = dev_Ey[lidEy];
	dev_Ez_zheng[lidzheng] = dev_Ez[lidEz];
	dev_Hx_zheng[lidzheng] = dev_Hx[lidHx];
	dev_Hy_zheng[lidzheng] = dev_Hy[lidHy];
	dev_Hz_zheng[lidzheng] = dev_Hz[lidHz];
	dev_Ex_zheng[ridzheng] = dev_Ex[ridEx];
	dev_Ey_zheng[ridzheng] = dev_Ey[ridEy];
	dev_Ez_zheng[ridzheng] = dev_Ez[ridEz];
	dev_Hx_zheng[ridzheng] = dev_Hx[ridHx];
	dev_Hy_zheng[ridzheng] = dev_Hy[ridHy];
	dev_Hz_zheng[ridzheng] = dev_Hz[ridHz];
}

__global__ void gpu_zheng_last(
	float *dev_Ex_zheng, float *dev_Ey_zheng, float *dev_Ez_zheng,
	float *dev_Hx_zheng, float *dev_Hy_zheng, float *dev_Hz_zheng,
	float *dev_Ex, float *dev_Ey, float *dev_Ez,
	float *dev_Hx, float *dev_Hy, float *dev_Hz)
{
	int ix = blockIdx.x;
	int iy = blockIdx.y;
	int iz = threadIdx.x;

	int lidzheng; //**_zheng_* 前半部分的位置
	int lidEx, lidEy, lidEz, lidHx, lidHy, lidHz;

	lidzheng =
		ix * (ny - 2 * npml) * (nz - 2 * npml) +
		iy * (nz - 2 * npml) +
		iz;
	lidEx =
		(ix + npml) * (ny + 1) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);
	lidEy =
		(ix + npml) * (ny + 0) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);
	lidEz =
		(ix + npml) * (ny + 1) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHx =
		(ix + npml) * (ny + 0) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHy =
		(ix + npml) * (ny + 1) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHz =
		(ix + npml) * (ny + 0) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);

	dev_Ex_zheng[lidzheng] = dev_Ex[lidEx];
	dev_Ey_zheng[lidzheng] = dev_Ey[lidEy];
	dev_Ez_zheng[lidzheng] = dev_Ez[lidEz];
	dev_Hx_zheng[lidzheng] = dev_Hx[lidHx];
	dev_Hy_zheng[lidzheng] = dev_Hy[lidHy];
	dev_Hz_zheng[lidzheng] = dev_Hz[lidHz];
}


__global__ void gpu_back_zheng_1(
	float *dev_Ex_zheng, float *dev_Ey_zheng, float *dev_Ez_zheng,
	float *dev_Hx_zheng, float *dev_Hy_zheng, float *dev_Hz_zheng,
	float *dev_Ex, float *dev_Ey, float *dev_Ez,
	float *dev_Hx, float *dev_Hy, float *dev_Hz,
	int j)
{
	int ix = blockIdx.x;
	int iy = blockIdx.y;
	int iz = threadIdx.x;

	int lidzheng; //**_zheng_* 前半部分的位置
	int ridzheng; //**_zheng_* 后半部分的位置
	int lidEx, lidEy, lidEz, lidHx, lidHy, lidHz;
	int ridEx, ridEy, ridEz, ridHx, ridHy, ridHz;

	lidzheng =
		j * (2 * npmlc) * (ny - 2 * npml) * (nz - 2 * npml) +
		ix * (ny - 2 * npml) * (nz - 2 * npml) +
		iy * (nz - 2 * npml) +
		iz;
	lidEx =
		(ix + npml) * (ny + 1) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);
	lidEy =
		(ix + npml) * (ny + 0) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);
	lidEz =
		(ix + npml) * (ny + 1) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHx =
		(ix + npml) * (ny + 0) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHy =
		(ix + npml) * (ny + 1) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHz =
		(ix + npml) * (ny + 0) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);

	ridzheng = lidzheng + (ny - 2 * npml) * (nz - 2 * npml) * (npmlc);
	ridEx = lidEx + (ny + 1) * (nz + 1) * (nx - 2 * npml - npmlc);
	ridEy = lidEy + (ny + 0) * (nz + 1) * (nx - 2 * npml - npmlc);
	ridEz = lidEz + (ny + 1) * (nz + 0) * (nx - 2 * npml - npmlc);
	ridHx = lidHx + (ny + 0) * (nz + 0) * (nx - 2 * npml - npmlc);
	ridHy = lidHy + (ny + 1) * (nz + 0) * (nx - 2 * npml - npmlc);
	ridHz = lidHz + (ny + 0) * (nz + 1) * (nx - 2 * npml - npmlc);

	dev_Ex[lidEx] = dev_Ex_zheng[lidzheng];
	dev_Ey[lidEy] = dev_Ey_zheng[lidzheng];
	dev_Ez[lidEz] = dev_Ez_zheng[lidzheng];
	dev_Hx[lidHx] = dev_Hx_zheng[lidzheng];
	dev_Hy[lidHy] = dev_Hy_zheng[lidzheng];
	dev_Hz[lidHz] = dev_Hz_zheng[lidzheng];
	dev_Ex[ridEx] = dev_Ex_zheng[ridzheng];
	dev_Ey[ridEy] = dev_Ey_zheng[ridzheng];
	dev_Ez[ridEz] = dev_Ez_zheng[ridzheng];
	dev_Hx[ridHx] = dev_Hx_zheng[ridzheng];
	dev_Hy[ridHy] = dev_Hy_zheng[ridzheng];
	dev_Hz[ridHz] = dev_Hz_zheng[ridzheng];
}

__global__ void gpu_back_zheng_2(
	float *dev_Ex_zheng, float *dev_Ey_zheng, float *dev_Ez_zheng,
	float *dev_Hx_zheng, float *dev_Hy_zheng, float *dev_Hz_zheng,
	float *dev_Ex, float *dev_Ey, float *dev_Ez,
	float *dev_Hx, float *dev_Hy, float *dev_Hz,
	int j)
{
	int ix = blockIdx.x;
	int iy = blockIdx.y;
	int iz = threadIdx.x;

	int lidzheng; //**_zheng_* 前半部分的位置
	int ridzheng; //**_zheng_* 后半部分的位置
	int lidEx, lidEy, lidEz, lidHx, lidHy, lidHz;
	int ridEx, ridEy, ridEz, ridHx, ridHy, ridHz;

	lidzheng =
		j * (nx - 2 * npml) * (2 * npmlc) * (nz - 2 * npml) +
		ix * (2 * npmlc) * (nz - 2 * npml) +
		iy * (nz - 2 * npml) +
		iz;
	lidEx =
		(ix + npml) * (ny + 1) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);
	lidEy =
		(ix + npml) * (ny + 0) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);
	lidEz =
		(ix + npml) * (ny + 1) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHx =
		(ix + npml) * (ny + 0) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHy =
		(ix + npml) * (ny + 1) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHz =
		(ix + npml) * (ny + 0) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);


	ridzheng = lidzheng + (nz - 2 * npml) * (npmlc);
	ridEx = lidEx + (nz + 1) * (ny - 2 * npml - npmlc);
	ridEy = lidEy + (nz + 1) * (ny - 2 * npml - npmlc);
	ridEz = lidEz + (nz + 0) * (ny - 2 * npml - npmlc);
	ridHx = lidHx + (nz + 0) * (ny - 2 * npml - npmlc);
	ridHy = lidHy + (nz + 0) * (ny - 2 * npml - npmlc);
	ridHz = lidHz + (nz + 1) * (ny - 2 * npml - npmlc);

	dev_Ex[lidEx] = dev_Ex_zheng[lidzheng];
	dev_Ey[lidEy] = dev_Ey_zheng[lidzheng];
	dev_Ez[lidEz] = dev_Ez_zheng[lidzheng];
	dev_Hx[lidHx] = dev_Hx_zheng[lidzheng];
	dev_Hy[lidHy] = dev_Hy_zheng[lidzheng];
	dev_Hz[lidHz] = dev_Hz_zheng[lidzheng];
	dev_Ex[ridEx] = dev_Ex_zheng[ridzheng];
	dev_Ey[ridEy] = dev_Ey_zheng[ridzheng];
	dev_Ez[ridEz] = dev_Ez_zheng[ridzheng];
	dev_Hx[ridHx] = dev_Hx_zheng[ridzheng];
	dev_Hy[ridHy] = dev_Hy_zheng[ridzheng];
	dev_Hz[ridHz] = dev_Hz_zheng[ridzheng];
}

__global__ void gpu_back_zheng_3(
	float *dev_Ex_zheng, float *dev_Ey_zheng, float *dev_Ez_zheng,
	float *dev_Hx_zheng, float *dev_Hy_zheng, float *dev_Hz_zheng,
	float *dev_Ex, float *dev_Ey, float *dev_Ez,
	float *dev_Hx, float *dev_Hy, float *dev_Hz,
	int j)
{
	int ix = blockIdx.x;
	int iy = blockIdx.y;
	int iz = threadIdx.x;

	int lidzheng; //**_zheng_* 前半部分的位置
	int ridzheng; //**_zheng_* 后半部分的位置
	int lidEx, lidEy, lidEz, lidHx, lidHy, lidHz;
	int ridEx, ridEy, ridEz, ridHx, ridHy, ridHz;

	lidzheng =
		j * (nx - 2 * npml) * (ny - 2 * npml) * (2 * npmlc) +
		ix * (ny - 2 * npml) * (2 * npmlc) +
		iy * (2 * npmlc) +
		iz;
	lidEx =
		(ix + npml) * (ny + 1) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);
	lidEy =
		(ix + npml) * (ny + 0) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);
	lidEz =
		(ix + npml) * (ny + 1) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHx =
		(ix + npml) * (ny + 0) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHy =
		(ix + npml) * (ny + 1) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHz =
		(ix + npml) * (ny + 0) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);


	ridzheng = lidzheng + (npmlc);
	ridEx = lidEx + (nz - 2 * npml - npmlc);
	ridEy = lidEy + (nz - 2 * npml - npmlc);
	ridEz = lidEz + (nz - 2 * npml - npmlc);
	ridHx = lidHx + (nz - 2 * npml - npmlc);
	ridHy = lidHy + (nz - 2 * npml - npmlc);
	ridHz = lidHz + (nz - 2 * npml - npmlc);

	dev_Ex[lidEx] = dev_Ex_zheng[lidzheng];
	dev_Ey[lidEy] = dev_Ey_zheng[lidzheng];
	dev_Ez[lidEz] = dev_Ez_zheng[lidzheng];
	dev_Hx[lidHx] = dev_Hx_zheng[lidzheng];
	dev_Hy[lidHy] = dev_Hy_zheng[lidzheng];
	dev_Hz[lidHz] = dev_Hz_zheng[lidzheng];
	dev_Ex[ridEx] = dev_Ex_zheng[ridzheng];
	dev_Ey[ridEy] = dev_Ey_zheng[ridzheng];
	dev_Ez[ridEz] = dev_Ez_zheng[ridzheng];
	dev_Hx[ridHx] = dev_Hx_zheng[ridzheng];
	dev_Hy[ridHy] = dev_Hy_zheng[ridzheng];
	dev_Hz[ridHz] = dev_Hz_zheng[ridzheng];
}

__global__ void gpu_back_zheng_last(
	float *dev_Ex_zheng, float *dev_Ey_zheng, float *dev_Ez_zheng,
	float *dev_Hx_zheng, float *dev_Hy_zheng, float *dev_Hz_zheng,
	float *dev_Ex, float *dev_Ey, float *dev_Ez,
	float *dev_Hx, float *dev_Hy, float *dev_Hz)
{
	int ix = blockIdx.x;
	int iy = blockIdx.y;
	int iz = threadIdx.x;

	int lidzheng; //**_zheng_* 前半部分的位置
	int lidEx, lidEy, lidEz, lidHx, lidHy, lidHz;

	lidzheng =
		ix * (ny - 2 * npml) * (nz - 2 * npml) +
		iy * (nz - 2 * npml) +
		iz;
	lidEx =
		(ix + npml) * (ny + 1) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);
	lidEy =
		(ix + npml) * (ny + 0) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);
	lidEz =
		(ix + npml) * (ny + 1) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHx =
		(ix + npml) * (ny + 0) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHy =
		(ix + npml) * (ny + 1) * (nz + 0) +
		(iy + npml) * (nz + 0) +
		(iz + npml);
	lidHz =
		(ix + npml) * (ny + 0) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);

	dev_Ex[lidEx] = dev_Ex_zheng[lidzheng];
	dev_Ey[lidEy] = dev_Ey_zheng[lidzheng];
	dev_Ez[lidEz] = dev_Ez_zheng[lidzheng];
	dev_Hx[lidHx] = dev_Hx_zheng[lidzheng];
	dev_Hy[lidHy] = dev_Hy_zheng[lidzheng];
	dev_Hz[lidHz] = dev_Hz_zheng[lidzheng];
}

dim3 grid_fan_huanyuan(nx - 2 * npml, ny - 2 * npml);
dim3 block_fan_huanyuan(nz - 2 * npml);
__global__ void gpu_fan_huanyuan(float *dev_dst, float *dev_Ex)
{
	int ix = blockIdx.x;
	int iy = blockIdx.y;
	int iz = threadIdx.x;

	int lidfan, lidEx; //**_zheng_* 前半部分的位置

	lidfan =
		ix * (ny - 2 * npml) * (nz - 2 * npml) +
		iy * (nz - 2 * npml) +
		iz;
	lidEx =
		(ix + npml) * (ny + 1) * (nz + 1) +
		(iy + npml) * (nz + 1) +
		(iz + npml);
	dev_dst[lidfan] = dev_Ex[lidEx];
}


dim3 grid_HE1(nx - np - np, ny - np - np);
dim3 block_HE1(nz - np - np);
__global__ void gpu_H1(
	float *dev_Hx1, float *dev_Hy1, float *dev_Hz1, 
	float *dev_Ex1, float *dev_Ey1, float *dev_Ez1, 
	float *dev_CPHx, float *dev_CPHy, float *dev_CPHz,
	float *dev_CQHx, float *dev_CQHy, float *dev_CQHz)
{
	int ix = blockIdx.x;
	int iy = blockIdx.y;
	int iz = blockIdx.z;
	int idxHx1 = (ix + np)*(ny + 0)*(nz + 0) + (iy + np)*(nz + 0) + (iz + np);
	int idxHy1 = (ix + np)*(ny + 1)*(nz + 0) + (iy + np)*(nz + 0) + (iz + np);
	int idxHz1 = (ix + np)*(ny + 0)*(nz + 1) + (iy + np)*(nz + 1) + (iz + np);
	int idxEx1 = (ix + np)*(ny + 1)*(nz + 1) + (iy + np)*(nz + 1) + (iz + np);
	int idxEy1 = (ix + np)*(ny + 0)*(nz + 1) + (iy + np)*(nz + 1) + (iz + np);
	int idxEz1 = (ix + np)*(ny + 1)*(nz + 0) + (iy + np)*(nz + 0) + (iz + np);
	int delEz1_Hx1 = nz;
	int delEy1_Hx1 = 1;
	int delEx1_Hy1 = 1;
	int delEz1_Hy1 = (ny + 1) * nz;
	int delEy1_Hz1 = ny * (nz + 1);
	int delEx1_Hz1 = nz + 1;

	const float rfCPHx = 1 / dev_CPHx[idxHx1];// 倒数reciprocal of fCPHx
	const float fCQHx = dev_CQHx[idxHx1]; 
	dev_Hx1[idxHx1] = rfCPHx * dev_Hx1[idxHx1]
		+ rfCPHx * fCQHx / dy * (dev_Ez1[idxEz1 + delEz1_Hx1] - dev_Ez1[idxEz1])
		- rfCPHx * fCQHx / dz * (dev_Ey1[idxEy1 + delEy1_Hx1] - dev_Ey1[idxEy1]);

	const float rfCPHy = 1 / dev_CPHy[idxHy1];// 倒数reciprocal of fCPHy
	const float fCQHy = dev_CQHy[idxHy1];
	dev_Hy1[idxHy1] = rfCPHy * dev_Hy1[idxHy1]
		+ rfCPHy * fCQHy / dz * (dev_Ex1[idxEx1 + delEx1_Hy1] - dev_Ex1[idxEx1])
		- rfCPHy * fCQHy / dx * (dev_Ez1[idxEz1 + delEz1_Hy1] - dev_Ez1[idxEz1]);

	const float rfCPHz = 1 / dev_CPHz[idxHz1];// 倒数reciprocal of fCPHz
	const float fCQHz = dev_CQHz[idxHz1];
	dev_Hz1[idxHz1] = rfCPHz * dev_Hz1[idxHz1]
		+ rfCPHz * fCQHz / dx * (dev_Ey1[idxEy1 + delEy1_Hz1] - dev_Ey1[idxEy1])
		- rfCPHz * fCQHz / dy * (dev_Ex1[idxEx1 + delEx1_Hz1] - dev_Ex1[idxEx1]);

}

__global__ void gpu_E1(
	float *dev_Hx1, float *dev_Hy1, float *dev_Hz1,
	float *dev_Ex1, float *dev_Ey1, float *dev_Ez1,
	float *dev_CAEx, float *dev_CAEy, float *dev_CAEz,
	float *dev_CBEx, float *dev_CBEy, float *dev_CBEz)
{
	int ix = blockIdx.x;
	int iy = blockIdx.y;
	int iz = blockIdx.z;
	int idxHx1 = (ix + np)*(ny + 0)*(nz + 0) + (iy + np)*(nz + 0) + (iz + np);
	int idxHy1 = (ix + np)*(ny + 1)*(nz + 0) + (iy + np)*(nz + 0) + (iz + np);
	int idxHz1 = (ix + np)*(ny + 0)*(nz + 1) + (iy + np)*(nz + 1) + (iz + np);
	int idxEx1 = (ix + np)*(ny + 1)*(nz + 1) + (iy + np)*(nz + 1) + (iz + np);
	int idxEy1 = (ix + np)*(ny + 0)*(nz + 1) + (iy + np)*(nz + 1) + (iz + np);
	int idxEz1 = (ix + np)*(ny + 1)*(nz + 0) + (iy + np)*(nz + 0) + (iz + np);
	int delHz1_Ex1 = nz + 1;
	int delHy1_Ex1 = 1;
	int delHx1_Ey1 = 1;
	int delHz1_Ey1 = ny * (nz + 1);
	int delHy1_Ez1 = (ny + 1) * nz;
	int delHx1_Ez1 = nz;

	const float rfCAEx = 1 / dev_CAEx[idxEx1];// 倒数reciprocal of fCAEx
	const float fCBEx = dev_CBEx[idxEx1];
	dev_Ex1[idxEx1] = rfCAEx * dev_Ex1[idxEx1]
		+ rfCAEx * fCBEx / dy * (dev_Hz1[idxHz1] - dev_Hz1[idxHz1 - delHz1_Ex1])
		- rfCAEx * fCBEx / dz * (dev_Hy1[idxHy1] - dev_Hy1[idxHy1 - delHy1_Ex1]);

	const float rfCAEy = 1 / dev_CAEy[idxEy1];// 倒数reciprocal of fCAEy
	const float fCBEy = dev_CBEy[idxEy1];
	dev_Ey1[idxEy1] = rfCAEy * dev_Ey1[idxEy1]
		+ rfCAEy * fCBEy / dz * (dev_Hx1[idxHx1] - dev_Hx1[idxHx1 - delHx1_Ey1])
		- rfCAEy * fCBEy / dx * (dev_Hz1[idxHz1] - dev_Hz1[idxHz1 - delHz1_Ey1]);

	const float rfCAEz = 1 / dev_CAEz[idxEz1];// 倒数reciprocal of fCAEz
	const float fCBEz = dev_CBEz[idxEz1];
	dev_Ez1[idxEz1] = rfCAEz * dev_Ez1[idxEz1]
		+ rfCAEz * fCBEz / dx * (dev_Hy1[idxHy1] - dev_Hy1[idxHy1 - delHy1_Ez1])
		- rfCAEz * fCBEz / dy * (dev_Hx1[idxHx1] - dev_Hx1[idxHx1 - delHx1_Ez1]);

}

dim3 grid_nzf(nx - 2 * npml, ny - 2 * npml);
dim3 block_nzf(nz - 2 * npml);
__global__ void gpu_nzf(float *dev_dst, float *dev_src1, float *dev_src2)
{
	int idx =
		blockIdx.x * (ny - 2 * npml) * (nz - 2 * npml) +
		blockIdx.y * (nz - 2 * npml) +
		threadIdx.x;
	dev_dst[idx] += dev_src1[idx] * dev_src2[idx];
}

void read_int(const char *name, int *a, int n1, int n2, int n3)
{
	FILE *fp = fopen(name, "r");
	if (fp == NULL) // 判断文件读入是否正确
	{
		printf("fopen %s error! \n", name);
		return;
	}
	printf("fopen %s ok! \n", name);
	for (int i = 0; i < n1; i++)
	{
		for (int k = 0; k < n3; k++)
		{
			for (int j = 0; j < n2; j++)
			{
				fscanf(fp, "%d", &a[i * n2*n3 + j * n3 + k]); // 读入a[i][j][k]

			}
		}
	}
	printf("read %s OK\n", name);

	fclose(fp);
	return;
}

void read_float(const char *name, float *a, int n1, int n2, int n3)
{
	FILE *fp = fopen(name, "r");
	if (fp == NULL) // 判断文件读入是否正确
	{
		printf("fopen %s error! \n", name);
		return;
	}
	printf("fopen %s ok! \n", name);
	for (int i = 0; i < n1; i++)
	{
		for (int k = 0; k < n3; k++)
		{
			for (int j = 0; j < n2; j++)
			{
				fscanf(fp, "%f", a + i * n2*n3 + j * n3 + k); // 读入a[i][j][k]			
			}

		}
	}
	printf("read %s OK\n", name);

	fclose(fp);
	return;
}

void print_nzf(const char *name, float *a, int n1, int n2, int n3)
{
	FILE *fp = fopen(name, "w+");
	if (fp == NULL) // 判断文件读入是否正确
	{
		printf("fopen %s error! \n", name);
		return;
	}
	printf("fopen %s ok! \n", name);
	for (int k = 0; k < n3; k++)
	{
		for (int j = 0; j < n2; j++)
		{
			for (int i = 0; i < n1; i++)
			{
				fprintf(fp, "%8f ", *(a + i * n2*n3 + j * n3 + k)); // 输出a[i][j][k]			
			}

		}
	}
	printf("print %s OK\n", name);

	fclose(fp);
	return;
}

void read_data_from_txt()
{
	if (isPianYi)
	{
		read_float("data_pianyi/CAEx.txt", (float*)CAEx, nx, ny + 1, nz + 1);
		read_float("data_pianyi/CBEx.txt", (float*)CBEx, nx, ny + 1, nz + 1);
		read_float("data_pianyi/RAEyz.txt", (float*)RAEyz, nx, 2 * (npml - 1), nz - 1);
		read_float("data_pianyi/RBEyz.txt", (float*)RBEyz, nx, 2 * (npml - 1), nz - 1);
		read_float("data_pianyi/RAEzy.txt", (float*)RAEzy, nx, ny - 1, 2 * (npml - 1));
		read_float("data_pianyi/RBEzy.txt", (float*)RBEzy, nx, ny - 1, 2 * (npml - 1));
		read_float("data_pianyi/CAEy.txt", (float*)CAEy, nx + 1, ny, nz + 1);
		read_float("data_pianyi/CBEy.txt", (float*)CBEy, nx + 1, ny, nz + 1);
		read_float("data_pianyi/RAEzx.txt", (float*)RAEzx, nx - 1, ny, 2 * (npml - 1));
		read_float("data_pianyi/RBEzx.txt", (float*)RBEzx, nx - 1, ny, 2 * (npml - 1));
		read_float("data_pianyi/RAExz.txt", (float*)RAExz, 2 * (npml - 1), ny, nz - 1);
		read_float("data_pianyi/RBExz.txt", (float*)RBExz, 2 * (npml - 1), ny, nz - 1);
		read_float("data_pianyi/CAEz.txt", (float*)CAEz, nx + 1, ny + 1, nz);
		read_float("data_pianyi/CBEz.txt", (float*)CBEz, nx + 1, ny + 1, nz);
		read_float("data_pianyi/RAExy.txt", (float*)RAExy, 2 * (npml - 1), ny - 1, nz);
		read_float("data_pianyi/RBExy.txt", (float*)RBExy, 2 * (npml - 1), ny - 1, nz);
		read_float("data_pianyi/RAEyx.txt", (float*)RAEyx, nx - 1, 2 * (npml - 1), nz);
		read_float("data_pianyi/RBEyx.txt", (float*)RBEyx, nx - 1, 2 * (npml - 1), nz);

		read_float("data_pianyi/CPHx.txt", (float*)CPHx, nx + 1, ny, nz);
		read_float("data_pianyi/CQHx.txt", (float*)CQHx, nx + 1, ny, nz);
		read_float("data_pianyi/RAHyz.txt", (float*)RAHyz, nx - 1, 2 * npml, nz);
		read_float("data_pianyi/RBHyz.txt", (float*)RBHyz, nx - 1, 2 * npml, nz);
		read_float("data_pianyi/RAHzy.txt", (float*)RAHzy, nx - 1, ny, 2 * npml);
		read_float("data_pianyi/RBHzy.txt", (float*)RBHzy, nx - 1, ny, 2 * npml);
		read_float("data_pianyi/CPHy.txt", (float*)CPHy, nx, ny + 1, nz);
		read_float("data_pianyi/CQHy.txt", (float*)CQHy, nx, ny + 1, nz);
		read_float("data_pianyi/RAHzx.txt", (float*)RAHzx, nx, ny - 1, 2 * npml);
		read_float("data_pianyi/RBHzx.txt", (float*)RBHzx, nx, ny - 1, 2 * npml);
		read_float("data_pianyi/RAHxz.txt", (float*)RAHxz, 2 * npml, ny - 1, nz);
		read_float("data_pianyi/RBHxz.txt", (float*)RBHxz, 2 * npml, ny - 1, nz);
		read_float("data_pianyi/CPHz.txt", (float*)CPHz, nx, ny, nz + 1);
		read_float("data_pianyi/CQHz.txt", (float*)CQHz, nx, ny, nz + 1);
		read_float("data_pianyi/RAHxy.txt", (float*)RAHxy, 2 * npml, ny, nz - 1);
		read_float("data_pianyi/RBHxy.txt", (float*)RBHxy, 2 * npml, ny, nz - 1);
		read_float("data_pianyi/RAHyx.txt", (float*)RAHyx, nx, 2 * npml, nz - 1);
		read_float("data_pianyi/RBHyx.txt", (float*)RBHyx, nx, 2 * npml, nz - 1);

		read_float("data_pianyi/kx_Ey.txt", (float*)kx_Ey, nx + 1, ny, nz + 1);
		read_float("data_pianyi/kx_Ez.txt", (float*)kx_Ez, nx + 1, ny + 1, nz);
		read_float("data_pianyi/ky_Ex.txt", (float*)ky_Ex, nx, ny + 1, nz + 1);
		read_float("data_pianyi/ky_Ez.txt", (float*)ky_Ez, nx + 1, ny + 1, nz);
		read_float("data_pianyi/kz_Ex.txt", (float*)kz_Ex, nx, ny + 1, nz + 1);
		read_float("data_pianyi/kz_Ey.txt", (float*)kz_Ey, nx + 1, ny, nz + 1);

		read_float("data_pianyi/kx_Hy.txt", (float*)kx_Hy, nx, ny + 1, nz);
		read_float("data_pianyi/kx_Hz.txt", (float*)kx_Hz, nx, ny, nz + 1);
		read_float("data_pianyi/ky_Hx.txt", (float*)ky_Hx, nx + 1, ny, nz);
		read_float("data_pianyi/ky_Hz.txt", (float*)ky_Hz, nx, ny, nz + 1);
		read_float("data_pianyi/kz_Hx.txt", (float*)kz_Hx, nx + 1, ny, nz);
		read_float("data_pianyi/kz_Hy.txt", (float*)kz_Hy, nx, ny + 1, nz);

		read_int("data_pianyi/fswzx.txt", (int*)fswzx, 1, 1, szfsw);
		read_int("data_pianyi/fswzy.txt", (int*)fswzy, 1, 1, szfsw);
		read_int("data_pianyi/fswzz.txt", (int*)fswzz, 1, 1, szfsw);
		read_int("data_pianyi/jswzx.txt", (int*)jswzx, 1, 1, szfsw);
		read_int("data_pianyi/jswzy.txt", (int*)jswzy, 1, 1, szfsw);
		read_int("data_pianyi/jswzz.txt", (int*)jswzz, 1, 1, szfsw);
		read_float("data_pianyi/source.txt", (float*)source, 1, 1, it);
		read_float("data_pianyi/E_obs.txt", (float*)source, 1, it, szfsw);
	}
	else
	{
		read_float("data_zhengyan/CAEx.txt", (float*)CAEx, nx, ny + 1, nz + 1);
		read_float("data_zhengyan/CBEx.txt", (float*)CBEx, nx, ny + 1, nz + 1);
		read_float("data_zhengyan/RAEyz.txt", (float*)RAEyz, nx, 2 * (npml - 1), nz - 1);
		read_float("data_zhengyan/RBEyz.txt", (float*)RBEyz, nx, 2 * (npml - 1), nz - 1);
		read_float("data_zhengyan/RAEzy.txt", (float*)RAEzy, nx, ny - 1, 2 * (npml - 1));
		read_float("data_zhengyan/RBEzy.txt", (float*)RBEzy, nx, ny - 1, 2 * (npml - 1));
		read_float("data_zhengyan/CAEy.txt", (float*)CAEy, nx + 1, ny, nz + 1);
		read_float("data_zhengyan/CBEy.txt", (float*)CBEy, nx + 1, ny, nz + 1);
		read_float("data_zhengyan/RAEzx.txt", (float*)RAEzx, nx - 1, ny, 2 * (npml - 1));
		read_float("data_zhengyan/RBEzx.txt", (float*)RBEzx, nx - 1, ny, 2 * (npml - 1));
		read_float("data_zhengyan/RAExz.txt", (float*)RAExz, 2 * (npml - 1), ny, nz - 1);
		read_float("data_zhengyan/RBExz.txt", (float*)RBExz, 2 * (npml - 1), ny, nz - 1);
		read_float("data_zhengyan/CAEz.txt", (float*)CAEz, nx + 1, ny + 1, nz);
		read_float("data_zhengyan/CBEz.txt", (float*)CBEz, nx + 1, ny + 1, nz);
		read_float("data_zhengyan/RAExy.txt", (float*)RAExy, 2 * (npml - 1), ny - 1, nz);
		read_float("data_zhengyan/RBExy.txt", (float*)RBExy, 2 * (npml - 1), ny - 1, nz);
		read_float("data_zhengyan/RAEyx.txt", (float*)RAEyx, nx - 1, 2 * (npml - 1), nz);
		read_float("data_zhengyan/RBEyx.txt", (float*)RBEyx, nx - 1, 2 * (npml - 1), nz);

		read_float("data_zhengyan/CPHx.txt", (float*)CPHx, nx + 1, ny, nz);
		read_float("data_zhengyan/CQHx.txt", (float*)CQHx, nx + 1, ny, nz);
		read_float("data_zhengyan/RAHyz.txt", (float*)RAHyz, nx - 1, 2 * npml, nz);
		read_float("data_zhengyan/RBHyz.txt", (float*)RBHyz, nx - 1, 2 * npml, nz);
		read_float("data_zhengyan/RAHzy.txt", (float*)RAHzy, nx - 1, ny, 2 * npml);
		read_float("data_zhengyan/RBHzy.txt", (float*)RBHzy, nx - 1, ny, 2 * npml);
		read_float("data_zhengyan/CPHy.txt", (float*)CPHy, nx, ny + 1, nz);
		read_float("data_zhengyan/CQHy.txt", (float*)CQHy, nx, ny + 1, nz);
		read_float("data_zhengyan/RAHzx.txt", (float*)RAHzx, nx, ny - 1, 2 * npml);
		read_float("data_zhengyan/RBHzx.txt", (float*)RBHzx, nx, ny - 1, 2 * npml);
		read_float("data_zhengyan/RAHxz.txt", (float*)RAHxz, 2 * npml, ny - 1, nz);
		read_float("data_zhengyan/RBHxz.txt", (float*)RBHxz, 2 * npml, ny - 1, nz);
		read_float("data_zhengyan/CPHz.txt", (float*)CPHz, nx, ny, nz + 1);
		read_float("data_zhengyan/CQHz.txt", (float*)CQHz, nx, ny, nz + 1);
		read_float("data_zhengyan/RAHxy.txt", (float*)RAHxy, 2 * npml, ny, nz - 1);
		read_float("data_zhengyan/RBHxy.txt", (float*)RBHxy, 2 * npml, ny, nz - 1);
		read_float("data_zhengyan/RAHyx.txt", (float*)RAHyx, nx, 2 * npml, nz - 1);
		read_float("data_zhengyan/RBHyx.txt", (float*)RBHyx, nx, 2 * npml, nz - 1);

		read_float("data_zhengyan/kx_Ey.txt", (float*)kx_Ey, nx + 1, ny, nz + 1);
		read_float("data_zhengyan/kx_Ez.txt", (float*)kx_Ez, nx + 1, ny + 1, nz);
		read_float("data_zhengyan/ky_Ex.txt", (float*)ky_Ex, nx, ny + 1, nz + 1);
		read_float("data_zhengyan/ky_Ez.txt", (float*)ky_Ez, nx + 1, ny + 1, nz);
		read_float("data_zhengyan/kz_Ex.txt", (float*)kz_Ex, nx, ny + 1, nz + 1);
		read_float("data_zhengyan/kz_Ey.txt", (float*)kz_Ey, nx + 1, ny, nz + 1);

		read_float("data_zhengyan/kx_Hy.txt", (float*)kx_Hy, nx, ny + 1, nz);
		read_float("data_zhengyan/kx_Hz.txt", (float*)kx_Hz, nx, ny, nz + 1);
		read_float("data_zhengyan/ky_Hx.txt", (float*)ky_Hx, nx + 1, ny, nz);
		read_float("data_zhengyan/ky_Hz.txt", (float*)ky_Hz, nx, ny, nz + 1);
		read_float("data_zhengyan/kz_Hx.txt", (float*)kz_Hx, nx + 1, ny, nz);
		read_float("data_zhengyan/kz_Hy.txt", (float*)kz_Hy, nx, ny + 1, nz);

		read_int("data_zhengyan/fswzx.txt", (int*)fswzx, 1, 1, szfsw);
		read_int("data_zhengyan/fswzy.txt", (int*)fswzy, 1, 1, szfsw);
		read_int("data_zhengyan/fswzz.txt", (int*)fswzz, 1, 1, szfsw);
		read_int("data_zhengyan/jswzx.txt", (int*)jswzx, 1, 1, szfsw);
		read_int("data_zhengyan/jswzy.txt", (int*)jswzy, 1, 1, szfsw);
		read_int("data_zhengyan/jswzz.txt", (int*)jswzz, 1, 1, szfsw);
		read_float("data_zhengyan/source.txt", (float*)source, 1, 1, it);
	}
}

void print_E_obs()
{
	const char *name = "output/E_obs.txt";
	FILE *fp = fopen(name, "w+");
	if (fp == NULL) // 判断文件读入是否正确
	{
		printf("fopen %s error! \n", name);
	}
	printf("print fopen %s ok! \n", name);

	fprintf(fp, "输出E_obs[%d][%d]\n", it, szfsw);
	fprintf(fp, "共有 %d 行 %d 列 \n", szfsw, it);

	for (int i = 0; i < szfsw; i++)
	{
		for (int j = 0; j < it; j++)
		{
			fprintf(fp, "%8f ", E_obs[j][i]);
		}
		fprintf(fp, "\n");
	}
	printf("print %s OK\n", name);

	fclose(fp);
	return;
}

void gpu_memory_malloc()
{
	hipError_t cudaStatus = hipSuccess;
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");  }
	//原来内存中存在的数组，数组大小用内存数组大小就行
	cudaStatus = hipMalloc((void**)&dev_CAEx, sizeof(CAEx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_CBEx, sizeof(CBEx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RAEyz, sizeof(RAEyz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RBEyz, sizeof(RBEyz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RAEzy, sizeof(RAEzy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RBEzy, sizeof(RBEzy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	cudaStatus = hipMalloc((void**)&dev_CAEy, sizeof(CAEy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_CBEy, sizeof(CBEy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RAExz, sizeof(RAExz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RBExz, sizeof(RBExz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RAEzx, sizeof(RAEzx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RBEzx, sizeof(RBEzx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	cudaStatus = hipMalloc((void**)&dev_CAEz, sizeof(CAEz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_CBEz, sizeof(CBEz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RAExy, sizeof(RAExy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RBExy, sizeof(RBExy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RAEyx, sizeof(RAEyx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RBEyx, sizeof(RBEyx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	cudaStatus = hipMalloc((void**)&dev_CPHx, sizeof(CPHx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_CQHx, sizeof(CQHx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RAHyz, sizeof(RAHyz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RBHyz, sizeof(RBHyz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RAHzy, sizeof(RAHzy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RBHzy, sizeof(RBHzy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	cudaStatus = hipMalloc((void**)&dev_CPHy, sizeof(CPHy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_CQHy, sizeof(CQHy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RAHxz, sizeof(RAHxz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RBHxz, sizeof(RBHxz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RAHzx, sizeof(RAHzx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RBHzx, sizeof(RBHzx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	cudaStatus = hipMalloc((void**)&dev_CPHz, sizeof(CPHz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_CQHz, sizeof(CQHz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RAHxy, sizeof(RAHxy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RBHxy, sizeof(RBHxy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RAHyx, sizeof(RAHyx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_RBHyx, sizeof(RBHyx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	cudaStatus = hipMalloc((void**)&dev_kx_Ey, sizeof(kx_Ey));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_kx_Ez, sizeof(kx_Ez));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_ky_Ex, sizeof(ky_Ex));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_ky_Ez, sizeof(ky_Ez));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_kz_Ex, sizeof(kz_Ex));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_kz_Ey, sizeof(kz_Ey));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	cudaStatus = hipMalloc((void**)&dev_kx_Hy, sizeof(kx_Hy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_kx_Hz, sizeof(kx_Hz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_ky_Hx, sizeof(ky_Hx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_ky_Hz, sizeof(ky_Hz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_kz_Hx, sizeof(kz_Hx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_kz_Hy, sizeof(kz_Hy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	//gpu显存新创建数组，原来内存中不存在
	int szEx = nx * (ny + 1)*(nz + 1);
	int szEy = (nx + 1)*ny*(nz + 1);
	int szEz = (nx + 1)*(ny + 1)*nz;
	int szHx = (nx + 1)*ny*nz;
	int szHy = nx * (ny + 1)*nz;
	int szHz = nx * ny*(nz + 1);

	cudaStatus = hipMalloc((void**)&dev_Ex, szEx * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_UEyz, szEx * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_UEzy, szEx * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	cudaStatus = hipMalloc((void**)&dev_Ey, szEy * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_UEzx, szEy * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_UExz, szEy * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	cudaStatus = hipMalloc((void**)&dev_Ez, szEz * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_UExy, szEz * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_UEyx, szEz * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	cudaStatus = hipMalloc((void**)&dev_Hx, szHx * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_UHyz, szHx * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_UHzy, szHx * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	cudaStatus = hipMalloc((void**)&dev_Hy, szHy * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_UHzx, szHy * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_UHxz, szHy * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	cudaStatus = hipMalloc((void**)&dev_Hz, szHz * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_UHxy, szHz * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_UHyx, szHz * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	cudaStatus = hipMalloc((void**)&dev_V, sizeof(V));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_E_obs, sizeof(E_obs));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_source, sizeof(source));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	// 第二部分并行需要用到的变量

	cudaStatus = hipMalloc((void**)&dev_fan, sizeof(fan));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_huanyuan, sizeof(huanyuan));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_ns, sizeof(ns));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_zv, sizeof(zv));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_fv, sizeof(fv));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	cudaStatus = hipMalloc((void**)&dev_Ex1, sizeof(Ex1));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_Ey1, sizeof(Ey1));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_Ez1, sizeof(Ez1));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	cudaStatus = hipMalloc((void**)&dev_Hx1, sizeof(Hx1));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_Hy1, sizeof(Hy1));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}
	cudaStatus = hipMalloc((void**)&dev_Hz1, sizeof(Hz1));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!");}

	// 超大数组

	cudaStatus = hipMalloc((void**)&dev_Ex_zheng_1, (it)*(2 * npmlc)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}
	cudaStatus = hipMalloc((void**)&dev_Ex_zheng_2, (it)*(nx - 2 * npml)*(2 * npmlc)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}
	cudaStatus = hipMalloc((void**)&dev_Ex_zheng_3, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npmlc) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}

	cudaStatus = hipMalloc((void**)&dev_Ey_zheng_1, (it)*(2 * npmlc)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}
	cudaStatus = hipMalloc((void**)&dev_Ey_zheng_2, (it)*(nx - 2 * npml)*(2 * npmlc)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}
	cudaStatus = hipMalloc((void**)&dev_Ey_zheng_3, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npmlc) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}

	cudaStatus = hipMalloc((void**)&dev_Ez_zheng_1, (it)*(2 * npmlc)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}
	cudaStatus = hipMalloc((void**)&dev_Ez_zheng_2, (it)*(nx - 2 * npml)*(2 * npmlc)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}
	cudaStatus = hipMalloc((void**)&dev_Ez_zheng_3, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npmlc) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}

	cudaStatus = hipMalloc((void**)&dev_Hx_zheng_1, (it)*(2 * npmlc)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}
	cudaStatus = hipMalloc((void**)&dev_Hx_zheng_2, (it)*(nx - 2 * npml)*(2 * npmlc)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}
	cudaStatus = hipMalloc((void**)&dev_Hx_zheng_3, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npmlc) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}

	cudaStatus = hipMalloc((void**)&dev_Hy_zheng_1, (it)*(2 * npmlc)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}
	cudaStatus = hipMalloc((void**)&dev_Hy_zheng_2, (it)*(nx - 2 * npml)*(2 * npmlc)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}
	cudaStatus = hipMalloc((void**)&dev_Hy_zheng_3, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npmlc) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}

	cudaStatus = hipMalloc((void**)&dev_Hz_zheng_1, (it)*(2 * npmlc)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}
	cudaStatus = hipMalloc((void**)&dev_Hz_zheng_2, (it)*(nx - 2 * npml)*(2 * npmlc)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}
	cudaStatus = hipMalloc((void**)&dev_Hz_zheng_3, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npmlc) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}

	cudaStatus = hipMalloc((void**)&dev_Ex_zheng_last, (nx - 2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}
	cudaStatus = hipMalloc((void**)&dev_Ey_zheng_last, (nx - 2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}
	cudaStatus = hipMalloc((void**)&dev_Ez_zheng_last, (nx - 2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}

	cudaStatus = hipMalloc((void**)&dev_Hx_zheng_last, (nx - 2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}
	cudaStatus = hipMalloc((void**)&dev_Hy_zheng_last, (nx - 2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}
	cudaStatus = hipMalloc((void**)&dev_Hz_zheng_last, (nx - 2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!");}
}

// flag == 0 将GPU显存中的E*, UE**, H*, UH**, (V, E_obs)置零
// flag == 1 将GPU显存中的E*, UE**, H*, UH**, (V, E*_zheng_*, H*_zheng_*, E*_zheng_last, H*_zheng_last, fan, huanyuan)置零
// flag == 2 将GPU显存中的E*, UE**, H*, UH**, (V, E*1, H*1)置零
void gpu_memory_set_zero(int flag)
{
	int szEx = nx * (ny + 1)*(nz + 1);
	int szEy = (nx + 1)*ny*(nz + 1);
	int szEz = (nx + 1)*(ny + 1)*nz;
	int szHx = (nx + 1)*ny*nz;
	int szHy = nx * (ny + 1)*nz;
	int szHz = nx * ny*(nz + 1);

	hipMemset(dev_Ex, 0, szEx * sizeof(float));
	hipMemset(dev_UEyz, 0, szEx * sizeof(float));
	hipMemset(dev_UEzy, 0, szEx * sizeof(float));

	hipMemset(dev_Ey, 0, szEy * sizeof(float));
	hipMemset(dev_UEzx, 0, szEy * sizeof(float));
	hipMemset(dev_UExz, 0, szEy * sizeof(float));

	hipMemset(dev_Ez, 0, szEz * sizeof(float));
	hipMemset(dev_UExy, 0, szEz * sizeof(float));
	hipMemset(dev_UEyx, 0, szEz * sizeof(float));

	hipMemset(dev_Hx, 0, szHx * sizeof(float));
	hipMemset(dev_UHyz, 0, szHx * sizeof(float));
	hipMemset(dev_UHzy, 0, szHx * sizeof(float));

	hipMemset(dev_Hy, 0, szHy * sizeof(float));
	hipMemset(dev_UHzx, 0, szHy * sizeof(float));
	hipMemset(dev_UHxz, 0, szHy * sizeof(float));

	hipMemset(dev_Hz, 0, szHz * sizeof(float));
	hipMemset(dev_UHxy, 0, szHz * sizeof(float));
	hipMemset(dev_UHyx, 0, szHz * sizeof(float));

	if (flag == 0)
	{
		hipMemset(dev_V, 0, sizeof(V));
		hipMemset(dev_E_obs, 0, sizeof(E_obs));
	}
	else if (flag == 1)
	{
		hipMemset(dev_V, 0, sizeof(V));

		hipMemset(dev_Ex_zheng_1, 0, (it)*(2 * npmlc)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Ex_zheng_2, 0, (it)*(nx - 2 * npml)*(2 * npmlc)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Ex_zheng_3, 0, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npmlc) * sizeof(float));

		hipMemset(dev_Ey_zheng_1, 0, (it)*(2 * npmlc)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Ey_zheng_2, 0, (it)*(nx - 2 * npml)*(2 * npmlc)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Ey_zheng_3, 0, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npmlc) * sizeof(float));

		hipMemset(dev_Ez_zheng_1, 0, (it)*(2 * npmlc)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Ez_zheng_2, 0, (it)*(nx - 2 * npml)*(2 * npmlc)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Ez_zheng_3, 0, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npmlc) * sizeof(float));

		hipMemset(dev_Hx_zheng_1, 0, (it)*(2 * npmlc)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Hx_zheng_2, 0, (it)*(nx - 2 * npml)*(2 * npmlc)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Hx_zheng_3, 0, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npmlc) * sizeof(float));

		hipMemset(dev_Hy_zheng_1, 0, (it)*(2 * npmlc)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Hy_zheng_2, 0, (it)*(nx - 2 * npml)*(2 * npmlc)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Hy_zheng_3, 0, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npmlc) * sizeof(float));

		hipMemset(dev_Hz_zheng_1, 0, (it)*(2 * npmlc)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Hz_zheng_2, 0, (it)*(nx - 2 * npml)*(2 * npmlc)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Hz_zheng_3, 0, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npmlc) * sizeof(float));

		size_t sz_last = (nx - 2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float);
		hipMemset(dev_Ex_zheng_last, 0, sz_last);
		hipMemset(dev_Ey_zheng_last, 0, sz_last);
		hipMemset(dev_Ez_zheng_last, 0, sz_last);

		hipMemset(dev_Hx_zheng_last, 0, sz_last);
		hipMemset(dev_Hy_zheng_last, 0, sz_last);
		hipMemset(dev_Hz_zheng_last, 0, sz_last);

		hipMemset(dev_fan, 0, sizeof(fan));
		hipMemset(dev_huanyuan, 0, sizeof(huanyuan));
	}
	else
	{
		hipMemset(dev_Ex1, 0, sizeof(Ex1));
		hipMemset(dev_Ey1, 0, sizeof(Ey1));
		hipMemset(dev_Ez1, 0, sizeof(Ez1));

		hipMemset(dev_Hx1, 0, sizeof(Hx1));
		hipMemset(dev_Hy1, 0, sizeof(Hy1));
		hipMemset(dev_Hz1, 0, sizeof(Hz1));
	}
}

// 将内存中的变量复制到显存中
// flag == 0 CAE CBE RAE RBE CPH CQH RAH RBH k*_E* k*_H* source
// flag == 1 CAE CBE RAE RBE CPH CQH RAH RBH k*_E* k*_H* source
void gpu_memory_copy()
{
	hipError_t cudaStatus;
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_CAEx, CAEx, sizeof(CAEx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_CBEx, CBEx, sizeof(CBEx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RAEyz, RAEyz, sizeof(RAEyz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RBEyz, RBEyz, sizeof(RBEyz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RAEzy, RAEzy, sizeof(RAEzy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RBEzy, RBEzy, sizeof(RBEzy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}

	cudaStatus = hipMemcpy(dev_CAEy, CAEy, sizeof(CAEy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_CBEy, CBEy, sizeof(CBEy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RAExz, RAExz, sizeof(RAExz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RBExz, RBExz, sizeof(RBExz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RAEzx, RAEzx, sizeof(RAEzx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RBEzx, RBEzx, sizeof(RBEzx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}

	cudaStatus = hipMemcpy(dev_CAEz, CAEz, sizeof(CAEz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_CBEz, CBEz, sizeof(CBEz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RAExy, RAExy, sizeof(RAExy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RBExy, RBExy, sizeof(RBExy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RAEyx, RAEyx, sizeof(RAEyx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RBEyx, RBEyx, sizeof(RBEyx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}

	cudaStatus = hipMemcpy(dev_CPHx, CPHx, sizeof(CPHx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_CQHx, CQHx, sizeof(CQHx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RAHyz, RAHyz, sizeof(RAHyz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RBHyz, RBHyz, sizeof(RBHyz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RAHzy, RAHzy, sizeof(RAHzy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RBHzy, RBHzy, sizeof(RBHzy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}

	cudaStatus = hipMemcpy(dev_CPHy, CPHy, sizeof(CPHy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_CQHy, CQHy, sizeof(CQHy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RAHxz, RAHxz, sizeof(RAHxz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RBHxz, RBHxz, sizeof(RBHxz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RAHzx, RAHzx, sizeof(RAHzx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RBHzx, RBHzx, sizeof(RBHzx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}

	cudaStatus = hipMemcpy(dev_CPHz, CPHz, sizeof(CPHz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_CQHz, CQHz, sizeof(CQHz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RAHxy, RAHxy, sizeof(RAHxy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RBHxy, RBHxy, sizeof(RBHxy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RAHyx, RAHyx, sizeof(RAHyx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_RBHyx, RBHyx, sizeof(RBHyx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}

	cudaStatus = hipMemcpy(dev_kx_Ey, kx_Ey, sizeof(kx_Ey), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_kx_Ez, kx_Ez, sizeof(kx_Ez), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_ky_Ex, ky_Ex, sizeof(ky_Ex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_ky_Ez, ky_Ez, sizeof(ky_Ez), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_kz_Ex, kz_Ex, sizeof(kz_Ex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_kz_Ey, kz_Ey, sizeof(kz_Ey), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}

	cudaStatus = hipMemcpy(dev_kx_Hy, kx_Hy, sizeof(kx_Hy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_kx_Hz, kx_Hz, sizeof(kx_Hz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_ky_Hx, ky_Hx, sizeof(ky_Hx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_ky_Hz, ky_Hz, sizeof(ky_Hz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_kz_Hx, kz_Hx, sizeof(kz_Hx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	cudaStatus = hipMemcpy(dev_kz_Hy, kz_Hy, sizeof(kz_Hy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}

	cudaStatus = hipMemcpy(dev_source, source, sizeof(source), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!");}
	if (isPianYi)
	{
		cudaStatus = hipMemcpy(dev_E_obs, E_obs, sizeof(E_obs), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); }
	}
}

// 释放显存空间
void gpu_memory_free()
{
	hipFree(dev_Ex);
	hipFree(dev_Ey);
	hipFree(dev_Ez);

	hipFree(dev_UEyz);
	hipFree(dev_UEzy);
	hipFree(dev_UExz);
	hipFree(dev_UEzx);
	hipFree(dev_UExy);
	hipFree(dev_UEyx);

	hipFree(dev_Hx);
	hipFree(dev_Hy);
	hipFree(dev_Hz);

	hipFree(dev_UHyz);
	hipFree(dev_UHzy);
	hipFree(dev_UHxz);
	hipFree(dev_UHzx);
	hipFree(dev_UHxy);
	hipFree(dev_UHyx);

	hipFree(dev_CAEx);
	hipFree(dev_CAEy);
	hipFree(dev_CAEz);

	hipFree(dev_CBEx);
	hipFree(dev_CBEy);
	hipFree(dev_CBEz);

	hipFree(dev_RAEyz);
	hipFree(dev_RAEzy);
	hipFree(dev_RAEzx);
	hipFree(dev_RAExz);
	hipFree(dev_RAExy);
	hipFree(dev_RAEyx);

	hipFree(dev_RBEyz);
	hipFree(dev_RBEzy);
	hipFree(dev_RBEzx);
	hipFree(dev_RBExz);
	hipFree(dev_RBExy);
	hipFree(dev_RBEyx);

	hipFree(dev_CPHx);
	hipFree(dev_CQHx);
	hipFree(dev_CPHy);
	hipFree(dev_CQHy);
	hipFree(dev_CPHz);
	hipFree(dev_CQHz);

	hipFree(dev_RAHyz);
	hipFree(dev_RAHzy);
	hipFree(dev_RAHzx);
	hipFree(dev_RAHxz);
	hipFree(dev_RAHxy);
	hipFree(dev_RAHyx);

	hipFree(dev_RBHyz);
	hipFree(dev_RBHzy);
	hipFree(dev_RBHzx);
	hipFree(dev_RBHxz);
	hipFree(dev_RBHxy);
	hipFree(dev_RBHyx);


	hipFree(fswzx);
	hipFree(fswzy);
	hipFree(fswzz);
	hipFree(jswzx);
	hipFree(jswzy);
	hipFree(jswzz);

	hipFree(dev_E_obs);
	hipFree(dev_V);
	hipFree(dev_source);

	hipFree(dev_kx_Ey);
	hipFree(dev_kx_Ez);
	hipFree(dev_ky_Ex);
	hipFree(dev_ky_Ez);
	hipFree(dev_kz_Ex);
	hipFree(dev_kz_Ey);

	hipFree(dev_kx_Hy);
	hipFree(dev_kx_Hz);
	hipFree(dev_ky_Hx);
	hipFree(dev_ky_Hz);
	hipFree(dev_kz_Hx);
	hipFree(dev_kz_Hy);

	hipFree(dev_Ex_zheng_1);
	hipFree(dev_Ex_zheng_2);
	hipFree(dev_Ex_zheng_3);

	hipFree(dev_Ey_zheng_1);
	hipFree(dev_Ey_zheng_2);
	hipFree(dev_Ey_zheng_3);

	hipFree(dev_Ez_zheng_1);
	hipFree(dev_Ez_zheng_2);
	hipFree(dev_Ez_zheng_3);

	hipFree(dev_Hx_zheng_1);
	hipFree(dev_Hx_zheng_2);
	hipFree(dev_Hx_zheng_3);

	hipFree(dev_Hy_zheng_1);
	hipFree(dev_Hy_zheng_2);
	hipFree(dev_Hy_zheng_3);

	hipFree(dev_Hz_zheng_1);
	hipFree(dev_Hz_zheng_2);
	hipFree(dev_Hz_zheng_3);

	hipFree(dev_Ex_zheng_last);
	hipFree(dev_Ey_zheng_last);
	hipFree(dev_Ez_zheng_last);

	hipFree(dev_Hx_zheng_last);
	hipFree(dev_Hy_zheng_last);
	hipFree(dev_Hz_zheng_last);

	hipFree(dev_fan);
	hipFree(dev_huanyuan);
	hipFree(dev_ns);
	hipFree(dev_zv);
	hipFree(dev_fv);
}

// gpu并行计算UH H UE E
void zheng_yan()
{
	hipError_t cudaStatus = hipSuccess;

	gpu_UHyz << < gridUHyz, blockUHyz >> > (dev_UHyz, dev_RBHyz, dev_RAHyz, dev_Ez);
	gpu_UHzy << < gridUHzy, blockUHzy >> > (dev_UHzy, dev_RBHzy, dev_RAHzy, dev_Ey);
	gpu_UHxy << < gridUHxy, blockUHxy >> > (dev_UHxy, dev_RBHxy, dev_RAHxy, dev_Ey);
	gpu_UHxz << < gridUHxz, blockUHxz >> > (dev_UHxz, dev_RBHxz, dev_RAHxz, dev_Ez);
	gpu_UHyx << < gridUHyx, blockUHyx >> > (dev_UHyx, dev_RBHyx, dev_RAHyx, dev_Ex);
	gpu_UHzx << < gridUHzx, blockUHzx >> > (dev_UHzx, dev_RBHzx, dev_RAHzx, dev_Ex);

	gpu_Hx << < gridHx, blockHx >> > (dev_Hx, dev_CPHx, dev_CQHx, dev_ky_Hx, dev_kz_Hx, dev_Ez, dev_Ey, dev_UHyz, dev_UHzy);
	gpu_Hy << < gridHy, blockHy >> > (dev_Hy, dev_CPHy, dev_CQHy, dev_kz_Hy, dev_kx_Hy, dev_Ex, dev_Ez, dev_UHzx, dev_UHxz);
	gpu_Hz << < gridHz, blockHz >> > (dev_Hz, dev_CPHz, dev_CQHz, dev_kx_Hz, dev_ky_Hz, dev_Ey, dev_Ex, dev_UHxy, dev_UHyx);

	gpu_UExy << < gridUExy, blockUExy >> > (dev_UExy, dev_RBExy, dev_RAExy, dev_Hy);
	gpu_UExz << < gridUExz, blockUExz >> > (dev_UExz, dev_RBExz, dev_RAExz, dev_Hz);
	gpu_UEyx << < gridUEyx, blockUEyx >> > (dev_UEyx, dev_RBEyx, dev_RAEyx, dev_Hx);
	gpu_UEyz << < gridUEyz, blockUEyz >> > (dev_UEyz, dev_RBEyz, dev_RAEyz, dev_Hz);
	gpu_UEzx << < gridUEzx, blockUEzx >> > (dev_UEzx, dev_RBEzx, dev_RAEzx, dev_Hx);
	gpu_UEzy << < gridUEzy, blockUEzy >> > (dev_UEzy, dev_RBEzy, dev_RAEzy, dev_Hy);

	gpu_Ex << < gridEx, blockEx >> > (dev_Ex, dev_CAEx, dev_CBEx, dev_ky_Ex, dev_kz_Ex, dev_Hz, dev_Hy, dev_UEyz, dev_UEzy);
	gpu_Ey << < gridEy, blockEy >> > (dev_Ey, dev_CAEy, dev_CBEy, dev_kz_Ey, dev_kx_Ey, dev_Hx, dev_Hz, dev_UEzx, dev_UExz);
	gpu_Ez << < gridEz, blockEz >> > (dev_Ez, dev_CAEz, dev_CBEz, dev_kx_Ez, dev_ky_Ez, dev_Hy, dev_Hx, dev_UExy, dev_UEyx);

	// 计算过程是否出错?
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("Zhengyan Calc Failed: %s\n", hipGetErrorString(cudaStatus));
	}
}


hipError_t gpu_parallel_one()
{
	hipError_t cudaStatus = hipSuccess;

	int i, j;
	for (i = 0; i < szfsw; i++)
	{
		gpu_memory_set_zero(0);	// flag == 0 将GPU显存中的E*, UE**, H*, UH**, (V, E_obs)置零

		for (j = 0; j < it; j++)
		{
			if (j % 10 == 0)
			{
				printf("i = %3d / %d,  j = %4d / %d\n", i, szfsw, j, it);
			}

			// matlab: Ex(fswzx(i),fswzy(i),fswzz(i))=source(j); 显存到显存
			int idxEx = (fswzx[i] - 1) * (ny + 1) * (nz + 1) + (fswzy[i] - 1) * (nz + 1) + (fswzz[i] - 1);
			cudaStatus = hipMemcpy(&(dev_Ex[idxEx]), &(dev_source[j]), sizeof(float), hipMemcpyDeviceToDevice);
			if (cudaStatus != hipSuccess) { printf("source --> Ex hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus)); return cudaStatus; };

			// 调用GPU运算正演
			zheng_yan();

			// matlab: V(j)=Ex(jswzx(i), jswzy(i), jswzz(i)); 显存到显存
			idxEx = (jswzx[i] - 1) * (ny + 1) * (nz + 1) + (jswzy[i] - 1) * (nz + 1) + (jswzz[i] - 1);
			cudaStatus = hipMemcpy(&(dev_V[j]), &(dev_Ex[idxEx]), sizeof(float), hipMemcpyDeviceToDevice);
			if (cudaStatus != hipSuccess) { printf("Ex --> V hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus)); return cudaStatus; };

			// matlab: E_obs(j,i) = V(j) 显存到内存
			cudaStatus = hipMemcpy(&(E_obs[j][i]), &(dev_V[j]), sizeof(float), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) { printf("V --> E_obs hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus)); return cudaStatus; };
		}
	}

	hipDeviceSynchronize();

	printf("finish calc 1 !\n");

	// 输出结果
	print_E_obs();

	return cudaStatus;
}

hipError_t gpu_parallel_two()
{
	hipError_t cudaStatus = hipSuccess;
	hipMemset(dev_ns, 0, sizeof(ns));
	hipMemset(dev_zv, 0, sizeof(zv));
	hipMemset(dev_fv, 0, sizeof(fv));
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("ns&zv&fv hipMemset Failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	int i, j;
	for (i = 0; i < szfsw; i++)
	{
		// 111111
		gpu_memory_set_zero(1); // flag == 1 将GPU显存中的E*, UE**, H*, UH**, (V, E*_zheng_*, H*_zheng_*, E*_zheng_last, H*_zheng_last, fan, huanyuan)置零
		for (j = 0; j < it; j++)
		{
			if (j % 50 == 0) { printf("i = %3d / %d,  j = %4d / %d\n", i, szfsw, j, it); }

			// 调用GPU运算正演
			zheng_yan();

			gpu_zheng_1 << <grid_zheng_1, block_zheng_1 >> > (
				dev_Ex_zheng_1, dev_Ey_zheng_1, dev_Ez_zheng_1,
				dev_Hx_zheng_1, dev_Hy_zheng_1, dev_Hz_zheng_1,
				dev_Ex, dev_Ey, dev_Ez,
				dev_Hx, dev_Hy, dev_Hz,
				j);

			gpu_zheng_2 << <grid_zheng_2, block_zheng_2 >> > (
				dev_Ex_zheng_2, dev_Ey_zheng_2, dev_Ez_zheng_2,
				dev_Hx_zheng_2, dev_Hy_zheng_2, dev_Hz_zheng_2,
				dev_Ex, dev_Ey, dev_Ez,
				dev_Hx, dev_Hy, dev_Hz,
				j);

			gpu_zheng_3 << <grid_zheng_3, block_zheng_3 >> > (
				dev_Ex_zheng_3, dev_Ey_zheng_3, dev_Ez_zheng_3,
				dev_Hx_zheng_3, dev_Hy_zheng_3, dev_Hz_zheng_3,
				dev_Ex, dev_Ey, dev_Ez,
				dev_Hx, dev_Hy, dev_Hz,
				j);

			gpu_zheng_last << <grid_zheng_last, block_zheng_last >> > (
				dev_Ex_zheng_last, dev_Ey_zheng_last, dev_Ez_zheng_last,
				dev_Hx_zheng_last, dev_Hy_zheng_last, dev_Hz_zheng_last,
				dev_Ex, dev_Ey, dev_Ez,
				dev_Hx, dev_Hy, dev_Hz);
		}

		// 222222
		gpu_memory_set_zero(2);
		for (j = it - 1; j >= 0; j--)
		{
			//if (j % 50 == 0) { printf("i = %3d / %d,  j = %4d / %d\n", i, szfsw, j, it); }

			//Ex(fswzx(i), fswzy(i), fswzz(i)) = E_obs(j, i);
			int idxEx = (fswzx[i] - 1) * (ny + 1) * (nz + 1) + (fswzy[i] - 1) * (nz + 1) + (fswzz[i] - 1);
			int idxE_obs = j * szfsw + i;
			cudaStatus = hipMemcpy(&(dev_Ex[idxEx]), &(dev_E_obs[idxE_obs]), sizeof(float), hipMemcpyDeviceToDevice);
			if (cudaStatus != hipSuccess) 
			{ 
				printf("E_obs --> Ex hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
				return cudaStatus; 
			}

			// 调用GPU运算正演
			zheng_yan();

			// matlab: fan=Ex(npml+1:nx-npml,npml+1:ny-npml,npml+1:nz-npml);
			gpu_fan_huanyuan << <grid_fan_huanyuan, block_fan_huanyuan >> > (dev_fan, dev_Ex);

			if (j == it - 1)
			{
				gpu_back_zheng_last << <grid_zheng_last, block_zheng_last >> > (
					dev_Ex_zheng_last, dev_Ey_zheng_last, dev_Ez_zheng_last,
					dev_Hx_zheng_last, dev_Hy_zheng_last, dev_Hz_zheng_last,
					dev_Ex1, dev_Ey1, dev_Ez1,
					dev_Hx1, dev_Hy1, dev_Hz1);
			}
			else //j < it - 1
			{
				gpu_back_zheng_1 << <grid_zheng_1, block_zheng_1 >> > (
					dev_Ex_zheng_1, dev_Ey_zheng_1, dev_Ez_zheng_1,
					dev_Hx_zheng_1, dev_Hy_zheng_1, dev_Hz_zheng_1,
					dev_Ex1, dev_Ey1, dev_Ez1,
					dev_Hx1, dev_Hy1, dev_Hz1,
					j);

				gpu_back_zheng_2 << <grid_zheng_2, block_zheng_2 >> > (
					dev_Ex_zheng_2, dev_Ey_zheng_2, dev_Ez_zheng_2,
					dev_Hx_zheng_2, dev_Hy_zheng_2, dev_Hz_zheng_2,
					dev_Ex1, dev_Ey1, dev_Ez1,
					dev_Hx1, dev_Hy1, dev_Hz1,
					j);

				gpu_back_zheng_3 << <grid_zheng_3, block_zheng_3 >> > (
					dev_Ex_zheng_3, dev_Ey_zheng_3, dev_Ez_zheng_3,
					dev_Hx_zheng_3, dev_Hy_zheng_3, dev_Hz_zheng_3,
					dev_Ex1, dev_Ey1, dev_Ez1,
					dev_Hx1, dev_Hy1, dev_Hz1,
					j);

				// matlab: Ex1(fswzx(i), fswzy(i), fswzz(i)) = source(j);
				int idxEx1 = (fswzx[i] - 1) * (ny + 1) * (nz + 1) + (fswzy[i] - 1) * (nz + 1) + (fswzz[i] - 1);
				cudaStatus = hipMemcpy(&(dev_Ex1[idxEx1]), &(dev_source[j]), sizeof(float), hipMemcpyDeviceToDevice);
				if (cudaStatus != hipSuccess) 
				{ 
					printf("source --> Ex1 hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus)); 
					return cudaStatus; 
				}

				gpu_H1 << <grid_HE1, block_HE1 >> > (
					dev_Hx1, dev_Hy1, dev_Hz1,
					dev_Ex1, dev_Ey1, dev_Ez1,
					dev_CPHx, dev_CPHy, dev_CPHz,
					dev_CQHx, dev_CQHy, dev_CQHz);
				gpu_E1 << <grid_HE1, block_HE1 >> > (
					dev_Hx1, dev_Hy1, dev_Hz1,
					dev_Ex1, dev_Ey1, dev_Ez1,
					dev_CAEx, dev_CAEy, dev_CAEz,
					dev_CBEx, dev_CBEy, dev_CBEz);
			}

			// matlab: huanyuan=Ex1(npml+1:nx-npml,npml+1:ny-npml,npml+1:nz-npml);
			gpu_fan_huanyuan << <grid_fan_huanyuan, block_fan_huanyuan >> > (dev_huanyuan, dev_Ex);
			gpu_nzf << <grid_nzf, block_nzf >> > (dev_ns, dev_huanyuan, dev_fan);
			gpu_nzf << <grid_nzf, block_nzf >> > (dev_zv, dev_huanyuan, dev_huanyuan);
			gpu_nzf << <grid_nzf, block_nzf >> > (dev_fv, dev_fan, dev_fan);
		}
	}
	cudaStatus = hipMemcpy(ns, dev_ns, sizeof(ns), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		printf("dev_ns --> ns hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(fv, dev_fv, sizeof(fv), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		printf("dev_fv --> fv hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(zv, dev_zv, sizeof(zv), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		printf("dev_zv --> ns hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}



	hipDeviceSynchronize();

	printf("finish calc 2!\n");

	print_nzf("nzf/ns.txt", (float*)ns, nx - 2 * npml, ny - 2 * npml, nz - 2 * npml);
	print_nzf("nzf/fv.txt", (float*)fv, nx - 2 * npml, ny - 2 * npml, nz - 2 * npml);
	print_nzf("nzf/zv.txt", (float*)zv, nx - 2 * npml, ny - 2 * npml, nz - 2 * npml);

	return cudaStatus;
}

/************************************************************************************
* 主函数
************************************************************************************/
int main()
{
	// 切换工作目录
	chdir(path); //linux
	//_chdir(path);
	char str[80];
	printf("Current Dir: %s \n",getcwd(str, 80)); //linux
	//printf("Current Dir: %s \n", _getcwd(str, 80));
	if (Hz_zheng_3 == NULL)
	{
		printf("malloc failed! \n");
		return 1;
	}
	else
	{
		printf("addr of Hz_zheng_3 is %p\n", Hz_zheng_3);
	}
	// 从matlab输出的文本文件中读取数据
	read_data_from_txt();
	printf("Read Data From Txt OK ! \n");

	// 选择运算使用的GPU
	hipError_t cudaStatus = hipSetDevice(cudaDevice);
	if (cudaStatus != hipSuccess) { printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?"); return 1; }
	else { printf("hipSetDevice success!\n"); }

	// 分配显存，把数据从内存传输到显存
	gpu_memory_malloc();
	gpu_memory_copy();

	// 调用gpu运算并输出到文件
	if (isPianYi)
	{
		cudaStatus = gpu_parallel_two();
		if (cudaStatus != hipSuccess) { printf("gpu_parallel_two failed!"); return 1; }
		else { printf("gpu_parallel_two success!\n"); }
	}
	else
	{
		cudaStatus = gpu_parallel_one();
		if (cudaStatus != hipSuccess) { printf("gpu_parallel_one failed!"); return 1; }
		else { printf("gpu_parallel_one success!\n"); }
	}


	// 释放显存空间
	gpu_memory_free();

	// 重置GPU
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) { printf("hipDeviceReset failed!"); return 1; }

	// 释放内存空间
	freeMemory();
	return 0;
}