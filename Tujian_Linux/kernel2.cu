#include "hip/hip_runtime.h"
/************************************************************************************
* Author: Tao Rui
* 版本: V1.0 单卡，Linux版
* 说明: 
*		计算第二部分的并行。
************************************************************************************/

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <string.h>
#include "global_variables.cpp"


/************************************************************************************
* 函数定义
************************************************************************************/

dim3 blockUHyz(nz);
dim3 gridUHyz(npml, nx - 1); //npml: blockIdx.x的变化范围， nx-1就是: blockIdx.y的变化范围
__global__ void calcUHyz(float *UHyz, float *RBHyz, float *RAHyz, float *Ez, const float dy)
{
	/*
	in0 UHyz  nx+1 ny     nz
	in1 RBHyz nx-1 2*npml nz
	in2 RAHyz nx-1 2*npml nz
	in3 Ez    nx+1  ny+1  nz
	UHyz = UHyz * RBHyz + RAHyz * (Ez - Ez) / dy
	运算块大小 nx-1 * npml * nz
	UHyz由5个矩阵相乘或相加得来。
	y维分为了两块

	UHyz(2:nx, [1:npml ny-npml+1:ny], :)=RBHyz .* UHyz(2:nx, [1:npml ny-npml+1:ny], :)...
	+RAHyz ./ dy .* (Ez(2:nx, [2:npml+1 ny-npml+2:ny+1], :) - Ez(2:nx, [1:npml ny-npml+1:ny], :));
	*/

	int ix = blockIdx.y;   // ix in [0, nx - 1)
	int iy = blockIdx.x;   // iy in [0, npml)
	int iz = threadIdx.x;  // iz in [0, nz)

	int lid0 = (ix + 1)*ny*nz + iy * nz + iz; // checked!
	int rid0 = (ix + 1)*ny*nz + (iy + ny - npml) * nz + iz;  //checked!

	int lid1 = ix * (2 * npml)*nz + iy * nz + iz; // checked!
	int rid1 = ix * (2 * npml)*nz + (iy + npml) * nz + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid3 = (ix + 1)*(ny + 1)*nz + (iy + 1)*nz + iz; // checked!
	int rid3 = (ix + 1)*(ny + 1)*nz + (iy + ny - npml + 1)*nz + iz; // checked!

	int lid4 = (ix + 1)*(ny + 1)*nz + iy * nz + iz; // checked!
	int rid4 = (ix + 1)*(ny + 1)*nz + (iy + ny - npml)*nz + iz; // checked!

	UHyz[lid0] = UHyz[lid0] * RBHyz[lid1] + RAHyz[lid2] * (Ez[lid3] - Ez[lid4]) / dy;
	UHyz[rid0] = UHyz[rid0] * RBHyz[rid1] + RAHyz[rid2] * (Ez[rid3] - Ez[rid4]) / dy;
}

dim3 blockUHzy(npml);
dim3 gridUHzy(nx - 1, ny);
__global__ void calcUHzy(float *UHzy, float *RBHzy, float *RAHzy, float *Ey, const float dz)
{
	/*
	in0 UHzy  --size--  nx+1  ny  nz
	in1 RBHzy --size--  nx-1  ny  2*npml
	in2 RAHzy --size--  nx-1  ny  2*npml
	in3 Ey    --size--  nx+1  ny  nz+1
	UHyz = UHyz * RBHyz + RAHyz * (Ez - Ez) / dy
	运算块大小 nx-1 * ny * (5 *npml)
	UHyz由5个矩阵相乘或相加得来。
	z维分为了两块
	UHzy(2:nx, :, [1:npml nz-npml+1:nz])=RBHzy.*UHzy(2:nx, :, [1:npml nz-npml+1:nz])
	+RAHzy./dz.*(Ey(2:nx, :, [2:npml+1 nz-npml+2:nz+1])-Ey(2:nx, :, [1:npml nz-npml+1:nz]));
	*/

	int ix = blockIdx.x;  // ix in [0, nx - 1)
	int iy = blockIdx.y;  // iy in [0, ny)
	int iz = threadIdx.x; // ix in [0, npml)

	int lid0 = (ix + 1) * ny * nz + iy * nz + iz; //checked!
	int rid0 = (ix + 1) * ny * nz + iy * nz + iz + nz - npml; //checked!

	int lid1 = ix * ny * (2 * npml) + iy * (2 * npml) + iz; //checked!
	int rid1 = ix * ny * (2 * npml) + iy * (2 * npml) + iz + npml; //checked!

	int lid2 = lid1;
	int rid2 = rid1;

	int lid4 = (ix + 1) * ny * (nz + 1) + iy * (nz + 1) + iz; //checked!
	int rid4 = (ix + 1) * ny * (nz + 1) + iy * (nz + 1) + iz + nz - npml; //checked!

	int lid3 = lid4 + 1;
	int rid3 = rid4 + 1;

	UHzy[lid0] = UHzy[lid0] * RBHzy[lid1] + RAHzy[lid2] * (Ey[lid3] - Ey[lid4]) / dz;
	UHzy[rid0] = UHzy[rid0] * RBHzy[rid1] + RAHzy[rid2] * (Ey[rid3] - Ey[rid4]) / dz;
}

dim3 blockUHzx(npml);
dim3 gridUHzx(nx, ny - 1);
__global__ void calcUHzx(float *UHzx, float *RBHzx, float *RAHzx, float *Ex, const float dz)
{
	/*
	in0 UHzx  --size--  nx   ny + 1  nz
	in1 RBHzx --size--  nx   ny - 1  2 * npml
	in2 RAHzx --size--  nx   ny - 1  2 * npml
	in3 Ex    --size--  nx   ny + 1  nz + 1
	UHzx = UHzx * RBHzx + RAHzx * (Ez - Ez) / dy
	运算块大小 nx * ny - 1 * npml
	UHzx由5个矩阵相乘或相加得来。
	z维分为了两块  1:npml    -npml:0
	UHzx(:, 2:ny, [1:npml nz - npml + 1:nz])=RBHzx. * UHzx(:, 2:ny, [1:npml nz - npml + 1:nz])
	+RAHzx./dz.*(Ex(:, 2:ny, [2:npml + 1 nz - npml + 2:nz + 1]) - Ex(:, 2:ny, [1:npml nz - npml + 1:nz]));
	*/

	int ix = blockIdx.x;  // ix in [0, nx)
	int iy = blockIdx.y;  // iy in [0, ny - 1)
	int iz = threadIdx.x; // iz in [0, npml)

	int lid0 = ix * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid0 = ix * (ny + 1) * nz + (iy + 1) * nz + iz + nz - npml; // checked!

	int lid1 = ix * (ny - 1) * (2 * npml) + iy * (2 * npml) + iz; // checked!
	int rid1 = ix * (ny - 1) * (2 * npml) + iy * (2 * npml) + iz + npml; // checked!

	int lid2 = lid1;
	int rid2 = rid1;

	int lid4 = ix * (ny + 1) * (nz + 1) + (iy + 1) * (nz + 1) + iz; // checked!
	int rid4 = ix * (ny + 1) * (nz + 1) + (iy + 1) * (nz + 1) + iz + nz - npml; // checked!

	int lid3 = lid4 + 1;
	int rid3 = rid4 + 1;

	UHzx[lid0] = UHzx[lid0] * RBHzx[lid1] + RAHzx[lid2] * (Ex[lid3] - Ex[lid4]) / dz;
	UHzx[rid0] = UHzx[rid0] * RBHzx[rid1] + RAHzx[rid2] * (Ex[rid3] - Ex[rid4]) / dz;
}

dim3 blockUHxz(nz);
dim3 gridUHxz(npml, ny - 1);
__global__ void calcUHxz(float *UHxz, float *RBHxz, float *RAHxz, float *Ez, const float dx)
{
	/*
	in0 UHxz  --size--  nx       ny + 1  nz
	in1 RBHxz --size--  2*npml   ny - 1  nz
	in2 RAHxz --size--  2*npml   ny - 1  nz
	in3 Ez    --size--  nx + 1   ny + 1  nz
	UHxz = UHxz * RBHxz + RAHxz * (Ez - Ez) / dx
	运算块大小 npml * ny - 1 * nz
	UHxz由5个矩阵相乘或相加得来。
	x维分为了两块  1:npml    -npml:0
	UHxz([1:npml nx-npml+1:nx], 2:ny, :)=RBHxz.*UHxz([1:npml nx-npml+1:nx], 2:ny, :)...
	+RAHxz./dx.*(Ez([2:npml+1 nx-npml+2:nx+1], 2:ny, :)-Ez([1:npml nx-npml+1:nx], 2:ny, :));
	*/
	int ix = blockIdx.x;  // ix in [0, npml)
	int iy = blockIdx.y;  // iy in [0, ny - 1)
	int iz = threadIdx.x; // iz in [0, nz)

	int lid0 = ix * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid0 = (ix + nx - npml) * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!

	int lid1 = ix * (ny - 1) * nz + iy * nz + iz; // checked!
	int rid1 = (ix + npml) * (ny - 1) * nz + iy * nz + iz; // checked!

	int lid2 = lid1;
	int rid2 = rid1;

	int lid4 = ix * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid4 = (ix + nx - npml) * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!

	int lid3 = lid4 + (ny + 1) * nz;
	int rid3 = rid4 + (ny + 1) * nz;

	UHxz[lid0] = UHxz[lid0] * RBHxz[lid1] + RAHxz[lid2] * (Ez[lid3] - Ez[lid4]) / dx;
	UHxz[rid0] = UHxz[rid0] * RBHxz[rid1] + RAHxz[rid2] * (Ez[rid3] - Ez[rid4]) / dx;
}

dim3 blockUHxy(nz - 1);
dim3 gridUHxy(npml, ny);
__global__ void calcUHxy(float *UHxy, float *RBHxy, float *RAHxy, float *Ey, const float dx)
{
	/*
	in0 UHxy  --size--  nx       ny      nz + 1
	in1 RBHxy --size--  2*npml   ny      nz - 1
	in2 RAHxy --size--  2*npml   ny      nz - 1
	in3 EY    --size--  nx + 1   ny      nz + 1
	UHxy = UHxy * RBHxy + RAHxy * (Ez - Ez) / dx
	运算块大小 npml * ny * nz - 1
	UHxy由5个矩阵相乘或相加得来。
	x维分为了两块  1:npml    -npml:0
	UHxy([1:npml nx-npml+1:nx], :, 2:nz)=RBHxy.*UHxy([1:npml nx-npml+1:nx], :, 2:nz)...
	+RAHxy./dx.*(Ey([2:npml+1 nx-npml+2:nx+1], :, 2:nz)-Ey([1:npml nx-npml+1:nx], :, 2:nz));
	*/
	int ix = blockIdx.x;  // ix in [0, npml)
	int iy = blockIdx.y;  // iy in [0, ny)
	int iz = threadIdx.x; // iz in [0, nz - 1)

	int lid0 = ix * ny * (nz + 1) + iy * (nz + 1) + iz + 1; // checked!
	int rid0 = (ix + nx - npml) * ny * (nz + 1) + iy * (nz + 1) + iz + 1; //checked

	int lid1 = ix * ny * (nz - 1) + iy * (nz - 1) + iz; // checked!
	int rid1 = (ix + npml) * ny * (nz - 1) + iy * (nz - 1) + iz; // checked!

	int lid2 = lid1;
	int rid2 = rid1;

	int lid4 = ix * ny * (nz + 1) + iy * (nz + 1) + iz + 1; // checked!
	int rid4 = (ix + nx - npml) * ny * (nz + 1) + iy * (nz + 1) + iz + 1; // checked!

	int lid3 = lid4 + ny * (nz + 1);
	int rid3 = rid4 + ny * (nz + 1);

	UHxy[lid0] = UHxy[lid0] * RBHxy[lid1] + RAHxy[lid2] * (Ey[lid3] - Ey[lid4]) / dx;
	UHxy[rid0] = UHxy[rid0] * RBHxy[rid1] + RAHxy[rid2] * (Ey[rid3] - Ey[rid4]) / dx;
}

dim3 blockUHyx(nz - 1);
dim3 gridUHyx(npml, nx);
__global__ void calcUHyx(float *UHyx, float *RBHyx, float *RAHyx, float *Ex, const float dy)
{
	/*
	in0 UHyx  nx   ny     nz + 1
	in1 RBHyx nx   2*npml nz - 1
	in2 RAHyx nx   2*npml nz - 1
	in3 Ex    nx   ny + 1 nz + 1
	UHyx = UHyx * RBHyx + RAHyx * (Ex - Ex) / dy
	运算块大小 nx * npml * nz - 1
	UHyx由5个矩阵相乘或相加得来。
	y维分为了两块

	UHyx(:, [1:npml ny-npml+1:ny], 2:nz)=RBHyx.*UHyx(:, [1:npml ny-npml+1:ny], 2:nz)...
	+RAHyx./dy.*(Ex(:, [2:npml+1 ny-npml+2:ny+1], 2:nz)-Ex(:, [1:npml ny-npml+1:ny], 2:nz));
	*/
	int ix = blockIdx.y;   // ix in [0, nx)
	int iy = blockIdx.x;   // iy in [0, npml)
	int iz = threadIdx.x;  // iz in [0, nz - 1)

	int lid0 = ix * ny * (nz + 1) + iy * (nz + 1) + iz + 1; // checked!
	int rid0 = ix * ny * (nz + 1) + (iy + ny - npml) * (nz + 1) + iz + 1;  //checked!

	int lid1 = ix * (2 * npml) * (nz - 1) + iy * (nz - 1) + iz; // checked!
	int rid1 = ix * (2 * npml) * (nz - 1) + (iy + npml) * (nz - 1) + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!


	int lid4 = ix * (ny + 1) * (nz + 1) + iy * (nz + 1) + iz + 1; // checked!
	int rid4 = ix * (ny + 1) * (nz + 1) + (iy + ny - npml) * (nz + 1) + iz + 1; // checked!


	int lid3 = lid4 + (nz + 1); // checked!
	int rid3 = rid4 + (nz + 1); // checked!

	UHyx[lid0] = UHyx[lid0] * RBHyx[lid1] + RAHyx[lid2] * (Ex[lid3] - Ex[lid4]) / dy;
	UHyx[rid0] = UHyx[rid0] * RBHyx[rid1] + RAHyx[rid2] * (Ex[rid3] - Ex[rid4]) / dy;
}

dim3 blockHx(nz);
dim3 gridHx(nx - 1, ny);
__global__ void calcHx(float *Hx, float *CPHx, float *CQHx, float *ky_Hx, float *kz_Hx, float *Ez, float *Ey, float *UHyz, float *UHzy, const float dy, const float dz)
{
	//
	// * 运算块大小 nx - 1 * ny * nz
	// * Hx(2:nx,:,:)
	//
	int ix = blockIdx.x + 1;
	int iy = blockIdx.y;
	int iz = threadIdx.x;

	int idx = ix * ny * nz + iy * nz + iz;
	int idxEz = ix * (ny + 1)*nz + iy * nz + iz;
	int idxEy = ix * ny * (nz + 1) + iy * (nz + 1) + iz;

	int deltaEz = nz;
	int deltaEy = 1;
	float CQH = CQHx[idx];

	Hx[idx] = Hx[idx] * CPHx[idx]
		- CQH / ky_Hx[idx] * (Ez[idxEz + deltaEz] - Ez[idxEz]) / dy
		+ CQH / kz_Hx[idx] * (Ey[idxEy + deltaEy] - Ey[idxEy]) / dz
		- CQH * UHyz[idx]
		+ CQH * UHzy[idx];
}

dim3 blockHy(nz);
dim3 gridHy(nx, ny - 1);
__global__ void calcHy(float *Hy, float *CPHy, float *CQHy, float *kz_Hy, float *kx_Hy, float *Ex, float *Ez, float *UHzx, float *UHxz, const float dz, const float dx)
{
	//
	// * 运算块大小 nx * ny -1 * nz
	// * Hy(:,2:ny,:)
	//
	int ix = blockIdx.x;
	int iy = blockIdx.y + 1;
	int iz = threadIdx.x;

	int idx = ix * (ny + 1)*nz + iy * nz + iz;
	int idxEx = ix * (ny + 1)*(nz + 1) + iy * (nz + 1) + iz;
	int idxEz = ix * (ny + 1)*nz + iy * nz + iz;

	int deltaEx = 1;
	int deltaEz = (ny + 1)*nz;
	float CQH = CQHy[idx];

	Hy[idx] = Hy[idx] * CPHy[idx]
		- CQH / kz_Hy[idx] * (Ex[idxEx + deltaEx] - Ex[idxEx]) / dz
		+ CQH / kx_Hy[idx] * (Ez[idxEz + deltaEz] - Ez[idxEz]) / dx
		- CQH * UHzx[idx]
		+ CQH * UHxz[idx];
}

dim3 blockHz(nz - 1);
dim3 gridHz(nx, ny);
__global__ void calcHz(float *Hz, float *CPHz, float *CQHz, float *kx_Hz, float *ky_Hz, float *Ey, float *Ex, float *UHxy, float *UHyx, const float dx, const float dy)
{
	//
	// * 运算块大小 nx * ny * nz -1
	// * Hz(:,;,2:nz)
	// * Hz大小为nx ny nz+1
	//
	int ix = blockIdx.x;
	int iy = blockIdx.y;
	int iz = threadIdx.x + 1;

	int idx = ix * ny * (nz + 1) + iy * (nz + 1) + iz;
	int idxEy = ix * ny * (nz + 1) + iy * (nz + 1) + iz;
	int idxEx = ix * (ny + 1) * (nz + 1) + iy * (nz + 1) + iz;
	int deltaEy = ny * (nz + 1);
	int deltaEx = nz + 1;
	float CQH = CQHz[idx];

	Hz[idx] = Hz[idx] * CPHz[idx]
		- CQH / kx_Hz[idx] * (Ey[idxEy + deltaEy] - Ey[idxEy]) / dx
		+ CQH / ky_Hz[idx] * (Ex[idxEx + deltaEx] - Ex[idxEx]) / dy
		- CQH * UHxy[idx]
		+ CQH * UHyx[idx];
}

dim3 blockUEyz(nz - 1);
dim3 gridUEyz(npml - 1, nx);
__global__ void calcUEyz(float *UEyz, float *RBEyz, float *RAEyz, float *Hz, const float dy)
{
	/*
	dim3 blockUEyz(nz - 1);
	dim3 gridUEyz(npml - 1, nx);

	in0 UEyz  nx   ny + 1     nz + 1
	in1 RBEyz nx   2*(npml-1) nz - 1
	in2 RAEyz nx   2*(npml-1) nz - 1
	in3 Hz    nx   ny         nz + 1

	运算块大小 nx * npml - 1 * nz - 1

	UEyz(:, [2:npml ny-npml+2:ny], 2:nz)=RBEyz .* UEyz(:, [2:npml ny-npml+2:ny], 2:nz)...
	+RAEyz ./ dy .* (Hz(:, [2:npml ny-npml+2:ny], 2:nz) - Hz(:, [1:npml-1 ny-npml+1:ny-1], 2:nz));
	*/
	int ix = blockIdx.y;   // ix in [0, nx)
	int iy = blockIdx.x;   // iy in [0, npml - 1)
	int iz = threadIdx.x;  // iz in [0, nz - 1)

	int lid0 = ix * (ny + 1) * (nz + 1) + (iy + 1) * (nz + 1) + (iz + 1); // checked!
	int rid0 = ix * (ny + 1) * (nz + 1) + (iy + 1 + ny - npml) * (nz + 1) + (iz + 1);  //checked!

	int lid1 = ix * (2 * (npml - 1)) * (nz - 1) + iy * (nz - 1) + iz; // checked!
	int rid1 = ix * (2 * (npml - 1)) * (nz - 1) + (iy + npml - 1) * (nz - 1) + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = ix * ny * (nz + 1) + iy * (nz + 1) + (iz + 1); // checked!
	int rid4 = ix * ny * (nz + 1) + (iy + ny - npml) * (nz + 1) + (iz + 1); // checked!

	int lid3 = lid4 + (nz + 1); // checked!
	int rid3 = rid4 + (nz + 1); // checked!

	UEyz[lid0] = UEyz[lid0] * RBEyz[lid1] + RAEyz[lid2] * (Hz[lid3] - Hz[lid4]) / dy;
	UEyz[rid0] = UEyz[rid0] * RBEyz[rid1] + RAEyz[rid2] * (Hz[rid3] - Hz[rid4]) / dy;
}

dim3 blockUEyx(nz - 1);
dim3 gridUEyx(npml - 1, nx);
__global__ void calcUEyx(float *UEyx, float *RBEyx, float *RAEyx, float *Hx, const float dy)
{
	/*
	dim3 blockUEyx(nz - 1);
	dim3 gridUEyx(npml - 1, nx);

	in0 UEyx  nx + 1 ny + 1     nz
	in1 RBEyx nx - 1 2*(npml-1) nz
	in2 RAEyx nx - 1 2*(npml-1) nz
	in3 Hx    nx + 1 ny         nz

	运算块大小 nx * npml-1 * nz-1

	UEyx(2:nx, [2:npml ny-npml+2:ny], :)=RBEyx .* UEyx(2:nx, [2:npml ny-npml+2:ny], :)...
	+RAEyx ./ dy .* (Hx(2:nx, [2:npml ny-npml+2:ny], :) - Hx(2:nx, [1:npml-1 ny-npml+1:ny-1], :));
	*/
	int ix = blockIdx.y;   // ix in [0, nx)
	int iy = blockIdx.x;   // iy in [0, npml - 1)
	int iz = threadIdx.x;  // iz in [0, nz - 1)

	int lid0 = (ix + 1) * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid0 = (ix + 1) * (ny + 1) * nz + (iy + 1 + ny - npml) * nz + iz;  //checked!

	int lid1 = ix * (2 * (npml - 1)) * nz + iy * nz + iz; // checked!
	int rid1 = ix * (2 * (npml - 1)) * nz + (iy + npml - 1) * nz + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = (ix + 1) * ny * nz + iy * nz + iz; // checked!
	int rid4 = (ix + 1) * ny * nz + (iy + ny - npml) * nz + iz; // checked!

	int lid3 = lid4 + nz; // checked!
	int rid3 = rid4 + nz; // checked!

	UEyx[lid0] = UEyx[lid0] * RBEyx[lid1] + RAEyx[lid2] * (Hx[lid3] - Hx[lid4]) / dy;
	UEyx[rid0] = UEyx[rid0] * RBEyx[rid1] + RAEyx[rid2] * (Hx[rid3] - Hx[rid4]) / dy;
}

dim3 blockUExy(nz);
dim3 gridUExy(npml - 1, ny - 1);
__global__ void calcUExy(float *UExy, float *RBExy, float *RAExy, float *Hy, const float dx)
{
	/*
	dim3 blockUExy(nz);
	dim3 gridUExy(npml - 1, ny - 1);

	in0 UExy  nx + 1     ny + 1 nz
	in1 RBExy 2*(npml-1) ny - 1 nz
	in2 RAExy 2*(npml-1) ny - 1 nz
	in3 Hy    nx         ny + 1 nz

	运算块大小 npml-1 * ny-1 * nz

	UExy([2:npml nx-npml+2:nx], 2:ny, :)=RBExy .* UExy([2:npml nx-npml+2:nx], 2:ny, :)...
	+RAExy ./ dx .* (Hy([2:npml nx-npml+2:nx], 2:ny, :) - Hy([1:npml-1 nx-npml+1:nx-1], 2:ny, :));
	*/
	int ix = blockIdx.x;   // ix in [0, npml - 1)
	int iy = blockIdx.y;   // iy in [0, ny - 1)
	int iz = threadIdx.x;  // iz in [0, nz)

	int lid0 = (ix + 1) * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid0 = (ix + 1 + nx - npml) * (ny + 1) * nz + (iy + 1) * nz + iz;  //checked!

	int lid1 = ix * (ny - 1) * nz + iy * nz + iz; // checked!
	int rid1 = (ix + npml - 1) * (ny - 1) * nz + iy * nz + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = ix * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid4 = (ix + nx - npml) * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!

	int lid3 = lid4 + (ny + 1) * nz; // checked!
	int rid3 = rid4 + (ny + 1) * nz; // checked!

	UExy[lid0] = UExy[lid0] * RBExy[lid1] + RAExy[lid2] * (Hy[lid3] - Hy[lid4]) / dx;
	UExy[rid0] = UExy[rid0] * RBExy[rid1] + RAExy[rid2] * (Hy[rid3] - Hy[rid4]) / dx;

}

dim3 blockUExz(nz - 1);
dim3 gridUExz(npml - 1, ny);
__global__ void calcUExz(float *UExz, float *RBExz, float *RAExz, float *Hz, const float dx)
{
	/*
	dim3 blockUExz(nz - 1);
	dim3 gridUExz(npml - 1, ny);

	in0 UExz  nx + 1     ny     nz + 1
	in1 RBExz 2*(npml-1) ny     nz - 1
	in2 RAExz 2*(npml-1) ny     nz - 1
	in3 Hz    nx         ny     nz + 1
	运算块大小 npml-1 * ny * nz-1

	UExz([2:npml nx-npml+2:nx], :, 2:nz)=RBExz .* UExz([2:npml nx-npml+2:nx], :, 2:nz)...
	+RAExz ./ dx .* (Hz([2:npml nx-npml+2:nx], :, 2:nz) - Hz([1:npml-1 nx-npml+1:nx-1], :, 2:nz));
	*/
	int ix = blockIdx.x;   // ix in [0, npml - 1)
	int iy = blockIdx.y;   // iy in [0, ny)
	int iz = threadIdx.x;  // iz in [0, nz - 1)

	int lid0 = (ix + 1) * ny * (nz + 1) + iy * (nz + 1) + (iz + 1); // checked!
	int rid0 = (ix + 1 + nx - npml) * ny * (nz + 1) + iy * (nz + 1) + (iz + 1);  //checked!

	int lid1 = ix * ny * (nz - 1) + iy * (nz - 1) + iz; // checked!
	int rid1 = (ix + npml - 1) * ny * (nz - 1) + iy * (nz - 1) + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = ix * ny * (nz + 1) + iy * (nz + 1) + (iz + 1); // checked!
	int rid4 = (ix + nx - npml) * ny * (nz + 1) + iy * (nz + 1) + (iz + 1); // checked!

	int lid3 = lid4 + ny * (nz + 1); // checked!
	int rid3 = rid4 + ny * (nz + 1); // checked!

	UExz[lid0] = UExz[lid0] * RBExz[lid1] + RAExz[lid2] * (Hz[lid3] - Hz[lid4]) / dx;
	UExz[rid0] = UExz[rid0] * RBExz[rid1] + RAExz[rid2] * (Hz[rid3] - Hz[rid4]) / dx;
}

dim3 blockUEzx(npml - 1);
dim3 gridUEzx(nx - 1, ny);
__global__ void calcUEzx(float *UEzx, float *RBEzx, float *RAEzx, float *Hx, const float dz)
{
	/*
	dim3 blockUEzx(npml - 1);
	dim3 gridUEzx(nx - 1, ny);

	in0 UEzx  nx + 1     ny     nz + 1
	in1 RBEzx nx - 1     ny     2*(npml-1)
	in2 RAEzx nx - 1     ny     2*(npml-1)
	in3 Hx    nx + 1     ny     nz

	运算块大小 nx-1 * ny * npml-1

	UEzx(2:nx, :, [2:npml nz-npml+2:nz])=RBEzx .* UEzx(2:nx, :, [2:npml nz-npml+2:nz])...
	+RAEzx ./ dz .* (Hx(2:nx, :, [2:npml nz-npml+2:nz]) - Hx(2:nx, :, [1:npml-1 nz-npml+1:nz-1]));
	*/
	int ix = blockIdx.x;   // ix in [0, nx)
	int iy = blockIdx.y;   // iy in [0, npml - 1)
	int iz = threadIdx.x;  // iz in [0, nz)

	int lid0 = (ix + 1) * ny * (nz + 1) + iy * (nz + 1) + (iz + 1); // checked!
	int rid0 = (ix + 1) * ny * (nz + 1) + iy * (nz + 1) + (iz + 1 + nz - npml);  //checked!

	int lid1 = ix * ny * (2 * (npml - 1)) + iy * (2 * (npml - 1)) + iz; // checked!
	int rid1 = ix * ny * (2 * (npml - 1)) + iy * (2 * (npml - 1)) + (iz + npml - 1); // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = (ix + 1) * ny * nz + iy * nz + iz; // checked!
	int rid4 = (ix + 1) * ny * nz + iy * nz + (iz + nz - npml); // checked!

	int lid3 = lid4 + 1; // checked!
	int rid3 = rid4 + 1; // checked!

	UEzx[lid0] = UEzx[lid0] * RBEzx[lid1] + RAEzx[lid2] * (Hx[lid3] - Hx[lid4]) / dz;
	UEzx[rid0] = UEzx[rid0] * RBEzx[rid1] + RAEzx[rid2] * (Hx[rid3] - Hx[rid4]) / dz;
}

dim3 blockUEzy(npml - 1);
dim3 gridUEzy(nx, ny - 1);
__global__ void calcUEzy(float *UEzy, float *RBEzy, float *RAEzy, float *Hy, const float dz)
{
	/*
	dim3 blockUEzy(npml - 1);
	dim3 gridUEzy(nx, ny - 1);

	in0 UEzy  nx      ny + 1    nz + 1
	in1 RBEzy nx      ny - 1    2*(npml-1)
	in2 RAEzy nx      ny - 1    2*(npml-1)
	in3 Hy    nx      ny + 1    nz

	运算块大小 nx * ny - 1 * npml-1

	UEzy(:, 2:ny, [2:npml nz-npml+2:nz])=RBEzy.*UEzy(:, 2:ny, [2:npml nz-npml+2:nz])...
	+RAEzy./dz.*(Hy(:, 2:ny, [2:npml nz-npml+2:nz])-Hy(:, 2:ny, [1:npml-1 nz-npml+1:nz-1]));
	*/
	int ix = blockIdx.x;   // ix in [0, nx)
	int iy = blockIdx.y;   // iy in [0, npml - 1)
	int iz = threadIdx.x;  // iz in [0, nz)

	int lid0 = ix * (ny + 1) * (nz + 1) + (iy + 1) * (nz + 1) + (iz + 1); // checked!
	int rid0 = ix * (ny + 1) * (nz + 1) + (iy + 1) * (nz + 1) + (iz + 1 + nz - npml);  //checked!

	int lid1 = ix * (ny - 1) * (2 * (npml - 1)) + iy * (2 * (npml - 1)) + iz; // checked!
	int rid1 = ix * (ny - 1) * (2 * (npml - 1)) + iy * (2 * (npml - 1)) + (iz + npml - 1); // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = ix * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid4 = ix * (ny + 1) * nz + (iy + 1) * nz + (iz + nz - npml); // checked!

	int lid3 = lid4 + 1; // checked!
	int rid3 = rid4 + 1; // checked!

	UEzy[lid0] = UEzy[lid0] * RBEzy[lid1] + RAEzy[lid2] * (Hy[lid3] - Hy[lid4]) / dz;
	UEzy[rid0] = UEzy[rid0] * RBEzy[rid1] + RAEzy[rid2] * (Hy[rid3] - Hy[rid4]) / dz;
}

dim3 blockEx(nz - 1);
dim3 gridEx(nx, ny - 1);
__global__ void calcEx(float *Ex, float *CAEx, float *CBEx, float *ky_Ex, float *kz_Ex, float *Hz, float *Hy, float *UEyz, float *UEzy, const float dy, const float dz)
{
	//
	// * dim3 blockEx(nz-1);
	// * dim3 gridEx(nx, ny-1);
	// * 运算块大小 nx * ny-1 * nz-1
	// * Ex(:, 2:ny, 2:nz)
	//
	int ix = blockIdx.x;      // ix in [0, nx)
	int iy = blockIdx.y + 1;  // iy in [1, ny)
	int iz = threadIdx.x + 1; // iz in [1, nz)

	int idx = ix * (ny + 1) * (nz + 1) + iy * (nz + 1) + iz;
	int idxHz = ix * ny * (nz + 1) + iy * (nz + 1) + iz;
	int idxHy = ix * (ny + 1)*nz + iy * nz + iz;
	int deltaHz = nz + 1;
	int deltaHy = 1;
	float CBE = CBEx[idx];

	Ex[idx] = Ex[idx] * CAEx[idx]
		+ CBE / ky_Ex[idx] * (Hz[idxHz] - Hz[idxHz - deltaHz]) / dy
		- CBE / kz_Ex[idx] * (Hy[idxHy] - Hy[idxHy - deltaHy]) / dz
		+ CBE * UEyz[idx]
		- CBE * UEzy[idx];
}

dim3 blockEy(nz - 1);
dim3 gridEy(nx - 1, ny);
__global__ void calcEy(float *Ey, float *CAEy, float *CBEy, float *kz_Ey, float *kx_Ey, float *Hx, float *Hz, float *UEzx, float *UExz, const float dz, const float dx)
{
	//
	// * dim3 blockEy(nz-1);
	// * dim3 gridEy(nx-1, ny);
	// * 运算块大小 nx-1 * ny * nz-1
	// * Ey(2:nx, :, 2:nz)
	//
	int ix = blockIdx.x + 1;  // ix in [1, nx)
	int iy = blockIdx.y;      // iy in [0, ny)
	int iz = threadIdx.x + 1; // iz in [1, nz)

	int idx = ix * ny * (nz + 1) + iy * (nz + 1) + iz;
	int idxHx = ix * ny * nz + iy * nz + iz;
	int idxHz = ix * ny * (nz + 1) + iy * (nz + 1) + iz;
	int deltaHx = 1;
	int deltaHz = ny * (nz + 1);
	float CBE = CBEy[idx];

	Ey[idx] = Ey[idx] * CAEy[idx]
		+ CBE / kz_Ey[idx] * (Hx[idxHx] - Hx[idxHx - deltaHx]) / dz
		- CBE / kx_Ey[idx] * (Hz[idxHz] - Hz[idxHz - deltaHz]) / dx
		+ CBE * UEzx[idx]
		- CBE * UExz[idx];
}

dim3 blockEz(nz);
dim3 gridEz(nx - 1, ny - 1);
__global__ void calcEz(float *Ez, float *CAEz, float *CBEz, float *kx_Ez, float *ky_Ez, float *Hy, float *Hx, float *UExy, float *UEyx, const float dx, const float dy)
{
	//
	// * dim3 blockEz(nz);
	// * dim3 gridEz(nx-1, ny-1);
	// * 运算块大小 nx-1 * ny-1 * nz
	// * Ez(2:nx, 2:ny, :)
	// * Ez大小为nx ny nz+1
	//
	int ix = blockIdx.x + 1; // ix in [1, nx)
	int iy = blockIdx.y + 1; // iy in [1, ny)
	int iz = threadIdx.x;    // iz in [0, nz)

	int idx = ix * (ny + 1) * nz + iy * nz + iz;
	int idxHy = ix * (ny + 1) * nz + iy * nz + iz;
	int idxHx = ix * ny * nz + iy * nz + iz;
	int deltaHy = (ny + 1) * nz;
	int deltaHx = nz;
	float CBE = CBEz[idx];

	Ez[idx] = Ez[idx] * CAEz[idx]
		+ CBE / kx_Ez[idx] * (Hy[idxHy] - Hy[idxHy - deltaHy]) / dx
		- CBE / ky_Ez[idx] * (Hx[idxHx] - Hx[idxHx - deltaHx]) / dy
		+ CBE * UExy[idx]
		- CBE * UEyx[idx];

}

__global__ void print_dev_matrix(float *A, int i,int j,int k,int xdim,int ydim,int zdim)
{
	int	idx = i * ydim*zdim + j * zdim + k;
	printf("dev_Matrix[%d][%d][%d] = %8f\n", i, j, k, A[idx]);
}

void readInteger(const char *name, int *a, int n1, int n2, int n3)
{
	FILE *fp = fopen(name, "r");
	if (fp == NULL) // 判断文件读入是否正确
	{
		printf("fopen %s error! \n", name);
		return;
	}
	printf("fopen %s ok! \n", name);
	for (int i = 0; i < n1; i++)
	{
		for (int k = 0; k < n3; k++)
		{
			for (int j = 0; j < n2; j++)
			{
				fscanf(fp, "%d", &a[i * n2*n3 + j * n3 + k]); // 读入a[i][j][k]

			}
		}
	}
	printf("read %s OK\n", name);

	fclose(fp);
	return;
}

void readFloat(const char *name, float *a, int n1, int n2, int n3)
{
	FILE *fp = fopen(name, "r");
	if (fp == NULL) // 判断文件读入是否正确
	{
		printf("fopen %s error! \n", name);
		return;
	}
	printf("fopen %s ok! \n", name);
	for (int i = 0; i < n1; i++)
	{
		for (int k = 0; k < n3; k++)
		{
			for (int j = 0; j < n2; j++)
			{
				fscanf(fp, "%f", a + i * n2*n3 + j * n3 + k); // 读入a[i][j][k]			
			}

		}
	}
	printf("read %s OK\n", name);

	fclose(fp);
	return;
}

void readAllData()
{

	readFloat("data/CAEx.txt", (float*)CAEx, nx, ny + 1, nz + 1);
	readFloat("data/CBEx.txt", (float*)CBEx, nx, ny + 1, nz + 1);
	readFloat("data/RAEyz.txt", (float*)RAEyz, nx, 2 * (npml - 1), nz - 1);
	readFloat("data/RBEyz.txt", (float*)RBEyz, nx, 2 * (npml - 1), nz - 1);
	readFloat("data/RAEzy.txt", (float*)RAEzy, nx, ny - 1, 2 * (npml - 1));
	readFloat("data/RBEzy.txt", (float*)RBEzy, nx, ny - 1, 2 * (npml - 1));
	readFloat("data/CAEy.txt", (float*)CAEy, nx + 1, ny, nz + 1);
	readFloat("data/CBEy.txt", (float*)CBEy, nx + 1, ny, nz + 1);
	readFloat("data/RAEzx.txt", (float*)RAEzx, nx - 1, ny, 2 * (npml - 1));
	readFloat("data/RBEzx.txt", (float*)RBEzx, nx - 1, ny, 2 * (npml - 1));
	readFloat("data/RAExz.txt", (float*)RAExz, 2 * (npml - 1), ny, nz - 1);
	readFloat("data/RBExz.txt", (float*)RBExz, 2 * (npml - 1), ny, nz - 1);
	readFloat("data/CAEz.txt", (float*)CAEz, nx + 1, ny + 1, nz);
	readFloat("data/CBEz.txt", (float*)CBEz, nx + 1, ny + 1, nz);
	readFloat("data/RAExy.txt", (float*)RAExy, 2 * (npml - 1), ny - 1, nz);
	readFloat("data/RBExy.txt", (float*)RBExy, 2 * (npml - 1), ny - 1, nz);
	readFloat("data/RAEyx.txt", (float*)RAEyx, nx - 1, 2 * (npml - 1), nz);
	readFloat("data/RBEyx.txt", (float*)RBEyx, nx - 1, 2 * (npml - 1), nz);

	readFloat("data/CPHx.txt", (float*)CPHx, nx + 1, ny, nz);
	readFloat("data/CQHx.txt", (float*)CQHx, nx + 1, ny, nz);
	readFloat("data/RAHyz.txt", (float*)RAHyz, nx - 1, 2 * npml, nz);
	readFloat("data/RBHyz.txt", (float*)RBHyz, nx - 1, 2 * npml, nz);
	readFloat("data/RAHzy.txt", (float*)RAHzy, nx - 1, ny, 2 * npml);
	readFloat("data/RBHzy.txt", (float*)RBHzy, nx - 1, ny, 2 * npml);
	readFloat("data/CPHy.txt", (float*)CPHy, nx, ny + 1, nz);
	readFloat("data/CQHy.txt", (float*)CQHy, nx, ny + 1, nz);
	readFloat("data/RAHzx.txt", (float*)RAHzx, nx, ny - 1, 2 * npml);
	readFloat("data/RBHzx.txt", (float*)RBHzx, nx, ny - 1, 2 * npml);
	readFloat("data/RAHxz.txt", (float*)RAHxz, 2 * npml, ny - 1, nz);
	readFloat("data/RBHxz.txt", (float*)RBHxz, 2 * npml, ny - 1, nz);
	readFloat("data/CPHz.txt", (float*)CPHz, nx, ny, nz + 1);
	readFloat("data/CQHz.txt", (float*)CQHz, nx, ny, nz + 1);
	readFloat("data/RAHxy.txt", (float*)RAHxy, 2 * npml, ny, nz - 1);
	readFloat("data/RBHxy.txt", (float*)RBHxy, 2 * npml, ny, nz - 1);
	readFloat("data/RAHyx.txt", (float*)RAHyx, nx, 2 * npml, nz - 1);
	readFloat("data/RBHyx.txt", (float*)RBHyx, nx, 2 * npml, nz - 1);

	readFloat("data/kx_Ey.txt", (float*)kx_Ey, nx + 1, ny, nz + 1);
	readFloat("data/kx_Ez.txt", (float*)kx_Ez, nx + 1, ny + 1, nz);
	readFloat("data/ky_Ex.txt", (float*)ky_Ex, nx, ny + 1, nz + 1);
	readFloat("data/ky_Ez.txt", (float*)ky_Ez, nx + 1, ny + 1, nz);
	readFloat("data/kz_Ex.txt", (float*)kz_Ex, nx, ny + 1, nz + 1);
	readFloat("data/kz_Ey.txt", (float*)kz_Ey, nx + 1, ny, nz + 1);

	readFloat("data/kx_Hy.txt", (float*)kx_Hy, nx, ny + 1, nz);
	readFloat("data/kx_Hz.txt", (float*)kx_Hz, nx, ny, nz + 1);
	readFloat("data/ky_Hx.txt", (float*)ky_Hx, nx + 1, ny, nz);
	readFloat("data/ky_Hz.txt", (float*)ky_Hz, nx, ny, nz + 1);
	readFloat("data/kz_Hx.txt", (float*)kz_Hx, nx + 1, ny, nz);
	readFloat("data/kz_Hy.txt", (float*)kz_Hy, nx, ny + 1, nz);

	readInteger("data/fswzx.txt", (int*)fswzx, 1, 1, szfsw);
	readInteger("data/fswzy.txt", (int*)fswzy, 1, 1, szfsw);
	readInteger("data/fswzz.txt", (int*)fswzz, 1, 1, szfsw);
	readInteger("data/jswzx.txt", (int*)jswzx, 1, 1, szfsw);
	readInteger("data/jswzy.txt", (int*)jswzy, 1, 1, szfsw);
	readInteger("data/jswzz.txt", (int*)jswzz, 1, 1, szfsw);
	readFloat("data/source.txt", (float*)source, 1, 1, it);
}

void printE_obs()
{
	const char *name = "output/E_obs.txt";
	FILE *fp = fopen(name, "w+");
	if (fp == NULL) // 判断文件读入是否正确
	{
		printf("fopen %s error! \n", name);
	}
	printf("print fopen %s ok! \n", name);

	fprintf(fp, "输出E_obs[%d][%d]\n", it, szfsw);
	fprintf(fp, "共有 %d 行 %d 列 \n", szfsw, it);

	for (int i = 0; i < szfsw; i++)
	{

		for (int j = 0; j < it; j++)
		{
			fprintf(fp, "%8f ", E_obs[j][i]);
		}
		fprintf(fp, "\n");
	}
	printf("print %s OK\n", name);

	fclose(fp);
	return;
}

void gpu_memory_malloc()
{
	//原来内存中存在的数组，数组大小用内存数组大小就行
	hipMalloc((void**)&dev_CAEx, sizeof(CAEx));
	hipMalloc((void**)&dev_CBEx, sizeof(CBEx));
	hipMalloc((void**)&dev_RAEyz, sizeof(RAEyz));
	hipMalloc((void**)&dev_RBEyz, sizeof(RBEyz));
	hipMalloc((void**)&dev_RAEzy, sizeof(RAEzy));
	hipMalloc((void**)&dev_RBEzy, sizeof(RBEzy));

	hipMalloc((void**)&dev_CAEy, sizeof(CAEy));
	hipMalloc((void**)&dev_CBEy, sizeof(CBEy));
	hipMalloc((void**)&dev_RAExz, sizeof(RAExz));
	hipMalloc((void**)&dev_RBExz, sizeof(RBExz));
	hipMalloc((void**)&dev_RAEzx, sizeof(RAEzx));
	hipMalloc((void**)&dev_RBEzx, sizeof(RBEzx));

	hipMalloc((void**)&dev_CAEz, sizeof(CAEz));
	hipMalloc((void**)&dev_CBEz, sizeof(CBEz));
	hipMalloc((void**)&dev_RAExy, sizeof(RAExy));
	hipMalloc((void**)&dev_RBExy, sizeof(RBExy));
	hipMalloc((void**)&dev_RAEyx, sizeof(RAEyx));
	hipMalloc((void**)&dev_RBEyx, sizeof(RBEyx));

	hipMalloc((void**)&dev_CPHx, sizeof(CPHx));
	hipMalloc((void**)&dev_CQHx, sizeof(CQHx));
	hipMalloc((void**)&dev_RAHyz, sizeof(RAHyz));
	hipMalloc((void**)&dev_RBHyz, sizeof(RBHyz));
	hipMalloc((void**)&dev_RAHzy, sizeof(RAHzy));
	hipMalloc((void**)&dev_RBHzy, sizeof(RBHzy));

	hipMalloc((void**)&dev_CPHy, sizeof(CPHy));
	hipMalloc((void**)&dev_CQHy, sizeof(CQHy));
	hipMalloc((void**)&dev_RAHxz, sizeof(RAHxz));
	hipMalloc((void**)&dev_RBHxz, sizeof(RBHxz));
	hipMalloc((void**)&dev_RAHzx, sizeof(RAHzx));
	hipMalloc((void**)&dev_RBHzx, sizeof(RBHzx));

	hipMalloc((void**)&dev_CPHz, sizeof(CPHz));
	hipMalloc((void**)&dev_CQHz, sizeof(CQHz));
	hipMalloc((void**)&dev_RAHxy, sizeof(RAHxy));
	hipMalloc((void**)&dev_RBHxy, sizeof(RBHxy));
	hipMalloc((void**)&dev_RAHyx, sizeof(RAHyx));
	hipMalloc((void**)&dev_RBHyx, sizeof(RBHyx));

	hipMalloc((void**)&dev_kx_Ey, sizeof(kx_Ey));
	hipMalloc((void**)&dev_kx_Ez, sizeof(kx_Ez));
	hipMalloc((void**)&dev_ky_Ex, sizeof(ky_Ex));
	hipMalloc((void**)&dev_ky_Ez, sizeof(ky_Ez));
	hipMalloc((void**)&dev_kz_Ex, sizeof(kz_Ex));
	hipMalloc((void**)&dev_kz_Ey, sizeof(kz_Ey));

	hipMalloc((void**)&dev_kx_Hy, sizeof(kx_Hy));
	hipMalloc((void**)&dev_kx_Hz, sizeof(kx_Hz));
	hipMalloc((void**)&dev_ky_Hx, sizeof(ky_Hx));
	hipMalloc((void**)&dev_ky_Hz, sizeof(ky_Hz));
	hipMalloc((void**)&dev_kz_Hx, sizeof(kz_Hx));
	hipMalloc((void**)&dev_kz_Hy, sizeof(kz_Hy));

	//gpu显存新创建数组，原来内存中不存在
	int szEx = nx * (ny + 1)*(nz + 1);
	int szEy = (nx + 1)*ny*(nz + 1);
	int szEz = (nx + 1)*(ny + 1)*nz;
	int szHx = (nx + 1)*ny*nz;
	int szHy = nx * (ny + 1)*nz;
	int szHz = nx * ny*(nz + 1);
	hipMalloc((void**)&dev_Ex, szEx * sizeof(float));
	hipMalloc((void**)&dev_UEyz, szEx * sizeof(float));
	hipMalloc((void**)&dev_UEzy, szEx * sizeof(float));

	hipMalloc((void**)&dev_Ey, szEy * sizeof(float));
	hipMalloc((void**)&dev_UEzx, szEy * sizeof(float));
	hipMalloc((void**)&dev_UExz, szEy * sizeof(float));

	hipMalloc((void**)&dev_Ez, szEz * sizeof(float));
	hipMalloc((void**)&dev_UExy, szEz * sizeof(float));
	hipMalloc((void**)&dev_UEyx, szEz * sizeof(float));

	hipMalloc((void**)&dev_Hx, szHx * sizeof(float));
	hipMalloc((void**)&dev_UHyz, szHx * sizeof(float));
	hipMalloc((void**)&dev_UHzy, szHx * sizeof(float));

	hipMalloc((void**)&dev_Hy, szHy * sizeof(float));
	hipMalloc((void**)&dev_UHzx, szHy * sizeof(float));
	hipMalloc((void**)&dev_UHxz, szHy * sizeof(float));

	hipMalloc((void**)&dev_Hz, szHz * sizeof(float));
	hipMalloc((void**)&dev_UHxy, szHz * sizeof(float));
	hipMalloc((void**)&dev_UHyx, szHz * sizeof(float));

	hipMalloc((void**)&dev_V, sizeof(V));
	hipMalloc((void**)&dev_E_obs, sizeof(E_obs));
	hipMalloc((void**)&dev_source, sizeof(source));

	// 第二部分并行需要用到的变量
	hipMalloc((void**)&dev_Ex_zheng_1, sizeof(Ex_zheng_1));
	hipMalloc((void**)&dev_Ex_zheng_2, sizeof(Ex_zheng_2));
	hipMalloc((void**)&dev_Ex_zheng_3, sizeof(Ex_zheng_3));

	hipMalloc((void**)&dev_Ey_zheng_1, sizeof(Ey_zheng_1));
	hipMalloc((void**)&dev_Ey_zheng_2, sizeof(Ey_zheng_2));
	hipMalloc((void**)&dev_Ey_zheng_3, sizeof(Ey_zheng_3));

	hipMalloc((void**)&dev_Ez_zheng_1, sizeof(Ez_zheng_1));
	hipMalloc((void**)&dev_Ez_zheng_2, sizeof(Ez_zheng_2));
	hipMalloc((void**)&dev_Ez_zheng_3, sizeof(Ez_zheng_3));

	hipMalloc((void**)&dev_Hx_zheng_1, sizeof(Hx_zheng_1));
	hipMalloc((void**)&dev_Hx_zheng_2, sizeof(Hx_zheng_2));
	hipMalloc((void**)&dev_Hx_zheng_3, sizeof(Hx_zheng_3));

	hipMalloc((void**)&dev_Hy_zheng_1, sizeof(Hy_zheng_1));
	hipMalloc((void**)&dev_Hy_zheng_2, sizeof(Hy_zheng_2));
	hipMalloc((void**)&dev_Hy_zheng_3, sizeof(Hy_zheng_3));

	hipMalloc((void**)&dev_Hz_zheng_1, sizeof(Hz_zheng_1));
	hipMalloc((void**)&dev_Hz_zheng_2, sizeof(Hz_zheng_2));
	hipMalloc((void**)&dev_Hz_zheng_3, sizeof(Hz_zheng_3));

	hipMalloc((void**)&dev_Ex_zheng_last, sizeof(Ex_zheng_last));
	hipMalloc((void**)&dev_Ey_zheng_last, sizeof(Ey_zheng_last));
	hipMalloc((void**)&dev_Ez_zheng_last, sizeof(Ez_zheng_last));
	hipMalloc((void**)&dev_Hx_zheng_last, sizeof(Hx_zheng_last));
	hipMalloc((void**)&dev_Hy_zheng_last, sizeof(Hy_zheng_last));
	hipMalloc((void**)&dev_Hz_zheng_last, sizeof(Hz_zheng_last));

	hipMalloc((void**)&dev_fan, sizeof(fan));
	hipMalloc((void**)&dev_huanyuan,sizeof(huanyuan));

	hipMalloc((void**)&dev_)
}


// flag == 0 将GPU显存中的E*, UE**, H*, UH**, (V, E_obs)置零
// flag == 1 将GPU显存中的E*, UE**, H*, UH**, (V, E*_zheng_*, H*_zheng_*, E*_zheng_last, H*_zheng_last, fan, huanyuan)置零
// flag == 2 将GPU显存中的E*, UE**, H*, UH**, (V, E*1, H*1, )置零
void gpu_memory_set_zero(int flag)
{
	int szEx = nx * (ny + 1)*(nz + 1);
	int szEy = (nx + 1)*ny*(nz + 1);
	int szEz = (nx + 1)*(ny + 1)*nz;
	int szHx = (nx + 1)*ny*nz;
	int szHy = nx * (ny + 1)*nz;
	int szHz = nx * ny*(nz + 1);


	//gpu显存新创建数组，原来内存中不存在
	hipMemset(dev_Ex, 0, szEx * sizeof(float));
	hipMemset(dev_UEyz, 0, szEx * sizeof(float));
	hipMemset(dev_UEzy, 0, szEx * sizeof(float));

	hipMemset(dev_Ey, 0, szEy * sizeof(float));
	hipMemset(dev_UEzx, 0, szEy * sizeof(float));
	hipMemset(dev_UExz, 0, szEy * sizeof(float));

	hipMemset(dev_Ez, 0, szEz * sizeof(float));
	hipMemset(dev_UExy, 0, szEz * sizeof(float));
	hipMemset(dev_UEyx, 0, szEz * sizeof(float));

	hipMemset(dev_Hx, 0, szHx * sizeof(float));
	hipMemset(dev_UHyz, 0, szHx * sizeof(float));
	hipMemset(dev_UHzy, 0, szHx * sizeof(float));

	hipMemset(dev_Hy, 0, szHy * sizeof(float));
	hipMemset(dev_UHzx, 0, szHy * sizeof(float));
	hipMemset(dev_UHxz, 0, szHy * sizeof(float));

	hipMemset(dev_Hz, 0, szHz * sizeof(float));
	hipMemset(dev_UHxy, 0, szHz * sizeof(float));
	hipMemset(dev_UHyx, 0, szHz * sizeof(float));



	if (flag == 0)
	{
		hipMemset(dev_V, 0, sizeof(V));
		hipMemset(dev_E_obs, 0, sizeof(E_obs));		
	} 
	else if (flag == 1)
	{
		hipMemset(dev_V, 0, sizeof(V));

		cudaMemSet(dev_Ex_zheng_1, 0, sizeof(Ex_zheng_1));
		cudaMemSet(dev_Ex_zheng_2, 0, sizeof(Ex_zheng_2));
		cudaMemSet(dev_Ex_zheng_3, 0, sizeof(Ex_zheng_3));

		cudaMemSet(dev_Ey_zheng_1, 0, sizeof(Ey_zheng_1));
		cudaMemSet(dev_Ey_zheng_2, 0, sizeof(Ey_zheng_2));
		cudaMemSet(dev_Ey_zheng_3, 0, sizeof(Ey_zheng_3));

		cudaMemSet(dev_Ez_zheng_1, 0, sizeof(Ez_zheng_1));
		cudaMemSet(dev_Ez_zheng_2, 0, sizeof(Ez_zheng_2));
		cudaMemSet(dev_Ez_zheng_3, 0, sizeof(Ez_zheng_3));

		cudaMemSet(dev_Hx_zheng_1, 0, sizeof(Hx_zheng_1));
		cudaMemSet(dev_Hx_zheng_2, 0, sizeof(Hx_zheng_2));
		cudaMemSet(dev_Hx_zheng_3, 0, sizeof(Hx_zheng_3));

		cudaMemSet(dev_Hy_zheng_1, 0, sizeof(Hy_zheng_1));
		cudaMemSet(dev_Hy_zheng_2, 0, sizeof(Hy_zheng_2));
		cudaMemSet(dev_Hy_zheng_3, 0, sizeof(Hy_zheng_3));

		cudaMemSet(dev_Hz_zheng_1, 0, sizeof(Hz_zheng_1));
		cudaMemSet(dev_Hz_zheng_2, 0, sizeof(Hz_zheng_2));
		cudaMemSet(dev_Hz_zheng_3, 0, sizeof(Hz_zheng_3));

		cudaMemSet(dev_Ex_zheng_last, 0, sizeof(Ex_zheng_last));
		cudaMemSet(dev_Ey_zheng_last, 0, sizeof(Ey_zheng_last));
		cudaMemSet(dev_Ez_zheng_last, 0, sizeof(Ez_zheng_last));
		cudaMemSet(dev_Hx_zheng_last, 0, sizeof(Hx_zheng_last));
		cudaMemSet(dev_Hy_zheng_last, 0, sizeof(Hy_zheng_last));
		cudaMemSet(dev_Hz_zheng_last, 0, sizeof(Hz_zheng_last));

		cudaMemSet(dev_fan, 0, sizeof(fan));
		cudaMemSet(dev_huanyuan, 0, sizeof(huanyuan));
	}
	else
	{
		hipMemset(dev_Ex1, 0, sizeof(Ex1));
		hipMemset(dev_Ey1, 0, sizeof(Ey1));
		hipMemset(dev_Ez1, 0, sizeof(Ez1));

		hipMemset(dev_Hx1, 0, sizeof(Hx1));
		hipMemset(dev_Hy1, 0, sizeof(Hy1));
		hipMemset(dev_Hz1, 0, sizeof(Hz1));
	}

}

// 将内存中的CAE CBE RAE RBE CPH CQH RAH CBH k*_E* k*_H* source复制到显存中
void gpu_memory_copy()
{
	hipError_t cudaStatus;
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_CAEx, CAEx, sizeof(CAEx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_CBEx, CBEx, sizeof(CBEx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAEyz, RAEyz, sizeof(RAEyz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBEyz, RBEyz, sizeof(RBEyz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAEzy, RAEzy, sizeof(RAEzy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBEzy, RBEzy, sizeof(RBEzy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }

	cudaStatus = hipMemcpy(dev_CAEy, CAEy, sizeof(CAEy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_CBEy, CBEy, sizeof(CBEy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAExz, RAExz, sizeof(RAExz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBExz, RBExz, sizeof(RBExz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAEzx, RAEzx, sizeof(RAEzx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBEzx, RBEzx, sizeof(RBEzx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }

	cudaStatus = hipMemcpy(dev_CAEz, CAEz, sizeof(CAEz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_CBEz, CBEz, sizeof(CBEz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAExy, RAExy, sizeof(RAExy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBExy, RBExy, sizeof(RBExy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAEyx, RAEyx, sizeof(RAEyx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBEyx, RBEyx, sizeof(RBEyx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }

	cudaStatus = hipMemcpy(dev_CPHx, CPHx, sizeof(CPHx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_CQHx, CQHx, sizeof(CQHx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAHyz, RAHyz, sizeof(RAHyz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBHyz, RBHyz, sizeof(RBHyz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAHzy, RAHzy, sizeof(RAHzy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBHzy, RBHzy, sizeof(RBHzy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }

	cudaStatus = hipMemcpy(dev_CPHy, CPHy, sizeof(CPHy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_CQHy, CQHy, sizeof(CQHy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAHxz, RAHxz, sizeof(RAHxz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBHxz, RBHxz, sizeof(RBHxz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAHzx, RAHzx, sizeof(RAHzx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBHzx, RBHzx, sizeof(RBHzx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }

	cudaStatus = hipMemcpy(dev_CPHz, CPHz, sizeof(CPHz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_CQHz, CQHz, sizeof(CQHz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAHxy, RAHxy, sizeof(RAHxy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBHxy, RBHxy, sizeof(RBHxy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAHyx, RAHyx, sizeof(RAHyx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBHyx, RBHyx, sizeof(RBHyx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }

	cudaStatus = hipMemcpy(dev_kx_Ey, kx_Ey, sizeof(kx_Ey), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_kx_Ez, kx_Ez, sizeof(kx_Ez), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_ky_Ex, ky_Ex, sizeof(ky_Ex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_ky_Ez, ky_Ez, sizeof(ky_Ez), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_kz_Ex, kz_Ex, sizeof(kz_Ex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_kz_Ey, kz_Ey, sizeof(kz_Ey), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }

	cudaStatus = hipMemcpy(dev_kx_Hy, kx_Hy, sizeof(kx_Hy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_kx_Hz, kx_Hz, sizeof(kx_Hz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_ky_Hx, ky_Hx, sizeof(ky_Hx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_ky_Hz, ky_Hz, sizeof(ky_Hz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_kz_Hx, kz_Hx, sizeof(kz_Hx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_kz_Hy, kz_Hy, sizeof(kz_Hy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }

	cudaStatus = hipMemcpy(dev_source, source, sizeof(source), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
Error:
	return;
}

void gpu_memory_free()
{
	hipFree(dev_CAEx);
	hipFree(dev_CBEx);
	hipFree(dev_RAEyz);
	hipFree(dev_RBEyz);
	hipFree(dev_RAEzy);
	hipFree(dev_RBEzy);

	hipFree(dev_CAEy);
	hipFree(dev_CBEy);
	hipFree(dev_RAExz);
	hipFree(dev_RBExz);
	hipFree(dev_RAEzx);
	hipFree(dev_RBEzx);

	hipFree(dev_CAEz);
	hipFree(dev_CBEz);
	hipFree(dev_RAExy);
	hipFree(dev_RBExy);
	hipFree(dev_RAEyx);
	hipFree(dev_RBEyx);

	hipFree(dev_CPHx);
	hipFree(dev_CQHx);
	hipFree(dev_RAHyz);
	hipFree(dev_RBHyz);
	hipFree(dev_RAHzy);
	hipFree(dev_RBHzy);

	hipFree(dev_CPHy);
	hipFree(dev_CQHy);
	hipFree(dev_RAHxz);
	hipFree(dev_RBHxz);
	hipFree(dev_RAHzx);
	hipFree(dev_RBHzx);

	hipFree(dev_CPHz);
	hipFree(dev_CQHz);
	hipFree(dev_RAHxy);
	hipFree(dev_RBHxy);
	hipFree(dev_RAHyx);
	hipFree(dev_RBHyx);

	hipFree(dev_kx_Ey);
	hipFree(dev_kx_Ez);
	hipFree(dev_ky_Ex);
	hipFree(dev_ky_Ez);
	hipFree(dev_kz_Ex);
	hipFree(dev_kz_Ey);

	hipFree(dev_kx_Hy);
	hipFree(dev_kx_Hz);
	hipFree(dev_ky_Hx);
	hipFree(dev_ky_Hz);
	hipFree(dev_kz_Hx);
	hipFree(dev_kz_Hy);

	hipFree(dev_Ex);
	hipFree(dev_UEyz);
	hipFree(dev_UEzy);

	hipFree(dev_Ey);
	hipFree(dev_UEzx);
	hipFree(dev_UExz);

	hipFree(dev_Ez);
	hipFree(dev_UExy);
	hipFree(dev_UEyx);

	hipFree(dev_Hx);
	hipFree(dev_UHyz);
	hipFree(dev_UHzy);

	hipFree(dev_Hy);
	hipFree(dev_UHzx);
	hipFree(dev_UHxz);

	hipFree(dev_Hz);
	hipFree(dev_UHxy);
	hipFree(dev_UHyx);

	hipFree(dev_V);
	hipFree(dev_E_obs);
	hipFree(dev_source);
}

// 计算UH H UE E
hipError_t zhengYan()
{
	hipError_t cudaStatus;	
	calcUHyz << < gridUHyz, blockUHyz >> > (dev_UHyz, dev_RBHyz, dev_RAHyz, dev_Ez, dy);
	calcUHzy << < gridUHzy, blockUHzy >> > (dev_UHzy, dev_RBHzy, dev_RAHzy, dev_Ey, dz);
	calcUHxy << < gridUHxy, blockUHxy >> > (dev_UHxy, dev_RBHxy, dev_RAHxy, dev_Ey, dx);
	calcUHxz << < gridUHxz, blockUHxz >> > (dev_UHxz, dev_RBHxz, dev_RAHxz, dev_Ez, dx);
	calcUHyx << < gridUHyx, blockUHyx >> > (dev_UHyx, dev_RBHyx, dev_RAHyx, dev_Ex, dy);
	calcUHzx << < gridUHzx, blockUHzx >> > (dev_UHzx, dev_RBHzx, dev_RAHzx, dev_Ex, dz);

	calcHx << < gridHx, blockHx >> > (dev_Hx, dev_CPHx, dev_CQHx, dev_ky_Hx, dev_kz_Hx, dev_Ez, dev_Ey, dev_UHyz, dev_UHzy, dy, dz);
	calcHy << < gridHy, blockHy >> > (dev_Hy, dev_CPHy, dev_CQHy, dev_kz_Hy, dev_kx_Hy, dev_Ex, dev_Ez, dev_UHzx, dev_UHxz, dz, dx);
	calcHz << < gridHz, blockHz >> > (dev_Hz, dev_CPHz, dev_CQHz, dev_kx_Hz, dev_ky_Hz, dev_Ey, dev_Ex, dev_UHxy, dev_UHyx, dx, dy);

	calcUExy << < gridUExy, blockUExy >> > (dev_UExy, dev_RBExy, dev_RAExy, dev_Hy, dx);
	calcUExz << < gridUExz, blockUExz >> > (dev_UExz, dev_RBExz, dev_RAExz, dev_Hz, dx);
	calcUEyx << < gridUEyx, blockUEyx >> > (dev_UEyx, dev_RBEyx, dev_RAEyx, dev_Hx, dy);
	calcUEyz << < gridUEyz, blockUEyz >> > (dev_UEyz, dev_RBEyz, dev_RAEyz, dev_Hz, dy);
	calcUEzx << < gridUEzx, blockUEzx >> > (dev_UEzx, dev_RBEzx, dev_RAEzx, dev_Hx, dz);
	calcUEzy << < gridUEzy, blockUEzy >> > (dev_UEzy, dev_RBEzy, dev_RAEzy, dev_Hy, dz);

	calcEx << < gridEx, blockEx >> > (dev_Ex, dev_CAEx, dev_CBEx, dev_ky_Ex, dev_kz_Ex, dev_Hz, dev_Hy, dev_UEyz, dev_UEzy, dy, dz);
	calcEy << < gridEy, blockEy >> > (dev_Ey, dev_CAEy, dev_CBEy, dev_kz_Ey, dev_kx_Ey, dev_Hx, dev_Hz, dev_UEzx, dev_UExz, dz, dx);
	calcEz << < gridEz, blockEz >> > (dev_Ez, dev_CAEz, dev_CBEz, dev_kx_Ez, dev_ky_Ez, dev_Hy, dev_Hx, dev_UExy, dev_UEyx, dx, dy);

	// 计算过程是否出错?
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) { printf("Zhengyan Calc Failed: %s\n", hipGetErrorString(cudaStatus)); return cudaStatus; }


}

hipError_t gpu_parallel_one()
{
	hipError_t cudaStatus;

	// 调用kernel函数计算。
	int i, j;
	for (i = 0; i < szfsw; i++)
	{
		// flag == 1代表后半部分的并行，将GPU显存中的E*, UE**, H*, UH**, (V, E*_zheng_*, H*_zheng_*, E*_zheng_last, H*_zheng_last, fan, huanyuan)置零
		gpu_memory_set_zero(1);
		for (j = 0; j < it; j++)
		{
			if (j % 200 == 0)
			{
				printf("i = %3d / %d,  j = %4d / %d\n", i, szfsw, j, it);
			}

			// 实现MATLAB中的Ex[fswzx[i] - 1][fswzy[i] - 1][fswzz[i] - 1] = source[j];
			int fidx = (fswzx[i] - 1)*(ny + 1)*(nz + 1) + (fswzy[i] - 1)*(nz + 1) + fswzz[i] - 1;
			cudaStatus = hipMemcpy(&(dev_Ex[fidx]), &(dev_source[j]), sizeof(float), hipMemcpyDeviceToDevice);

			// 实现MATLAB中的V(j)=Ex(jswzx(i), jswzy(i), jswzz(i));
			int jidx = (jswzx[i] - 1)*(ny + 1)*(nz + 1) + (jswzy[i] - 1)*(nz + 1) + jswzz[i] - 1;
			cudaStatus = hipMemcpy(&(dev_V[j]), &(dev_Ex[jidx]), sizeof(float), hipMemcpyDeviceToDevice);
			if (cudaStatus != hipSuccess) { printf("V hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus)); return cudaStatus; };

			cudaStatus = hipMemcpy(&(E_obs[j][i]), &(dev_V[j]), sizeof(float), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) { printf("V hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus)); return cudaStatus; };
		}
	}

	printf("finish calc !\n");

	hipDeviceSynchronize();

	return cudaStatus;
}

hipError_t gpu_parallel_two()
{
	hipError_t cudaStatus;

	// 调用kernel函数计算。
	int i, j;
	for (i = 0; i < szfsw; i++)
	{
		// flag == 1代表后半部分的并行，将GPU显存中的E*, UE**, H*, UH**, (V, E*_zheng_*, H*_zheng_*, E*_zheng_last, H*_zheng_last, fan, huanyuan)置零
		gpu_memory_set_zero(1);
		for (j = 0; j < it; j++)
		{
			if (j % 200 == 0)
			{
				printf("i = %3d / %d,  j = %4d / %d\n", i, szfsw, j, it);
			}

			// 实现MATLAB中的Ex[fswzx[i] - 1][fswzy[i] - 1][fswzz[i] - 1] = source[j];
			int fidx = (fswzx[i] - 1)*(ny + 1)*(nz + 1) + (fswzy[i] - 1)*(nz + 1) + fswzz[i] - 1;
			cudaStatus = hipMemcpy(&(dev_Ex[fidx]), &(dev_source[j]), sizeof(float), hipMemcpyDeviceToDevice);

			// 实现MATLAB中的V(j)=Ex(jswzx(i), jswzy(i), jswzz(i));
			int jidx = (jswzx[i] - 1)*(ny + 1)*(nz + 1) + (jswzy[i] - 1)*(nz + 1) + jswzz[i] - 1;
			cudaStatus = hipMemcpy(&(dev_V[j]), &(dev_Ex[jidx]), sizeof(float), hipMemcpyDeviceToDevice);
			if (cudaStatus != hipSuccess) { printf("V hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus)); return cudaStatus; };

			cudaStatus = hipMemcpy(&(E_obs[j][i]), &(dev_V[j]), sizeof(float), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) { printf("V hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus)); return cudaStatus; };
		}
	}

	printf("finish calc !\n");

	hipDeviceSynchronize();

	return cudaStatus;
}

/************************************************************************************
* 主函数
************************************************************************************/
int main()
{
	printf("c = %e\n", dt);
	readAllData();
	printf("Read All Data OK ! \n");

	// 选择运算使用的GPU
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) { printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?"); return 1; }
	else { printf("hipSetDevice success!\n"); }

	gpu_memory_malloc();
	gpu_memory_copy();

	// 调用gpu运算
	cudaStatus = calcWithCuda();
	if (cudaStatus != hipSuccess) { printf("calcWithCuda failed!"); return 1; }
	else { printf("calcWithCudasuccess!\n"); }

	gpu_memory_free();

	// 重置GPU
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) { printf("hipDeviceReset failed!"); return 1; }

	// 输出结果
	printE_obs();

	return 0;
}