#include "hip/hip_runtime.h"
/************************************************************************************
* Author: Tao Rui
* 版本: V1.0 单卡，Linux版
* 说明: 
*		计算第二部分的并行。
************************************************************************************/

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <string.h>
#include "unistd.h"
#include "global_variables.cpp"


/************************************************************************************
* GPU计算单个矩阵的函数
************************************************************************************/

dim3 blockUHyz(nz);
dim3 gridUHyz(npml, nx - 1); //npml: blockIdx.x的变化范围， nx-1就是: blockIdx.y的变化范围
__global__ void calc_UHyz(float *UHyz, float *RBHyz, float *RAHyz, float *Ez, const float dy)
{
	/*
	in0 UHyz  nx+1 ny     nz
	in1 RBHyz nx-1 2*npml nz
	in2 RAHyz nx-1 2*npml nz
	in3 Ez    nx+1  ny+1  nz
	UHyz = UHyz * RBHyz + RAHyz * (Ez - Ez) / dy
	运算块大小 nx-1 * npml * nz
	UHyz由5个矩阵相乘或相加得来。
	y维分为了两块

	UHyz(2:nx, [1:npml ny-npml+1:ny], :)=RBHyz .* UHyz(2:nx, [1:npml ny-npml+1:ny], :)...
	+RAHyz ./ dy .* (Ez(2:nx, [2:npml+1 ny-npml+2:ny+1], :) - Ez(2:nx, [1:npml ny-npml+1:ny], :));
	*/

	int ix = blockIdx.y;   // ix in [0, nx - 1)
	int iy = blockIdx.x;   // iy in [0, npml)
	int iz = threadIdx.x;  // iz in [0, nz)

	int lid0 = (ix + 1)*ny*nz + iy * nz + iz; // checked!
	int rid0 = (ix + 1)*ny*nz + (iy + ny - npml) * nz + iz;  //checked!

	int lid1 = ix * (2 * npml)*nz + iy * nz + iz; // checked!
	int rid1 = ix * (2 * npml)*nz + (iy + npml) * nz + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid3 = (ix + 1)*(ny + 1)*nz + (iy + 1)*nz + iz; // checked!
	int rid3 = (ix + 1)*(ny + 1)*nz + (iy + ny - npml + 1)*nz + iz; // checked!

	int lid4 = (ix + 1)*(ny + 1)*nz + iy * nz + iz; // checked!
	int rid4 = (ix + 1)*(ny + 1)*nz + (iy + ny - npml)*nz + iz; // checked!

	UHyz[lid0] = UHyz[lid0] * RBHyz[lid1] + RAHyz[lid2] * (Ez[lid3] - Ez[lid4]) / dy;
	UHyz[rid0] = UHyz[rid0] * RBHyz[rid1] + RAHyz[rid2] * (Ez[rid3] - Ez[rid4]) / dy;
}

dim3 blockUHzy(npml);
dim3 gridUHzy(nx - 1, ny);
__global__ void calc_UHzy(float *UHzy, float *RBHzy, float *RAHzy, float *Ey, const float dz)
{
	/*
	in0 UHzy  --size--  nx+1  ny  nz
	in1 RBHzy --size--  nx-1  ny  2*npml
	in2 RAHzy --size--  nx-1  ny  2*npml
	in3 Ey    --size--  nx+1  ny  nz+1
	UHyz = UHyz * RBHyz + RAHyz * (Ez - Ez) / dy
	运算块大小 nx-1 * ny * (5 *npml)
	UHyz由5个矩阵相乘或相加得来。
	z维分为了两块
	UHzy(2:nx, :, [1:npml nz-npml+1:nz])=RBHzy.*UHzy(2:nx, :, [1:npml nz-npml+1:nz])
	+RAHzy./dz.*(Ey(2:nx, :, [2:npml+1 nz-npml+2:nz+1])-Ey(2:nx, :, [1:npml nz-npml+1:nz]));
	*/

	int ix = blockIdx.x;  // ix in [0, nx - 1)
	int iy = blockIdx.y;  // iy in [0, ny)
	int iz = threadIdx.x; // ix in [0, npml)

	int lid0 = (ix + 1) * ny * nz + iy * nz + iz; //checked!
	int rid0 = (ix + 1) * ny * nz + iy * nz + iz + nz - npml; //checked!

	int lid1 = ix * ny * (2 * npml) + iy * (2 * npml) + iz; //checked!
	int rid1 = ix * ny * (2 * npml) + iy * (2 * npml) + iz + npml; //checked!

	int lid2 = lid1;
	int rid2 = rid1;

	int lid4 = (ix + 1) * ny * (nz + 1) + iy * (nz + 1) + iz; //checked!
	int rid4 = (ix + 1) * ny * (nz + 1) + iy * (nz + 1) + iz + nz - npml; //checked!

	int lid3 = lid4 + 1;
	int rid3 = rid4 + 1;

	UHzy[lid0] = UHzy[lid0] * RBHzy[lid1] + RAHzy[lid2] * (Ey[lid3] - Ey[lid4]) / dz;
	UHzy[rid0] = UHzy[rid0] * RBHzy[rid1] + RAHzy[rid2] * (Ey[rid3] - Ey[rid4]) / dz;
}

dim3 blockUHzx(npml);
dim3 gridUHzx(nx, ny - 1);
__global__ void calc_UHzx(float *UHzx, float *RBHzx, float *RAHzx, float *Ex, const float dz)
{
	/*
	in0 UHzx  --size--  nx   ny + 1  nz
	in1 RBHzx --size--  nx   ny - 1  2 * npml
	in2 RAHzx --size--  nx   ny - 1  2 * npml
	in3 Ex    --size--  nx   ny + 1  nz + 1
	UHzx = UHzx * RBHzx + RAHzx * (Ez - Ez) / dy
	运算块大小 nx * ny - 1 * npml
	UHzx由5个矩阵相乘或相加得来。
	z维分为了两块  1:npml    -npml:0
	UHzx(:, 2:ny, [1:npml nz - npml + 1:nz])=RBHzx. * UHzx(:, 2:ny, [1:npml nz - npml + 1:nz])
	+RAHzx./dz.*(Ex(:, 2:ny, [2:npml + 1 nz - npml + 2:nz + 1]) - Ex(:, 2:ny, [1:npml nz - npml + 1:nz]));
	*/

	int ix = blockIdx.x;  // ix in [0, nx)
	int iy = blockIdx.y;  // iy in [0, ny - 1)
	int iz = threadIdx.x; // iz in [0, npml)

	int lid0 = ix * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid0 = ix * (ny + 1) * nz + (iy + 1) * nz + iz + nz - npml; // checked!

	int lid1 = ix * (ny - 1) * (2 * npml) + iy * (2 * npml) + iz; // checked!
	int rid1 = ix * (ny - 1) * (2 * npml) + iy * (2 * npml) + iz + npml; // checked!

	int lid2 = lid1;
	int rid2 = rid1;

	int lid4 = ix * (ny + 1) * (nz + 1) + (iy + 1) * (nz + 1) + iz; // checked!
	int rid4 = ix * (ny + 1) * (nz + 1) + (iy + 1) * (nz + 1) + iz + nz - npml; // checked!

	int lid3 = lid4 + 1;
	int rid3 = rid4 + 1;

	UHzx[lid0] = UHzx[lid0] * RBHzx[lid1] + RAHzx[lid2] * (Ex[lid3] - Ex[lid4]) / dz;
	UHzx[rid0] = UHzx[rid0] * RBHzx[rid1] + RAHzx[rid2] * (Ex[rid3] - Ex[rid4]) / dz;
}

dim3 blockUHxz(nz);
dim3 gridUHxz(npml, ny - 1);
__global__ void calc_UHxz(float *UHxz, float *RBHxz, float *RAHxz, float *Ez, const float dx)
{
	/*
	in0 UHxz  --size--  nx       ny + 1  nz
	in1 RBHxz --size--  2*npml   ny - 1  nz
	in2 RAHxz --size--  2*npml   ny - 1  nz
	in3 Ez    --size--  nx + 1   ny + 1  nz
	UHxz = UHxz * RBHxz + RAHxz * (Ez - Ez) / dx
	运算块大小 npml * ny - 1 * nz
	UHxz由5个矩阵相乘或相加得来。
	x维分为了两块  1:npml    -npml:0
	UHxz([1:npml nx-npml+1:nx], 2:ny, :)=RBHxz.*UHxz([1:npml nx-npml+1:nx], 2:ny, :)...
	+RAHxz./dx.*(Ez([2:npml+1 nx-npml+2:nx+1], 2:ny, :)-Ez([1:npml nx-npml+1:nx], 2:ny, :));
	*/
	int ix = blockIdx.x;  // ix in [0, npml)
	int iy = blockIdx.y;  // iy in [0, ny - 1)
	int iz = threadIdx.x; // iz in [0, nz)

	int lid0 = ix * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid0 = (ix + nx - npml) * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!

	int lid1 = ix * (ny - 1) * nz + iy * nz + iz; // checked!
	int rid1 = (ix + npml) * (ny - 1) * nz + iy * nz + iz; // checked!

	int lid2 = lid1;
	int rid2 = rid1;

	int lid4 = ix * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid4 = (ix + nx - npml) * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!

	int lid3 = lid4 + (ny + 1) * nz;
	int rid3 = rid4 + (ny + 1) * nz;

	UHxz[lid0] = UHxz[lid0] * RBHxz[lid1] + RAHxz[lid2] * (Ez[lid3] - Ez[lid4]) / dx;
	UHxz[rid0] = UHxz[rid0] * RBHxz[rid1] + RAHxz[rid2] * (Ez[rid3] - Ez[rid4]) / dx;
}

dim3 blockUHxy(nz - 1);
dim3 gridUHxy(npml, ny);
__global__ void calc_UHxy(float *UHxy, float *RBHxy, float *RAHxy, float *Ey, const float dx)
{
	/*
	in0 UHxy  --size--  nx       ny      nz + 1
	in1 RBHxy --size--  2*npml   ny      nz - 1
	in2 RAHxy --size--  2*npml   ny      nz - 1
	in3 EY    --size--  nx + 1   ny      nz + 1
	UHxy = UHxy * RBHxy + RAHxy * (Ez - Ez) / dx
	运算块大小 npml * ny * nz - 1
	UHxy由5个矩阵相乘或相加得来。
	x维分为了两块  1:npml    -npml:0
	UHxy([1:npml nx-npml+1:nx], :, 2:nz)=RBHxy.*UHxy([1:npml nx-npml+1:nx], :, 2:nz)...
	+RAHxy./dx.*(Ey([2:npml+1 nx-npml+2:nx+1], :, 2:nz)-Ey([1:npml nx-npml+1:nx], :, 2:nz));
	*/
	int ix = blockIdx.x;  // ix in [0, npml)
	int iy = blockIdx.y;  // iy in [0, ny)
	int iz = threadIdx.x; // iz in [0, nz - 1)

	int lid0 = ix * ny * (nz + 1) + iy * (nz + 1) + iz + 1; // checked!
	int rid0 = (ix + nx - npml) * ny * (nz + 1) + iy * (nz + 1) + iz + 1; //checked

	int lid1 = ix * ny * (nz - 1) + iy * (nz - 1) + iz; // checked!
	int rid1 = (ix + npml) * ny * (nz - 1) + iy * (nz - 1) + iz; // checked!

	int lid2 = lid1;
	int rid2 = rid1;

	int lid4 = ix * ny * (nz + 1) + iy * (nz + 1) + iz + 1; // checked!
	int rid4 = (ix + nx - npml) * ny * (nz + 1) + iy * (nz + 1) + iz + 1; // checked!

	int lid3 = lid4 + ny * (nz + 1);
	int rid3 = rid4 + ny * (nz + 1);

	UHxy[lid0] = UHxy[lid0] * RBHxy[lid1] + RAHxy[lid2] * (Ey[lid3] - Ey[lid4]) / dx;
	UHxy[rid0] = UHxy[rid0] * RBHxy[rid1] + RAHxy[rid2] * (Ey[rid3] - Ey[rid4]) / dx;
}

dim3 blockUHyx(nz - 1);
dim3 gridUHyx(npml, nx);
__global__ void calc_UHyx(float *UHyx, float *RBHyx, float *RAHyx, float *Ex, const float dy)
{
	/*
	in0 UHyx  nx   ny     nz + 1
	in1 RBHyx nx   2*npml nz - 1
	in2 RAHyx nx   2*npml nz - 1
	in3 Ex    nx   ny + 1 nz + 1
	UHyx = UHyx * RBHyx + RAHyx * (Ex - Ex) / dy
	运算块大小 nx * npml * nz - 1
	UHyx由5个矩阵相乘或相加得来。
	y维分为了两块

	UHyx(:, [1:npml ny-npml+1:ny], 2:nz)=RBHyx.*UHyx(:, [1:npml ny-npml+1:ny], 2:nz)...
	+RAHyx./dy.*(Ex(:, [2:npml+1 ny-npml+2:ny+1], 2:nz)-Ex(:, [1:npml ny-npml+1:ny], 2:nz));
	*/
	int ix = blockIdx.y;   // ix in [0, nx)
	int iy = blockIdx.x;   // iy in [0, npml)
	int iz = threadIdx.x;  // iz in [0, nz - 1)

	int lid0 = ix * ny * (nz + 1) + iy * (nz + 1) + iz + 1; // checked!
	int rid0 = ix * ny * (nz + 1) + (iy + ny - npml) * (nz + 1) + iz + 1;  //checked!

	int lid1 = ix * (2 * npml) * (nz - 1) + iy * (nz - 1) + iz; // checked!
	int rid1 = ix * (2 * npml) * (nz - 1) + (iy + npml) * (nz - 1) + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!


	int lid4 = ix * (ny + 1) * (nz + 1) + iy * (nz + 1) + iz + 1; // checked!
	int rid4 = ix * (ny + 1) * (nz + 1) + (iy + ny - npml) * (nz + 1) + iz + 1; // checked!


	int lid3 = lid4 + (nz + 1); // checked!
	int rid3 = rid4 + (nz + 1); // checked!

	UHyx[lid0] = UHyx[lid0] * RBHyx[lid1] + RAHyx[lid2] * (Ex[lid3] - Ex[lid4]) / dy;
	UHyx[rid0] = UHyx[rid0] * RBHyx[rid1] + RAHyx[rid2] * (Ex[rid3] - Ex[rid4]) / dy;
}

dim3 blockHx(nz);
dim3 gridHx(nx - 1, ny);
__global__ void calc_Hx(float *Hx, float *CPHx, float *CQHx, float *ky_Hx, float *kz_Hx, float *Ez, float *Ey, float *UHyz, float *UHzy, const float dy, const float dz)
{
	//
	// * 运算块大小 nx - 1 * ny * nz
	// * Hx(2:nx,:,:)
	//
	int ix = blockIdx.x + 1;
	int iy = blockIdx.y;
	int iz = threadIdx.x;

	int idx = ix * ny * nz + iy * nz + iz;
	int idxEz = ix * (ny + 1)*nz + iy * nz + iz;
	int idxEy = ix * ny * (nz + 1) + iy * (nz + 1) + iz;

	int deltaEz = nz;
	int deltaEy = 1;
	float CQH = CQHx[idx];

	Hx[idx] = Hx[idx] * CPHx[idx]
		- CQH / ky_Hx[idx] * (Ez[idxEz + deltaEz] - Ez[idxEz]) / dy
		+ CQH / kz_Hx[idx] * (Ey[idxEy + deltaEy] - Ey[idxEy]) / dz
		- CQH * UHyz[idx]
		+ CQH * UHzy[idx];
}

dim3 blockHy(nz);
dim3 gridHy(nx, ny - 1);
__global__ void calc_Hy(float *Hy, float *CPHy, float *CQHy, float *kz_Hy, float *kx_Hy, float *Ex, float *Ez, float *UHzx, float *UHxz, const float dz, const float dx)
{
	//
	// * 运算块大小 nx * ny -1 * nz
	// * Hy(:,2:ny,:)
	//
	int ix = blockIdx.x;
	int iy = blockIdx.y + 1;
	int iz = threadIdx.x;

	int idx = ix * (ny + 1)*nz + iy * nz + iz;
	int idxEx = ix * (ny + 1)*(nz + 1) + iy * (nz + 1) + iz;
	int idxEz = ix * (ny + 1)*nz + iy * nz + iz;

	int deltaEx = 1;
	int deltaEz = (ny + 1)*nz;
	float CQH = CQHy[idx];

	Hy[idx] = Hy[idx] * CPHy[idx]
		- CQH / kz_Hy[idx] * (Ex[idxEx + deltaEx] - Ex[idxEx]) / dz
		+ CQH / kx_Hy[idx] * (Ez[idxEz + deltaEz] - Ez[idxEz]) / dx
		- CQH * UHzx[idx]
		+ CQH * UHxz[idx];
}

dim3 blockHz(nz - 1);
dim3 gridHz(nx, ny);
__global__ void calc_Hz(float *Hz, float *CPHz, float *CQHz, float *kx_Hz, float *ky_Hz, float *Ey, float *Ex, float *UHxy, float *UHyx, const float dx, const float dy)
{
	//
	// * 运算块大小 nx * ny * nz -1
	// * Hz(:,;,2:nz)
	// * Hz大小为nx ny nz+1
	//
	int ix = blockIdx.x;
	int iy = blockIdx.y;
	int iz = threadIdx.x + 1;

	int idx = ix * ny * (nz + 1) + iy * (nz + 1) + iz;
	int idxEy = ix * ny * (nz + 1) + iy * (nz + 1) + iz;
	int idxEx = ix * (ny + 1) * (nz + 1) + iy * (nz + 1) + iz;
	int deltaEy = ny * (nz + 1);
	int deltaEx = nz + 1;
	float CQH = CQHz[idx];

	Hz[idx] = Hz[idx] * CPHz[idx]
		- CQH / kx_Hz[idx] * (Ey[idxEy + deltaEy] - Ey[idxEy]) / dx
		+ CQH / ky_Hz[idx] * (Ex[idxEx + deltaEx] - Ex[idxEx]) / dy
		- CQH * UHxy[idx]
		+ CQH * UHyx[idx];
}

dim3 blockUEyz(nz - 1);
dim3 gridUEyz(npml - 1, nx);
__global__ void calc_UEyz(float *UEyz, float *RBEyz, float *RAEyz, float *Hz, const float dy)
{
	/*
	dim3 blockUEyz(nz - 1);
	dim3 gridUEyz(npml - 1, nx);

	in0 UEyz  nx   ny + 1     nz + 1
	in1 RBEyz nx   2*(npml-1) nz - 1
	in2 RAEyz nx   2*(npml-1) nz - 1
	in3 Hz    nx   ny         nz + 1

	运算块大小 nx * npml - 1 * nz - 1

	UEyz(:, [2:npml ny-npml+2:ny], 2:nz)=RBEyz .* UEyz(:, [2:npml ny-npml+2:ny], 2:nz)...
	+RAEyz ./ dy .* (Hz(:, [2:npml ny-npml+2:ny], 2:nz) - Hz(:, [1:npml-1 ny-npml+1:ny-1], 2:nz));
	*/
	int ix = blockIdx.y;   // ix in [0, nx)
	int iy = blockIdx.x;   // iy in [0, npml - 1)
	int iz = threadIdx.x;  // iz in [0, nz - 1)

	int lid0 = ix * (ny + 1) * (nz + 1) + (iy + 1) * (nz + 1) + (iz + 1); // checked!
	int rid0 = ix * (ny + 1) * (nz + 1) + (iy + 1 + ny - npml) * (nz + 1) + (iz + 1);  //checked!

	int lid1 = ix * (2 * (npml - 1)) * (nz - 1) + iy * (nz - 1) + iz; // checked!
	int rid1 = ix * (2 * (npml - 1)) * (nz - 1) + (iy + npml - 1) * (nz - 1) + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = ix * ny * (nz + 1) + iy * (nz + 1) + (iz + 1); // checked!
	int rid4 = ix * ny * (nz + 1) + (iy + ny - npml) * (nz + 1) + (iz + 1); // checked!

	int lid3 = lid4 + (nz + 1); // checked!
	int rid3 = rid4 + (nz + 1); // checked!

	UEyz[lid0] = UEyz[lid0] * RBEyz[lid1] + RAEyz[lid2] * (Hz[lid3] - Hz[lid4]) / dy;
	UEyz[rid0] = UEyz[rid0] * RBEyz[rid1] + RAEyz[rid2] * (Hz[rid3] - Hz[rid4]) / dy;
}

dim3 blockUEyx(nz - 1);
dim3 gridUEyx(npml - 1, nx);
__global__ void calc_UEyx(float *UEyx, float *RBEyx, float *RAEyx, float *Hx, const float dy)
{
	/*
	dim3 blockUEyx(nz - 1);
	dim3 gridUEyx(npml - 1, nx);

	in0 UEyx  nx + 1 ny + 1     nz
	in1 RBEyx nx - 1 2*(npml-1) nz
	in2 RAEyx nx - 1 2*(npml-1) nz
	in3 Hx    nx + 1 ny         nz

	运算块大小 nx * npml-1 * nz-1

	UEyx(2:nx, [2:npml ny-npml+2:ny], :)=RBEyx .* UEyx(2:nx, [2:npml ny-npml+2:ny], :)...
	+RAEyx ./ dy .* (Hx(2:nx, [2:npml ny-npml+2:ny], :) - Hx(2:nx, [1:npml-1 ny-npml+1:ny-1], :));
	*/
	int ix = blockIdx.y;   // ix in [0, nx)
	int iy = blockIdx.x;   // iy in [0, npml - 1)
	int iz = threadIdx.x;  // iz in [0, nz - 1)

	int lid0 = (ix + 1) * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid0 = (ix + 1) * (ny + 1) * nz + (iy + 1 + ny - npml) * nz + iz;  //checked!

	int lid1 = ix * (2 * (npml - 1)) * nz + iy * nz + iz; // checked!
	int rid1 = ix * (2 * (npml - 1)) * nz + (iy + npml - 1) * nz + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = (ix + 1) * ny * nz + iy * nz + iz; // checked!
	int rid4 = (ix + 1) * ny * nz + (iy + ny - npml) * nz + iz; // checked!

	int lid3 = lid4 + nz; // checked!
	int rid3 = rid4 + nz; // checked!

	UEyx[lid0] = UEyx[lid0] * RBEyx[lid1] + RAEyx[lid2] * (Hx[lid3] - Hx[lid4]) / dy;
	UEyx[rid0] = UEyx[rid0] * RBEyx[rid1] + RAEyx[rid2] * (Hx[rid3] - Hx[rid4]) / dy;
}

dim3 blockUExy(nz);
dim3 gridUExy(npml - 1, ny - 1);
__global__ void calc_UExy(float *UExy, float *RBExy, float *RAExy, float *Hy, const float dx)
{
	/*
	dim3 blockUExy(nz);
	dim3 gridUExy(npml - 1, ny - 1);

	in0 UExy  nx + 1     ny + 1 nz
	in1 RBExy 2*(npml-1) ny - 1 nz
	in2 RAExy 2*(npml-1) ny - 1 nz
	in3 Hy    nx         ny + 1 nz

	运算块大小 npml-1 * ny-1 * nz

	UExy([2:npml nx-npml+2:nx], 2:ny, :)=RBExy .* UExy([2:npml nx-npml+2:nx], 2:ny, :)...
	+RAExy ./ dx .* (Hy([2:npml nx-npml+2:nx], 2:ny, :) - Hy([1:npml-1 nx-npml+1:nx-1], 2:ny, :));
	*/
	int ix = blockIdx.x;   // ix in [0, npml - 1)
	int iy = blockIdx.y;   // iy in [0, ny - 1)
	int iz = threadIdx.x;  // iz in [0, nz)

	int lid0 = (ix + 1) * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid0 = (ix + 1 + nx - npml) * (ny + 1) * nz + (iy + 1) * nz + iz;  //checked!

	int lid1 = ix * (ny - 1) * nz + iy * nz + iz; // checked!
	int rid1 = (ix + npml - 1) * (ny - 1) * nz + iy * nz + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = ix * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid4 = (ix + nx - npml) * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!

	int lid3 = lid4 + (ny + 1) * nz; // checked!
	int rid3 = rid4 + (ny + 1) * nz; // checked!

	UExy[lid0] = UExy[lid0] * RBExy[lid1] + RAExy[lid2] * (Hy[lid3] - Hy[lid4]) / dx;
	UExy[rid0] = UExy[rid0] * RBExy[rid1] + RAExy[rid2] * (Hy[rid3] - Hy[rid4]) / dx;
}

dim3 blockUExz(nz - 1);
dim3 gridUExz(npml - 1, ny);
__global__ void calc_UExz(float *UExz, float *RBExz, float *RAExz, float *Hz, const float dx)
{
	/*
	dim3 blockUExz(nz - 1);
	dim3 gridUExz(npml - 1, ny);

	in0 UExz  nx + 1     ny     nz + 1
	in1 RBExz 2*(npml-1) ny     nz - 1
	in2 RAExz 2*(npml-1) ny     nz - 1
	in3 Hz    nx         ny     nz + 1
	运算块大小 npml-1 * ny * nz-1

	UExz([2:npml nx-npml+2:nx], :, 2:nz)=RBExz .* UExz([2:npml nx-npml+2:nx], :, 2:nz)...
	+RAExz ./ dx .* (Hz([2:npml nx-npml+2:nx], :, 2:nz) - Hz([1:npml-1 nx-npml+1:nx-1], :, 2:nz));
	*/
	int ix = blockIdx.x;   // ix in [0, npml - 1)
	int iy = blockIdx.y;   // iy in [0, ny)
	int iz = threadIdx.x;  // iz in [0, nz - 1)

	int lid0 = (ix + 1) * ny * (nz + 1) + iy * (nz + 1) + (iz + 1); // checked!
	int rid0 = (ix + 1 + nx - npml) * ny * (nz + 1) + iy * (nz + 1) + (iz + 1);  //checked!

	int lid1 = ix * ny * (nz - 1) + iy * (nz - 1) + iz; // checked!
	int rid1 = (ix + npml - 1) * ny * (nz - 1) + iy * (nz - 1) + iz; // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = ix * ny * (nz + 1) + iy * (nz + 1) + (iz + 1); // checked!
	int rid4 = (ix + nx - npml) * ny * (nz + 1) + iy * (nz + 1) + (iz + 1); // checked!

	int lid3 = lid4 + ny * (nz + 1); // checked!
	int rid3 = rid4 + ny * (nz + 1); // checked!

	UExz[lid0] = UExz[lid0] * RBExz[lid1] + RAExz[lid2] * (Hz[lid3] - Hz[lid4]) / dx;
	UExz[rid0] = UExz[rid0] * RBExz[rid1] + RAExz[rid2] * (Hz[rid3] - Hz[rid4]) / dx;
}

dim3 blockUEzx(npml - 1);
dim3 gridUEzx(nx - 1, ny);
__global__ void calc_UEzx(float *UEzx, float *RBEzx, float *RAEzx, float *Hx, const float dz)
{
	/*
	dim3 blockUEzx(npml - 1);
	dim3 gridUEzx(nx - 1, ny);

	in0 UEzx  nx + 1     ny     nz + 1
	in1 RBEzx nx - 1     ny     2*(npml-1)
	in2 RAEzx nx - 1     ny     2*(npml-1)
	in3 Hx    nx + 1     ny     nz

	运算块大小 nx-1 * ny * npml-1

	UEzx(2:nx, :, [2:npml nz-npml+2:nz])=RBEzx .* UEzx(2:nx, :, [2:npml nz-npml+2:nz])...
	+RAEzx ./ dz .* (Hx(2:nx, :, [2:npml nz-npml+2:nz]) - Hx(2:nx, :, [1:npml-1 nz-npml+1:nz-1]));
	*/
	int ix = blockIdx.x;   // ix in [0, nx)
	int iy = blockIdx.y;   // iy in [0, npml - 1)
	int iz = threadIdx.x;  // iz in [0, nz)

	int lid0 = (ix + 1) * ny * (nz + 1) + iy * (nz + 1) + (iz + 1); // checked!
	int rid0 = (ix + 1) * ny * (nz + 1) + iy * (nz + 1) + (iz + 1 + nz - npml);  //checked!

	int lid1 = ix * ny * (2 * (npml - 1)) + iy * (2 * (npml - 1)) + iz; // checked!
	int rid1 = ix * ny * (2 * (npml - 1)) + iy * (2 * (npml - 1)) + (iz + npml - 1); // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = (ix + 1) * ny * nz + iy * nz + iz; // checked!
	int rid4 = (ix + 1) * ny * nz + iy * nz + (iz + nz - npml); // checked!

	int lid3 = lid4 + 1; // checked!
	int rid3 = rid4 + 1; // checked!

	UEzx[lid0] = UEzx[lid0] * RBEzx[lid1] + RAEzx[lid2] * (Hx[lid3] - Hx[lid4]) / dz;
	UEzx[rid0] = UEzx[rid0] * RBEzx[rid1] + RAEzx[rid2] * (Hx[rid3] - Hx[rid4]) / dz;
}

dim3 blockUEzy(npml - 1);
dim3 gridUEzy(nx, ny - 1);
__global__ void calc_UEzy(float *UEzy, float *RBEzy, float *RAEzy, float *Hy, const float dz)
{
	/*
	dim3 blockUEzy(npml - 1);
	dim3 gridUEzy(nx, ny - 1);

	in0 UEzy  nx      ny + 1    nz + 1
	in1 RBEzy nx      ny - 1    2*(npml-1)
	in2 RAEzy nx      ny - 1    2*(npml-1)
	in3 Hy    nx      ny + 1    nz

	运算块大小 nx * ny - 1 * npml-1

	UEzy(:, 2:ny, [2:npml nz-npml+2:nz])=RBEzy.*UEzy(:, 2:ny, [2:npml nz-npml+2:nz])...
	+RAEzy./dz.*(Hy(:, 2:ny, [2:npml nz-npml+2:nz])-Hy(:, 2:ny, [1:npml-1 nz-npml+1:nz-1]));
	*/
	int ix = blockIdx.x;   // ix in [0, nx)
	int iy = blockIdx.y;   // iy in [0, npml - 1)
	int iz = threadIdx.x;  // iz in [0, nz)

	int lid0 = ix * (ny + 1) * (nz + 1) + (iy + 1) * (nz + 1) + (iz + 1); // checked!
	int rid0 = ix * (ny + 1) * (nz + 1) + (iy + 1) * (nz + 1) + (iz + 1 + nz - npml);  //checked!

	int lid1 = ix * (ny - 1) * (2 * (npml - 1)) + iy * (2 * (npml - 1)) + iz; // checked!
	int rid1 = ix * (ny - 1) * (2 * (npml - 1)) + iy * (2 * (npml - 1)) + (iz + npml - 1); // checked!

	int lid2 = lid1; // checked!
	int rid2 = rid1; // checked!

	int lid4 = ix * (ny + 1) * nz + (iy + 1) * nz + iz; // checked!
	int rid4 = ix * (ny + 1) * nz + (iy + 1) * nz + (iz + nz - npml); // checked!

	int lid3 = lid4 + 1; // checked!
	int rid3 = rid4 + 1; // checked!

	UEzy[lid0] = UEzy[lid0] * RBEzy[lid1] + RAEzy[lid2] * (Hy[lid3] - Hy[lid4]) / dz;
	UEzy[rid0] = UEzy[rid0] * RBEzy[rid1] + RAEzy[rid2] * (Hy[rid3] - Hy[rid4]) / dz;
}

dim3 blockEx(nz - 1);
dim3 gridEx(nx, ny - 1);
__global__ void calc_Ex(float *Ex, float *CAEx, float *CBEx, float *ky_Ex, float *kz_Ex, float *Hz, float *Hy, float *UEyz, float *UEzy, const float dy, const float dz)
{
	//
	// * dim3 blockEx(nz-1);
	// * dim3 gridEx(nx, ny-1);
	// * 运算块大小 nx * ny-1 * nz-1
	// * Ex(:, 2:ny, 2:nz)
	//
	int ix = blockIdx.x;      // ix in [0, nx)
	int iy = blockIdx.y + 1;  // iy in [1, ny)
	int iz = threadIdx.x + 1; // iz in [1, nz)

	int idx = ix * (ny + 1) * (nz + 1) + iy * (nz + 1) + iz;
	int idxHz = ix * ny * (nz + 1) + iy * (nz + 1) + iz;
	int idxHy = ix * (ny + 1)*nz + iy * nz + iz;
	int deltaHz = nz + 1;
	int deltaHy = 1;
	float CBE = CBEx[idx];

	Ex[idx] = Ex[idx] * CAEx[idx]
		+ CBE / ky_Ex[idx] * (Hz[idxHz] - Hz[idxHz - deltaHz]) / dy
		- CBE / kz_Ex[idx] * (Hy[idxHy] - Hy[idxHy - deltaHy]) / dz
		+ CBE * UEyz[idx]
		- CBE * UEzy[idx];
}

dim3 blockEy(nz - 1);
dim3 gridEy(nx - 1, ny);
__global__ void calc_Ey(float *Ey, float *CAEy, float *CBEy, float *kz_Ey, float *kx_Ey, float *Hx, float *Hz, float *UEzx, float *UExz, const float dz, const float dx)
{
	//
	// * dim3 blockEy(nz-1);
	// * dim3 gridEy(nx-1, ny);
	// * 运算块大小 nx-1 * ny * nz-1
	// * Ey(2:nx, :, 2:nz)
	//
	int ix = blockIdx.x + 1;  // ix in [1, nx)
	int iy = blockIdx.y;      // iy in [0, ny)
	int iz = threadIdx.x + 1; // iz in [1, nz)

	int idx = ix * ny * (nz + 1) + iy * (nz + 1) + iz;
	int idxHx = ix * ny * nz + iy * nz + iz;
	int idxHz = ix * ny * (nz + 1) + iy * (nz + 1) + iz;
	int deltaHx = 1;
	int deltaHz = ny * (nz + 1);
	float CBE = CBEy[idx];

	Ey[idx] = Ey[idx] * CAEy[idx]
		+ CBE / kz_Ey[idx] * (Hx[idxHx] - Hx[idxHx - deltaHx]) / dz
		- CBE / kx_Ey[idx] * (Hz[idxHz] - Hz[idxHz - deltaHz]) / dx
		+ CBE * UEzx[idx]
		- CBE * UExz[idx];
}

dim3 blockEz(nz);
dim3 gridEz(nx - 1, ny - 1);
__global__ void calc_Ez(float *Ez, float *CAEz, float *CBEz, float *kx_Ez, float *ky_Ez, float *Hy, float *Hx, float *UExy, float *UEyx, const float dx, const float dy)
{
	//
	// * dim3 blockEz(nz);
	// * dim3 gridEz(nx-1, ny-1);
	// * 运算块大小 nx-1 * ny-1 * nz
	// * Ez(2:nx, 2:ny, :)
	// * Ez大小为nx ny nz+1
	//
	int ix = blockIdx.x + 1; // ix in [1, nx)
	int iy = blockIdx.y + 1; // iy in [1, ny)
	int iz = threadIdx.x;    // iz in [0, nz)

	int idx = ix * (ny + 1) * nz + iy * nz + iz;
	int idxHy = ix * (ny + 1) * nz + iy * nz + iz;
	int idxHx = ix * ny * nz + iy * nz + iz;
	int deltaHy = (ny + 1) * nz;
	int deltaHx = nz;
	float CBE = CBEz[idx];

	Ez[idx] = Ez[idx] * CAEz[idx]
		+ CBE / kx_Ez[idx] * (Hy[idxHy] - Hy[idxHy - deltaHy]) / dx
		- CBE / ky_Ez[idx] * (Hx[idxHx] - Hx[idxHx - deltaHx]) / dy
		+ CBE * UExy[idx]
		- CBE * UEyx[idx];
}

// 用src矩阵中x*y*z大小的块填充dst矩阵
// 矩阵块在src矩阵中的位置为(x_offset, y_offset, z_offset)
__global__ void gpu_copy_data_3D(float *dst, int dst_xsize, int dst_ysize, int dst_zsize, 
								 float *src, int src_xsize, int src_ysize, int src_zsize, 
								 int x, int y, int z, 
								 int x_offset, int y_offset, int z_offset)
{
	int ix = blockIdx.x;
	int iy = blockIdx.y;
	int iz = threadIdx.x;    

	int src_idx = ix * dst_ysize * dst_zsize + iy * dst_zsize + iz;
	int dst_idx = (ix + y_offset) * src_ysize * src_zsize + (iy + y_offset) * src_zsize + (iz + z_offset);
	dst[dst_idx] = src[src_idx];
}

__global__ void print_dev_matrix(float *A, int i,int j,int k,int xdim,int ydim,int zdim)
{
	int	idx = i * ydim*zdim + j * zdim + k;
	printf("dev_Matrix[%d][%d][%d] = %8f\n", i, j, k, A[idx]);
}

void read_int(const char *name, int *a, int n1, int n2, int n3)
{
	FILE *fp = fopen(name, "r");
	if (fp == NULL) // 判断文件读入是否正确
	{
		printf("fopen %s error! \n", name);
		return;
	}
	printf("fopen %s ok! \n", name);
	for (int i = 0; i < n1; i++)
	{
		for (int k = 0; k < n3; k++)
		{
			for (int j = 0; j < n2; j++)
			{
				fscanf(fp, "%d", &a[i * n2*n3 + j * n3 + k]); // 读入a[i][j][k]

			}
		}
	}
	printf("read %s OK\n", name);

	fclose(fp);
	return;
}

void read_float(const char *name, float *a, int n1, int n2, int n3)
{
	FILE *fp = fopen(name, "r");
	if (fp == NULL) // 判断文件读入是否正确
	{
		printf("fopen %s error! \n", name);
		return;
	}
	printf("fopen %s ok! \n", name);
	for (int i = 0; i < n1; i++)
	{
		for (int k = 0; k < n3; k++)
		{
			for (int j = 0; j < n2; j++)
			{
				fscanf(fp, "%f", a + i * n2*n3 + j * n3 + k); // 读入a[i][j][k]			
			}

		}
	}
	printf("read %s OK\n", name);

	fclose(fp);
	return;
}

void read_data_from_txt()
{

	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\CAEx.txt", (float*)CAEx, nx, ny + 1, nz + 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\CBEx.txt", (float*)CBEx, nx, ny + 1, nz + 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RAEyz.txt", (float*)RAEyz, nx, 2 * (npml - 1), nz - 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RBEyz.txt", (float*)RBEyz, nx, 2 * (npml - 1), nz - 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RAEzy.txt", (float*)RAEzy, nx, ny - 1, 2 * (npml - 1));
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RBEzy.txt", (float*)RBEzy, nx, ny - 1, 2 * (npml - 1));
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\CAEy.txt", (float*)CAEy, nx + 1, ny, nz + 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\CBEy.txt", (float*)CBEy, nx + 1, ny, nz + 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RAEzx.txt", (float*)RAEzx, nx - 1, ny, 2 * (npml - 1));
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RBEzx.txt", (float*)RBEzx, nx - 1, ny, 2 * (npml - 1));
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RAExz.txt", (float*)RAExz, 2 * (npml - 1), ny, nz - 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RBExz.txt", (float*)RBExz, 2 * (npml - 1), ny, nz - 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\CAEz.txt", (float*)CAEz, nx + 1, ny + 1, nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\CBEz.txt", (float*)CBEz, nx + 1, ny + 1, nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RAExy.txt", (float*)RAExy, 2 * (npml - 1), ny - 1, nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RBExy.txt", (float*)RBExy, 2 * (npml - 1), ny - 1, nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RAEyx.txt", (float*)RAEyx, nx - 1, 2 * (npml - 1), nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RBEyx.txt", (float*)RBEyx, nx - 1, 2 * (npml - 1), nz);

	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\CPHx.txt", (float*)CPHx, nx + 1, ny, nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\CQHx.txt", (float*)CQHx, nx + 1, ny, nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RAHyz.txt", (float*)RAHyz, nx - 1, 2 * npml, nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RBHyz.txt", (float*)RBHyz, nx - 1, 2 * npml, nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RAHzy.txt", (float*)RAHzy, nx - 1, ny, 2 * npml);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RBHzy.txt", (float*)RBHzy, nx - 1, ny, 2 * npml);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\CPHy.txt", (float*)CPHy, nx, ny + 1, nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\CQHy.txt", (float*)CQHy, nx, ny + 1, nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RAHzx.txt", (float*)RAHzx, nx, ny - 1, 2 * npml);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RBHzx.txt", (float*)RBHzx, nx, ny - 1, 2 * npml);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RAHxz.txt", (float*)RAHxz, 2 * npml, ny - 1, nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RBHxz.txt", (float*)RBHxz, 2 * npml, ny - 1, nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\CPHz.txt", (float*)CPHz, nx, ny, nz + 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\CQHz.txt", (float*)CQHz, nx, ny, nz + 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RAHxy.txt", (float*)RAHxy, 2 * npml, ny, nz - 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RBHxy.txt", (float*)RBHxy, 2 * npml, ny, nz - 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RAHyx.txt", (float*)RAHyx, nx, 2 * npml, nz - 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\RBHyx.txt", (float*)RBHyx, nx, 2 * npml, nz - 1);

	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\kx_Ey.txt", (float*)kx_Ey, nx + 1, ny, nz + 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\kx_Ez.txt", (float*)kx_Ez, nx + 1, ny + 1, nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\ky_Ex.txt", (float*)ky_Ex, nx, ny + 1, nz + 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\ky_Ez.txt", (float*)ky_Ez, nx + 1, ny + 1, nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\kz_Ex.txt", (float*)kz_Ex, nx, ny + 1, nz + 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\kz_Ey.txt", (float*)kz_Ey, nx + 1, ny, nz + 1);

	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\kx_Hy.txt", (float*)kx_Hy, nx, ny + 1, nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\kx_Hz.txt", (float*)kx_Hz, nx, ny, nz + 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\ky_Hx.txt", (float*)ky_Hx, nx + 1, ny, nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\ky_Hz.txt", (float*)ky_Hz, nx, ny, nz + 1);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\kz_Hx.txt", (float*)kz_Hx, nx + 1, ny, nz);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\kz_Hy.txt", (float*)kz_Hy, nx, ny + 1, nz);

	read_int("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\fswzx.txt", (int*)fswzx, 1, 1, szfsw);
	read_int("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\fswzy.txt", (int*)fswzy, 1, 1, szfsw);
	read_int("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\fswzz.txt", (int*)fswzz, 1, 1, szfsw);
	read_int("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\jswzx.txt", (int*)jswzx, 1, 1, szfsw);
	read_int("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\jswzy.txt", (int*)jswzy, 1, 1, szfsw);
	read_int("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\jswzz.txt", (int*)jswzz, 1, 1, szfsw);
	read_float("C:\\Users\\sky\\Desktop\\Tujian_VS\\data\\source.txt", (float*)source, 1, 1, it);
}

void print_E_obs()
{
	const char *name = "C:\\Users\\sky\\Desktop\\Tujian_VS\\output\\E_obs.txt";
	FILE *fp = fopen(name, "w+");
	if (fp == NULL) // 判断文件读入是否正确
	{
		printf("fopen %s error! \n", name);
	}
	printf("print fopen %s ok! \n", name);

	fprintf(fp, "输出E_obs[%d][%d]\n", it, szfsw);
	fprintf(fp, "共有 %d 行 %d 列 \n", szfsw, it);

	for (int i = 0; i < szfsw; i++)
	{

		for (int j = 0; j < it; j++)
		{
			fprintf(fp, "%8f ", E_obs[j][i]);
		}
		fprintf(fp, "\n");
	}
	printf("print %s OK\n", name);

	fclose(fp);
	return;
}

void gpu_memory_malloc()
{
	hipError_t cudaStatus = hipSuccess;
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	//原来内存中存在的数组，数组大小用内存数组大小就行
	cudaStatus = hipMalloc((void**)&dev_CAEx, sizeof(CAEx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_CBEx, sizeof(CBEx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RAEyz, sizeof(RAEyz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RBEyz, sizeof(RBEyz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RAEzy, sizeof(RAEzy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RBEzy, sizeof(RBEzy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_CAEy, sizeof(CAEy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_CBEy, sizeof(CBEy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RAExz, sizeof(RAExz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RBExz, sizeof(RBExz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RAEzx, sizeof(RAEzx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RBEzx, sizeof(RBEzx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_CAEz, sizeof(CAEz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_CBEz, sizeof(CBEz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RAExy, sizeof(RAExy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RBExy, sizeof(RBExy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RAEyx, sizeof(RAEyx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RBEyx, sizeof(RBEyx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_CPHx, sizeof(CPHx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_CQHx, sizeof(CQHx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RAHyz, sizeof(RAHyz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RBHyz, sizeof(RBHyz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RAHzy, sizeof(RAHzy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RBHzy, sizeof(RBHzy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_CPHy, sizeof(CPHy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_CQHy, sizeof(CQHy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RAHxz, sizeof(RAHxz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RBHxz, sizeof(RBHxz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RAHzx, sizeof(RAHzx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RBHzx, sizeof(RBHzx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_CPHz, sizeof(CPHz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_CQHz, sizeof(CQHz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RAHxy, sizeof(RAHxy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RBHxy, sizeof(RBHxy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RAHyx, sizeof(RAHyx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_RBHyx, sizeof(RBHyx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_kx_Ey, sizeof(kx_Ey));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_kx_Ez, sizeof(kx_Ez));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_ky_Ex, sizeof(ky_Ex));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_ky_Ez, sizeof(ky_Ez));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_kz_Ex, sizeof(kz_Ex));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_kz_Ey, sizeof(kz_Ey));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_kx_Hy, sizeof(kx_Hy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_kx_Hz, sizeof(kx_Hz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_ky_Hx, sizeof(ky_Hx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_ky_Hz, sizeof(ky_Hz));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_kz_Hx, sizeof(kz_Hx));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_kz_Hy, sizeof(kz_Hy));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	//gpu显存新创建数组，原来内存中不存在
	int szEx = nx * (ny + 1)*(nz + 1);
	int szEy = (nx + 1)*ny*(nz + 1);
	int szEz = (nx + 1)*(ny + 1)*nz;
	int szHx = (nx + 1)*ny*nz;
	int szHy = nx * (ny + 1)*nz;
	int szHz = nx * ny*(nz + 1);

	cudaStatus = hipMalloc((void**)&dev_Ex, szEx * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_UEyz, szEx * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_UEzy, szEx * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_Ey, szEy * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_UEzx, szEy * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_UExz, szEy * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_Ez, szEz * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_UExy, szEz * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_UEyx, szEz * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_Hx, szHx * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_UHyz, szHx * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_UHzy, szHx * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_Hy, szHy * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_UHzx, szHy * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_UHxz, szHy * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_Hz, szHz * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_UHxy, szHz * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_UHyx, szHz * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_V, sizeof(V));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_E_obs, sizeof(E_obs));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_source, sizeof(source));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	// 第二部分并行需要用到的变量

	cudaStatus = hipMalloc((void**)&dev_fan, sizeof(fan));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_huanyuan, sizeof(huanyuan));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_Ex1, sizeof(Ex1));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Ey1, sizeof(Ey1));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Ez1, sizeof(Ez1));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_Hx1, sizeof(Hx1));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Hy1, sizeof(Hy1));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Hz1, sizeof(Hz1));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); goto Error; }

	// 超大数组

	cudaStatus = hipMalloc((void**)&dev_Ex_zheng_1, (it)*(2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Ex_zheng_2, (it)*(nx - 2 * npml)*(2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Ex_zheng_3, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_Ey_zheng_1, (it)*(2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Ey_zheng_2, (it)*(nx - 2 * npml)*(2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Ey_zheng_3, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)Ez_zheng_1, (it)*(2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)Ez_zheng_2, (it)*(nx - 2 * npml)*(2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)Ez_zheng_3, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_Hx_zheng_1, (it)*(2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Hx_zheng_2, (it)*(nx - 2 * npml)*(2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Hx_zheng_3, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_Hy_zheng_1, (it)*(2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Hy_zheng_2, (it)*(nx - 2 * npml)*(2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Hy_zheng_3, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_Hz_zheng_1, (it)*(2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Hz_zheng_2, (it)*(nx - 2 * npml)*(2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Hz_zheng_3, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_Ex_zheng_last, (nx - 2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Ey_zheng_last, (nx - 2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Ez_zheng_last, (nx - 2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_Hx_zheng_last, (nx - 2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Hy_zheng_last, (nx - 2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_Hz_zheng_last, (nx - 2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
	if (cudaStatus != hipSuccess) { printf("hipMalloc Super Big Array failed!"); goto Error; }
Error:
	return;
}

// flag == 0 将GPU显存中的E*, UE**, H*, UH**, (V, E_obs)置零
// flag == 1 将GPU显存中的E*, UE**, H*, UH**, (V, E*_zheng_*, H*_zheng_*, E*_zheng_last, H*_zheng_last, fan, huanyuan)置零
// flag == 2 将GPU显存中的E*, UE**, H*, UH**, (V, E*1, H*1, )置零
void gpu_memory_set_zero(int flag)
{
	int szEx = nx * (ny + 1)*(nz + 1);
	int szEy = (nx + 1)*ny*(nz + 1);
	int szEz = (nx + 1)*(ny + 1)*nz;
	int szHx = (nx + 1)*ny*nz;
	int szHy = nx * (ny + 1)*nz;
	int szHz = nx * ny*(nz + 1);

	//gpu显存新创建数组，原来内存中不存在
	hipMemset(dev_Ex, 0, szEx * sizeof(float));
	hipMemset(dev_UEyz, 0, szEx * sizeof(float));
	hipMemset(dev_UEzy, 0, szEx * sizeof(float));

	hipMemset(dev_Ey, 0, szEy * sizeof(float));
	hipMemset(dev_UEzx, 0, szEy * sizeof(float));
	hipMemset(dev_UExz, 0, szEy * sizeof(float));

	hipMemset(dev_Ez, 0, szEz * sizeof(float));
	hipMemset(dev_UExy, 0, szEz * sizeof(float));
	hipMemset(dev_UEyx, 0, szEz * sizeof(float));

	hipMemset(dev_Hx, 0, szHx * sizeof(float));
	hipMemset(dev_UHyz, 0, szHx * sizeof(float));
	hipMemset(dev_UHzy, 0, szHx * sizeof(float));

	hipMemset(dev_Hy, 0, szHy * sizeof(float));
	hipMemset(dev_UHzx, 0, szHy * sizeof(float));
	hipMemset(dev_UHxz, 0, szHy * sizeof(float));

	hipMemset(dev_Hz, 0, szHz * sizeof(float));
	hipMemset(dev_UHxy, 0, szHz * sizeof(float));
	hipMemset(dev_UHyx, 0, szHz * sizeof(float));

	if (flag == 0)
	{
		hipMemset(dev_V, 0, sizeof(V));
		hipMemset(dev_E_obs, 0, sizeof(E_obs));		
	} 
	else if (flag == 1)
	{
		hipMemset(dev_V, 0, sizeof(V));

		hipMemset(dev_Ex_zheng_1, 0, (it)*(2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Ex_zheng_2, 0, (it)*(nx - 2 * npml)*(2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Ex_zheng_3, 0, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npml) * sizeof(float));

		hipMemset(dev_Ey_zheng_1, 0, (it)*(2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Ey_zheng_2, 0, (it)*(nx - 2 * npml)*(2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Ey_zheng_3, 0, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npml) * sizeof(float));

		hipMemset(dev_Ez_zheng_1, 0, (it)*(2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Ez_zheng_2, 0, (it)*(nx - 2 * npml)*(2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Ez_zheng_3, 0, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npml) * sizeof(float));

		hipMemset(dev_Hx_zheng_1, 0, (it)*(2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Hx_zheng_2, 0, (it)*(nx - 2 * npml)*(2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Hx_zheng_3, 0, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npml) * sizeof(float));

		hipMemset(dev_Hy_zheng_1, 0, (it)*(2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Hy_zheng_2, 0, (it)*(nx - 2 * npml)*(2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Hy_zheng_3, 0, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npml) * sizeof(float));

		hipMemset(dev_Hz_zheng_1, 0, (it)*(2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Hz_zheng_2, 0, (it)*(nx - 2 * npml)*(2 * npml)*(nz - 2 * npml) * sizeof(float));
		hipMemset(dev_Hz_zheng_3, 0, (it)*(nx - 2 * npml)*(ny - 2 * npml)*(2 * npml) * sizeof(float));

		size_t sz_last = (nx - 2 * npml)*(ny - 2 * npml)*(nz - 2 * npml) * sizeof(float);
		hipMemset(dev_Ex_zheng_last, 0, sz_last);
		hipMemset(dev_Ey_zheng_last, 0, sz_last);
		hipMemset(dev_Ez_zheng_last, 0, sz_last);

		hipMemset(dev_Hx_zheng_last, 0, sz_last);
		hipMemset(dev_Hy_zheng_last, 0, sz_last);
		hipMemset(dev_Hz_zheng_last, 0, sz_last);

		hipMemset(dev_fan, 0, sizeof(fan));
		hipMemset(dev_huanyuan, 0, sizeof(huanyuan));
	}
	else
	{
		hipMemset(dev_Ex1, 0, sizeof(Ex1));
		hipMemset(dev_Ey1, 0, sizeof(Ey1));
		hipMemset(dev_Ez1, 0, sizeof(Ez1));

		hipMemset(dev_Hx1, 0, sizeof(Hx1));
		hipMemset(dev_Hy1, 0, sizeof(Hy1));
		hipMemset(dev_Hz1, 0, sizeof(Hz1));
	}
}

// 将内存中的CAE CBE RAE RBE CPH CQH RAH CBH k*_E* k*_H* source复制到显存中
void gpu_memory_copy()
{
	hipError_t cudaStatus;
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_CAEx, CAEx, sizeof(CAEx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_CBEx, CBEx, sizeof(CBEx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAEyz, RAEyz, sizeof(RAEyz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBEyz, RBEyz, sizeof(RBEyz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAEzy, RAEzy, sizeof(RAEzy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBEzy, RBEzy, sizeof(RBEzy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }

	cudaStatus = hipMemcpy(dev_CAEy, CAEy, sizeof(CAEy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_CBEy, CBEy, sizeof(CBEy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAExz, RAExz, sizeof(RAExz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBExz, RBExz, sizeof(RBExz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAEzx, RAEzx, sizeof(RAEzx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBEzx, RBEzx, sizeof(RBEzx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }

	cudaStatus = hipMemcpy(dev_CAEz, CAEz, sizeof(CAEz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_CBEz, CBEz, sizeof(CBEz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAExy, RAExy, sizeof(RAExy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBExy, RBExy, sizeof(RBExy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAEyx, RAEyx, sizeof(RAEyx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBEyx, RBEyx, sizeof(RBEyx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }

	cudaStatus = hipMemcpy(dev_CPHx, CPHx, sizeof(CPHx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_CQHx, CQHx, sizeof(CQHx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAHyz, RAHyz, sizeof(RAHyz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBHyz, RBHyz, sizeof(RBHyz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAHzy, RAHzy, sizeof(RAHzy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBHzy, RBHzy, sizeof(RBHzy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }

	cudaStatus = hipMemcpy(dev_CPHy, CPHy, sizeof(CPHy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_CQHy, CQHy, sizeof(CQHy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAHxz, RAHxz, sizeof(RAHxz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBHxz, RBHxz, sizeof(RBHxz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAHzx, RAHzx, sizeof(RAHzx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBHzx, RBHzx, sizeof(RBHzx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }

	cudaStatus = hipMemcpy(dev_CPHz, CPHz, sizeof(CPHz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_CQHz, CQHz, sizeof(CQHz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAHxy, RAHxy, sizeof(RAHxy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBHxy, RBHxy, sizeof(RBHxy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RAHyx, RAHyx, sizeof(RAHyx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_RBHyx, RBHyx, sizeof(RBHyx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }

	cudaStatus = hipMemcpy(dev_kx_Ey, kx_Ey, sizeof(kx_Ey), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_kx_Ez, kx_Ez, sizeof(kx_Ez), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_ky_Ex, ky_Ex, sizeof(ky_Ex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_ky_Ez, ky_Ez, sizeof(ky_Ez), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_kz_Ex, kz_Ex, sizeof(kz_Ex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_kz_Ey, kz_Ey, sizeof(kz_Ey), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }

	cudaStatus = hipMemcpy(dev_kx_Hy, kx_Hy, sizeof(kx_Hy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_kx_Hz, kx_Hz, sizeof(kx_Hz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_ky_Hx, ky_Hx, sizeof(ky_Hx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_ky_Hz, ky_Hz, sizeof(ky_Hz), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_kz_Hx, kz_Hx, sizeof(kz_Hx), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_kz_Hy, kz_Hy, sizeof(kz_Hy), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }

	cudaStatus = hipMemcpy(dev_source, source, sizeof(source), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); goto Error; }
	
	Error:
		return;
}

// 释放显存空间
void gpu_memory_free()
{
	hipFree(dev_Ex);
	hipFree(dev_Ey);
	hipFree(dev_Ez);

	hipFree(dev_UEyz);
	hipFree(dev_UEzy);
	hipFree(dev_UExz);
	hipFree(dev_UEzx);
	hipFree(dev_UExy);
	hipFree(dev_UEyx);

	hipFree(dev_Hx);
	hipFree(dev_Hy);
	hipFree(dev_Hz);

	hipFree(dev_UHyz);
	hipFree(dev_UHzy);
	hipFree(dev_UHxz);
	hipFree(dev_UHzx);
	hipFree(dev_UHxy);
	hipFree(dev_UHyx);

	hipFree(dev_CAEx);
	hipFree(dev_CAEy);
	hipFree(dev_CAEz);

	hipFree(dev_CBEx);
	hipFree(dev_CBEy);
	hipFree(dev_CBEz);

	hipFree(dev_RAEyz);
	hipFree(dev_RAEzy);
	hipFree(dev_RAEzx);
	hipFree(dev_RAExz);
	hipFree(dev_RAExy);
	hipFree(dev_RAEyx);

	hipFree(dev_RBEyz);
	hipFree(dev_RBEzy);
	hipFree(dev_RBEzx);
	hipFree(dev_RBExz);
	hipFree(dev_RBExy);
	hipFree(dev_RBEyx);

	hipFree(dev_CPHx);
	hipFree(dev_CQHx);
	hipFree(dev_CPHy);
	hipFree(dev_CQHy);
	hipFree(dev_CPHz);
	hipFree(dev_CQHz);

	hipFree(dev_RAHyz);
	hipFree(dev_RAHzy);
	hipFree(dev_RAHzx);
	hipFree(dev_RAHxz);
	hipFree(dev_RAHxy);
	hipFree(dev_RAHyx);

	hipFree(dev_RBHyz);
	hipFree(dev_RBHzy);
	hipFree(dev_RBHzx);
	hipFree(dev_RBHxz);
	hipFree(dev_RBHxy);
	hipFree(dev_RBHyx);


	hipFree(fswzx);
	hipFree(fswzy);
	hipFree(fswzz);
	hipFree(jswzx);
	hipFree(jswzy);
	hipFree(jswzz);

	hipFree(dev_E_obs);
	hipFree(dev_V);
	hipFree(dev_source);

	hipFree(dev_kx_Ey);
	hipFree(dev_kx_Ez);
	hipFree(dev_ky_Ex);
	hipFree(dev_ky_Ez);
	hipFree(dev_kz_Ex);
	hipFree(dev_kz_Ey);

	hipFree(dev_kx_Hy);
	hipFree(dev_kx_Hz);
	hipFree(dev_ky_Hx);
	hipFree(dev_ky_Hz);
	hipFree(dev_kz_Hx);
	hipFree(dev_kz_Hy);

	hipFree(dev_Ex_zheng_1);
	hipFree(dev_Ex_zheng_2);
	hipFree(dev_Ex_zheng_3);

	hipFree(dev_Ey_zheng_1);
	hipFree(dev_Ey_zheng_2);
	hipFree(dev_Ey_zheng_3);

	hipFree(dev_Ez_zheng_1);
	hipFree(dev_Ez_zheng_2);
	hipFree(dev_Ez_zheng_3);

	hipFree(dev_Hx_zheng_1);
	hipFree(dev_Hx_zheng_2);
	hipFree(dev_Hx_zheng_3);

	hipFree(dev_Hy_zheng_1);
	hipFree(dev_Hy_zheng_2);
	hipFree(dev_Hy_zheng_3);

	hipFree(dev_Hz_zheng_1);
	hipFree(dev_Hz_zheng_2);
	hipFree(dev_Hz_zheng_3);

	hipFree(dev_Ex_zheng_last);
	hipFree(dev_Ey_zheng_last);
	hipFree(dev_Ez_zheng_last);

	hipFree(dev_Hx_zheng_last);
	hipFree(dev_Hy_zheng_last);
	hipFree(dev_Hz_zheng_last);

	hipFree(dev_fan);
	hipFree(dev_huanyuan);
}

// gpu并行计算UH H UE E
hipError_t gpu_zheng_yan()
{
	hipError_t cudaStatus = hipSuccess;

	calc_UHyz << < gridUHyz, blockUHyz >> > (dev_UHyz, dev_RBHyz, dev_RAHyz, dev_Ez, dy);
	calc_UHzy << < gridUHzy, blockUHzy >> > (dev_UHzy, dev_RBHzy, dev_RAHzy, dev_Ey, dz);
	calc_UHxy << < gridUHxy, blockUHxy >> > (dev_UHxy, dev_RBHxy, dev_RAHxy, dev_Ey, dx);
	calc_UHxz << < gridUHxz, blockUHxz >> > (dev_UHxz, dev_RBHxz, dev_RAHxz, dev_Ez, dx);
	calc_UHyx << < gridUHyx, blockUHyx >> > (dev_UHyx, dev_RBHyx, dev_RAHyx, dev_Ex, dy);
	calc_UHzx << < gridUHzx, blockUHzx >> > (dev_UHzx, dev_RBHzx, dev_RAHzx, dev_Ex, dz);

	calc_Hx << < gridHx, blockHx >> > (dev_Hx, dev_CPHx, dev_CQHx, dev_ky_Hx, dev_kz_Hx, dev_Ez, dev_Ey, dev_UHyz, dev_UHzy, dy, dz);
	calc_Hy << < gridHy, blockHy >> > (dev_Hy, dev_CPHy, dev_CQHy, dev_kz_Hy, dev_kx_Hy, dev_Ex, dev_Ez, dev_UHzx, dev_UHxz, dz, dx);
	calc_Hz << < gridHz, blockHz >> > (dev_Hz, dev_CPHz, dev_CQHz, dev_kx_Hz, dev_ky_Hz, dev_Ey, dev_Ex, dev_UHxy, dev_UHyx, dx, dy);

	calc_UExy << < gridUExy, blockUExy >> > (dev_UExy, dev_RBExy, dev_RAExy, dev_Hy, dx);
	calc_UExz << < gridUExz, blockUExz >> > (dev_UExz, dev_RBExz, dev_RAExz, dev_Hz, dx);
	calc_UEyx << < gridUEyx, blockUEyx >> > (dev_UEyx, dev_RBEyx, dev_RAEyx, dev_Hx, dy);
	calc_UEyz << < gridUEyz, blockUEyz >> > (dev_UEyz, dev_RBEyz, dev_RAEyz, dev_Hz, dy);
	calc_UEzx << < gridUEzx, blockUEzx >> > (dev_UEzx, dev_RBEzx, dev_RAEzx, dev_Hx, dz);
	calc_UEzy << < gridUEzy, blockUEzy >> > (dev_UEzy, dev_RBEzy, dev_RAEzy, dev_Hy, dz);

	calc_Ex << < gridEx, blockEx >> > (dev_Ex, dev_CAEx, dev_CBEx, dev_ky_Ex, dev_kz_Ex, dev_Hz, dev_Hy, dev_UEyz, dev_UEzy, dy, dz);
	calc_Ey << < gridEy, blockEy >> > (dev_Ey, dev_CAEy, dev_CBEy, dev_kz_Ey, dev_kx_Ey, dev_Hx, dev_Hz, dev_UEzx, dev_UExz, dz, dx);
	calc_Ez << < gridEz, blockEz >> > (dev_Ez, dev_CAEz, dev_CBEz, dev_kx_Ez, dev_ky_Ez, dev_Hy, dev_Hx, dev_UExy, dev_UEyx, dx, dy);

	// 计算过程是否出错?
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) { printf("Zhengyan Calc Failed: %s\n", hipGetErrorString(cudaStatus)); return cudaStatus; }
}

hipError_t gpu_parallel_one()
{
	hipError_t cudaStatus;

	int i, j;
	for (i = 0; i < szfsw; i++)
	{
		gpu_memory_set_zero(0);	// flag == 0 将GPU显存中的E*, UE**, H*, UH**, (V, E_obs)置零

		for (j = 0; j < it; j++)
		{
			if (j % 10 == 0)
			{
				printf("i = %3d / %d,  j = %4d / %d\n", i, szfsw, j, it);
			}

			// 实现MATLAB中的Ex[fswzx[i] - 1][fswzy[i] - 1][fswzz[i] - 1] = source[j];
			int fidx = (fswzx[i] - 1)*(ny + 1)*(nz + 1) + (fswzy[i] - 1)*(nz + 1) + fswzz[i] - 1;
			cudaStatus = hipMemcpy(&(dev_Ex[fidx]), &(dev_source[j]), sizeof(float), hipMemcpyDeviceToDevice);

			// 调用GPU运算正演
			gpu_zheng_yan();

			// 实现MATLAB中的V(j)=Ex(jswzx(i), jswzy(i), jswzz(i));
			int jidx = (jswzx[i] - 1)*(ny + 1)*(nz + 1) + (jswzy[i] - 1)*(nz + 1) + jswzz[i] - 1;
			cudaStatus = hipMemcpy(&(dev_V[j]), &(dev_Ex[jidx]), sizeof(float), hipMemcpyDeviceToDevice);
			if (cudaStatus != hipSuccess) { printf("V hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus)); return cudaStatus; };

			cudaStatus = hipMemcpy(&(E_obs[j][i]), &(dev_V[j]), sizeof(float), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) { printf("V hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus)); return cudaStatus; };
		}
	}

	printf("finish calc 1 !\n");

	hipDeviceSynchronize();

	return cudaStatus;
}

hipError_t gpu_parallel_two()
{
	hipError_t cudaStatus;

	int i, j;
	for (i = 0; i < szfsw; i++)
	{
		gpu_memory_set_zero(1); // flag == 1 将GPU显存中的E*, UE**, H*, UH**, (V, E*_zheng_*, H*_zheng_*, E*_zheng_last, H*_zheng_last, fan, huanyuan)置零
		for (j = 0; j < it; j++)
		{
			if (j % 10 == 0) { printf("i = %3d / %d,  j = %4d / %d\n", i, szfsw, j, it); }

			// 实现MATLAB中的Ex[fswzx[i] - 1][fswzy[i] - 1][fswzz[i] - 1] = source[j];
			int fidx = (fswzx[i] - 1)*(ny + 1)*(nz + 1) + (fswzy[i] - 1)*(nz + 1) + fswzz[i] - 1;
			cudaStatus = hipMemcpy(&(dev_Ex[fidx]), &(dev_source[j]), sizeof(float), hipMemcpyDeviceToDevice);

			// 调用GPU运算正演
			gpu_zheng_yan();
			size_t numBytes = (nz-2*npml) * sizeof(float);

			// 复制的块大小 [npml,ny-2*npml,nz-2*npml]
			// Ex_zheng_1(:,:,:,j)=Ex(npml+1:npml+npml      ,npml+1:ny-npml,npml+1:nz-npml);
			// Ex_zheng_1(:,:,:,j)=Ex(nx-npml-npml+1:nx-npml,npml+1:ny-npml,npml+1:nz-npml);
			/*
			__global__ void gpu_copy_data_3D(float *dst, int dst_xsize, int dst_ysize, int dst_zsize, 
											 float *src, int src_xsize, int src_ysize, int src_zsize, 
											 int x, int y, int z, 
											 int x_offset, int y_offset, int z_offst);
											 */
			dim3 blockSize(npml);
			dim3 gridSize(ny-2*npml, nz-2*npml);
			gpu_copy_data_3D << <gridSize, blockSize >> > (dev_Ex_zheng_1 + j * (ny - 2 * npml)*(nz - 2 * npml), 2 * npml, ny - 2 * npml, nz - 2 * npml,
				dev_Ex, nx, ny + 1, nz + 1,
				npml, ny - 2 * npml, nz - 2 * npml,
				npml, npml, npml);

			// 实现MATLAB中的V(j)=Ex(jswzx(i), jswzy(i), jswzz(i));
			//int jidx = (jswzx[i] - 1)*(ny + 1)*(nz + 1) + (jswzy[i] - 1)*(nz + 1) + jswzz[i] - 1;
			//cudaStatus = hipMemcpy(&(dev_V[j]), &(dev_Ex[jidx]), sizeof(float), hipMemcpyDeviceToDevice);
			//if (cudaStatus != hipSuccess) { printf("V hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus)); return cudaStatus; };

			//cudaStatus = hipMemcpy(&(E_obs[j][i]), &(dev_V[j]), sizeof(float), hipMemcpyDeviceToHost);
			//if (cudaStatus != hipSuccess) { printf("V hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus)); return cudaStatus; };
		}
	}

	printf("finish calc !\n");

	hipDeviceSynchronize();

	return cudaStatus;
}

/************************************************************************************
* 主函数
************************************************************************************/
int main()
{
	// 切换工作目录
	//chdir(path);
	//printf("Current Dir: %s \n",getcwd(NULL，NULL));
	if (Hz_zheng_3 == NULL)
	{
		printf("malloc failed! \n");
		return 1;
	}
	else
	{
		printf("addr of Hz_zheng_3 is %p\n",Hz_zheng_3);
	}
	// 从matlab输出的文本文件中读取数据
	read_data_from_txt();
	printf("Read Data From Txt OK ! \n");

	// 选择运算使用的GPU
	hipError_t cudaStatus = hipSetDevice(cudaDevice);
	if (cudaStatus != hipSuccess) { printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?"); return 1; }
	else { printf("hipSetDevice success!\n"); }

	// 分配显存，把数据从内存传输到显存
	gpu_memory_malloc();
	gpu_memory_copy();

	// 调用gpu运算
	cudaStatus = gpu_parallel_two();
	if (cudaStatus != hipSuccess) { printf("gpu_parallel_two failed!"); return 1; }
	else { printf("gpu_parallel_two success!\n"); }

	// 释放显存空间
	gpu_memory_free();

	// 重置GPU
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) { printf("hipDeviceReset failed!"); return 1; }

	// 输出结果
	print_E_obs();

	// 释放内存
	freeMemory();
	return 0;
}